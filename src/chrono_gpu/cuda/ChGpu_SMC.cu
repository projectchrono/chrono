#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Conlain Kelly, Nic Olsen, Dan Negrut, Ruochun Zhang
// =============================================================================

#include <cmath>
#include <numeric>

#include "chrono_gpu/cuda/ChGpu_SMC.cuh"
#include "chrono_gpu/utils/ChGpuUtilities.h"

namespace chrono {
namespace gpu {

__host__ float ChSystemGpu_impl::computeArray3SquaredSum(std::vector<float, cudallocator<float>>& arrX,
                                                         std::vector<float, cudallocator<float>>& arrY,
                                                         std::vector<float, cudallocator<float>>& arrZ,
                                                         size_t nSpheres) {
    const unsigned int threadsPerBlock = 1024;
    unsigned int nBlocks = (nSpheres + threadsPerBlock - 1) / threadsPerBlock;
    elementalArray3Squared<float><<<nBlocks, threadsPerBlock>>>(sphere_data->sphere_stats_buffer, arrX.data(),
                                                                arrY.data(), arrZ.data(), nSpheres);
    gpuErrchk(hipDeviceSynchronize());

    // Use CUB to reduce. And put the reduced result at the last element of sphere_stats_buffer array.
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(NULL, temp_storage_bytes, sphere_data->sphere_stats_buffer,
                           sphere_data->sphere_stats_buffer + nSpheres, nSpheres);
    void* d_scratch_space = (void*)stateOfSolver_resources.pDeviceMemoryScratchSpace(temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_scratch_space, temp_storage_bytes, sphere_data->sphere_stats_buffer,
                           sphere_data->sphere_stats_buffer + nSpheres, nSpheres);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    return *(sphere_data->sphere_stats_buffer + nSpheres);
}

__host__ double ChSystemGpu_impl::GetMaxParticleZ(bool getMax) {
    size_t nSpheres = sphere_local_pos_Z.size();
    if (nSpheres == 0)
        CHGPU_ERROR("ERROR! 0 particle in system! Please call this method after Initialize().\n");

    const unsigned int threadsPerBlock = 1024;
    unsigned int nBlocks = (nSpheres + threadsPerBlock - 1) / threadsPerBlock;
    elementalZLocalToGlobal<<<nBlocks, threadsPerBlock>>>(sphere_data->sphere_stats_buffer, sphere_data, nSpheres,
                                                          gran_params);
    gpuErrchk(hipDeviceSynchronize());

    // Use CUB to find the max or min Z.
    size_t temp_storage_bytes = 0;
    if (getMax) {
        hipcub::DeviceReduce::Max(NULL, temp_storage_bytes, sphere_data->sphere_stats_buffer,
                               sphere_data->sphere_stats_buffer + nSpheres, nSpheres);
        void* d_scratch_space = (void*)stateOfSolver_resources.pDeviceMemoryScratchSpace(temp_storage_bytes);
        hipcub::DeviceReduce::Max(d_scratch_space, temp_storage_bytes, sphere_data->sphere_stats_buffer,
                               sphere_data->sphere_stats_buffer + nSpheres, nSpheres);
    } else {
        hipcub::DeviceReduce::Min(NULL, temp_storage_bytes, sphere_data->sphere_stats_buffer,
                               sphere_data->sphere_stats_buffer + nSpheres, nSpheres);
        void* d_scratch_space = (void*)stateOfSolver_resources.pDeviceMemoryScratchSpace(temp_storage_bytes);
        hipcub::DeviceReduce::Min(d_scratch_space, temp_storage_bytes, sphere_data->sphere_stats_buffer,
                               sphere_data->sphere_stats_buffer + nSpheres, nSpheres);
    }
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    return *(sphere_data->sphere_stats_buffer + nSpheres);
}

// Reset broadphase data structures
void ChSystemGpu_impl::resetBroadphaseInformation() {
    // Set all the offsets to zero
    gpuErrchk(hipMemset(SD_NumSpheresTouching.data(), 0, SD_NumSpheresTouching.size() * sizeof(unsigned int)));
    gpuErrchk(hipMemset(SD_SphereCompositeOffsets.data(), 0, SD_SphereCompositeOffsets.size() * sizeof(unsigned int)));
    // For each SD, all the spheres touching that SD should have their ID be NULL_CHGPU_ID
    gpuErrchk(hipMemset(spheres_in_SD_composite.data(), NULL_CHGPU_ID,
                         spheres_in_SD_composite.size() * sizeof(unsigned int)));
    gpuErrchk(hipDeviceSynchronize());
}

// Reset sphere acceleration data structures
void ChSystemGpu_impl::resetSphereAccelerations() {
    // cache past acceleration data
    if (time_integrator == CHGPU_TIME_INTEGRATOR::CHUNG) {
        gpuErrchk(hipMemcpy(sphere_acc_X_old.data(), sphere_acc_X.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_acc_Y_old.data(), sphere_acc_Y.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_acc_Z_old.data(), sphere_acc_Z.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        // if we have multistep AND friction, cache old alphas
        if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
            gpuErrchk(hipMemcpy(sphere_ang_acc_X_old.data(), sphere_ang_acc_X.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
            gpuErrchk(hipMemcpy(sphere_ang_acc_Y_old.data(), sphere_ang_acc_Y.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
            gpuErrchk(hipMemcpy(sphere_ang_acc_Z_old.data(), sphere_ang_acc_Z.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
        }
        gpuErrchk(hipDeviceSynchronize());
    }

    // reset current accelerations to zero to zero
    gpuErrchk(hipMemset(sphere_acc_X.data(), 0, nSpheres * sizeof(float)));
    gpuErrchk(hipMemset(sphere_acc_Y.data(), 0, nSpheres * sizeof(float)));
    gpuErrchk(hipMemset(sphere_acc_Z.data(), 0, nSpheres * sizeof(float)));

    // reset torques to zero, if applicable
    if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
        gpuErrchk(hipMemset(sphere_ang_acc_X.data(), 0, nSpheres * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Y.data(), 0, nSpheres * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Z.data(), 0, nSpheres * sizeof(float)));
    }
}

__global__ void compute_absv(const unsigned int nSpheres,
                             const float* velX,
                             const float* velY,
                             const float* velZ,
                             float* d_absv) {
    unsigned int my_sphere = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_sphere < nSpheres) {
        float v[3] = {velX[my_sphere], velY[my_sphere], velZ[my_sphere]};
        d_absv[my_sphere] = sqrt(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);
    }
}

__host__ float ChSystemGpu_impl::get_max_vel() const {
    float* d_absv;
    float* d_max_vel;
    float h_max_vel;
    gpuErrchk(hipMalloc(&d_absv, nSpheres * sizeof(float)));
    gpuErrchk(hipMalloc(&d_max_vel, sizeof(float)));

    compute_absv<<<(nSpheres + 255) / 256, 256>>>(nSpheres, pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), d_absv);

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nSpheres);
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nSpheres);
    gpuErrchk(hipMemcpy(&h_max_vel, d_max_vel, sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_absv));
    gpuErrchk(hipFree(d_max_vel));

    return h_max_vel;
}

__host__ int3 ChSystemGpu_impl::getSDTripletFromID(unsigned int SD_ID) const {
    return SDIDTriplet(SD_ID, gran_params);
}
/// Sort sphere positions by subdomain id
/// Occurs entirely on host, not intended to be efficient
/// ONLY DO AT BEGINNING OF SIMULATION
__host__ void ChSystemGpu_impl::defragment_initial_positions() {
    // key and value pointers
    std::vector<unsigned int, cudallocator<unsigned int>> sphere_ids;

    // load sphere indices
    sphere_ids.resize(nSpheres);
    std::iota(sphere_ids.begin(), sphere_ids.end(), 0);

    // sort sphere ids by owner SD
    std::sort(sphere_ids.begin(), sphere_ids.end(),
              [&](std::size_t i, std::size_t j) { return sphere_owner_SDs.at(i) < sphere_owner_SDs.at(j); });

    std::vector<int, cudallocator<int>> sphere_pos_x_tmp;
    std::vector<int, cudallocator<int>> sphere_pos_y_tmp;
    std::vector<int, cudallocator<int>> sphere_pos_z_tmp;

    std::vector<float, cudallocator<float>> sphere_vel_x_tmp;
    std::vector<float, cudallocator<float>> sphere_vel_y_tmp;
    std::vector<float, cudallocator<float>> sphere_vel_z_tmp;

    std::vector<float, cudallocator<float>> sphere_angv_x_tmp;
    std::vector<float, cudallocator<float>> sphere_angv_y_tmp;
    std::vector<float, cudallocator<float>> sphere_angv_z_tmp;

    std::vector<not_stupid_bool, cudallocator<not_stupid_bool>> sphere_fixed_tmp;
    std::vector<unsigned int, cudallocator<unsigned int>> sphere_owner_SDs_tmp;

    sphere_pos_x_tmp.resize(nSpheres);
    sphere_pos_y_tmp.resize(nSpheres);
    sphere_pos_z_tmp.resize(nSpheres);

    sphere_vel_x_tmp.resize(nSpheres);
    sphere_vel_y_tmp.resize(nSpheres);
    sphere_vel_z_tmp.resize(nSpheres);

    if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
        sphere_angv_x_tmp.resize(nSpheres);
        sphere_angv_y_tmp.resize(nSpheres);
        sphere_angv_z_tmp.resize(nSpheres);
    }

    sphere_fixed_tmp.resize(nSpheres);
    sphere_owner_SDs_tmp.resize(nSpheres);

    // reorder values into new sorted
    for (unsigned int i = 0; i < nSpheres; i++) {
        sphere_pos_x_tmp.at(i) = sphere_local_pos_X.at(sphere_ids.at(i));
        sphere_pos_y_tmp.at(i) = sphere_local_pos_Y.at(sphere_ids.at(i));
        sphere_pos_z_tmp.at(i) = sphere_local_pos_Z.at(sphere_ids.at(i));

        sphere_vel_x_tmp.at(i) = (float)pos_X_dt.at(sphere_ids.at(i));
        sphere_vel_y_tmp.at(i) = (float)pos_Y_dt.at(sphere_ids.at(i));
        sphere_vel_z_tmp.at(i) = (float)pos_Z_dt.at(sphere_ids.at(i));

        if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
            sphere_angv_x_tmp.at(i) = (float)sphere_Omega_X.at(sphere_ids.at(i));
            sphere_angv_y_tmp.at(i) = (float)sphere_Omega_Y.at(sphere_ids.at(i));
            sphere_angv_z_tmp.at(i) = (float)sphere_Omega_Z.at(sphere_ids.at(i));
        }

        sphere_fixed_tmp.at(i) = sphere_fixed.at(sphere_ids.at(i));
        sphere_owner_SDs_tmp.at(i) = sphere_owner_SDs.at(sphere_ids.at(i));
    }

    // swap into the correct data structures
	sphere_local_pos_X.swap(sphere_pos_x_tmp);
    sphere_local_pos_Y.swap(sphere_pos_y_tmp);
    sphere_local_pos_Z.swap(sphere_pos_z_tmp);

    pos_X_dt.swap(sphere_vel_x_tmp);
    pos_Y_dt.swap(sphere_vel_y_tmp);
    pos_Z_dt.swap(sphere_vel_z_tmp);

    if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
        sphere_Omega_X.swap(sphere_angv_x_tmp);
        sphere_Omega_Y.swap(sphere_angv_y_tmp);
        sphere_Omega_Z.swap(sphere_angv_z_tmp);
    }

    sphere_fixed.swap(sphere_fixed_tmp);
    sphere_owner_SDs.swap(sphere_owner_SDs_tmp);
}

/// Same defragment function, but this time for the contact friction history arrays.
/// It is stand-alone because it should rarely be needed, so let us save some time by
/// not calling it in most of our simulations.
__host__ void ChSystemGpu_impl::defragment_friction_history(unsigned int history_offset) {
    // key and value pointers
    std::vector<unsigned int, cudallocator<unsigned int>> sphere_ids;

    // load sphere indices
    sphere_ids.resize(nSpheres);
    std::iota(sphere_ids.begin(), sphere_ids.end(), 0);

    // sort sphere ids by owner SD
    std::sort(sphere_ids.begin(), sphere_ids.end(),
              [&](std::size_t i, std::size_t j) { return sphere_owner_SDs.at(i) < sphere_owner_SDs.at(j); });

    std::vector<float3, cudallocator<float3>> history_tmp;
    std::vector<unsigned int, cudallocator<unsigned int>> partners_tmp;

    history_tmp.resize(history_offset * nSpheres);
    partners_tmp.resize(history_offset * nSpheres);

    // reorder values into new sorted
    for (unsigned int i = 0; i < nSpheres; i++) {
        for (unsigned int j = 0; j < history_offset; j++) {
            history_tmp.at(history_offset * i + j) = contact_history_map.at(history_offset * sphere_ids.at(i) + j);
            partners_tmp.at(history_offset * i + j) = contact_partners_map.at(history_offset * sphere_ids.at(i) + j);
        }
    }

    contact_history_map.swap(history_tmp);
    contact_partners_map.swap(partners_tmp);
}

__host__ void ChSystemGpu_impl::setupSphereDataStructures() {
    // Each fills user_sphere_positions with positions to be copied
    if (user_sphere_positions.size() == 0) {
        CHGPU_ERROR("ERROR! no sphere positions given!\n");
    }

    nSpheres = (unsigned int)user_sphere_positions.size();
    INFO_PRINTF("%u balls added!\n", nSpheres);
    gran_params->nSpheres = nSpheres;

    TRACK_VECTOR_RESIZE(sphere_owner_SDs, nSpheres, "sphere_owner_SDs", NULL_CHGPU_ID);

    // Allocate space for new bodies
    TRACK_VECTOR_RESIZE(sphere_local_pos_X, nSpheres, "sphere_local_pos_X", 0);
    TRACK_VECTOR_RESIZE(sphere_local_pos_Y, nSpheres, "sphere_local_pos_Y", 0);
    TRACK_VECTOR_RESIZE(sphere_local_pos_Z, nSpheres, "sphere_local_pos_Z", 0);

    TRACK_VECTOR_RESIZE(sphere_fixed, nSpheres, "sphere_fixed", 0);

    TRACK_VECTOR_RESIZE(pos_X_dt, nSpheres, "pos_X_dt", 0);
    TRACK_VECTOR_RESIZE(pos_Y_dt, nSpheres, "pos_Y_dt", 0);
    TRACK_VECTOR_RESIZE(pos_Z_dt, nSpheres, "pos_Z_dt", 0);

    // temporarily store global positions as 64-bit, discard as soon as local positions are loaded
    {
        bool user_provided_fixed = user_sphere_fixed.size() != 0;
        bool user_provided_vel = user_sphere_vel.size() != 0;
        if (user_provided_fixed && user_sphere_fixed.size() != nSpheres)
            CHGPU_ERROR("Provided fixity array has length %zu, but there are %u spheres!\n", user_sphere_fixed.size(),
                        nSpheres);
        if (user_provided_vel && user_sphere_vel.size() != nSpheres)
            CHGPU_ERROR("Provided velocity array has length %zu, but there are %u spheres!\n", user_sphere_vel.size(),
                        nSpheres);

        std::vector<int64_t, cudallocator<int64_t>> sphere_global_pos_X;
        std::vector<int64_t, cudallocator<int64_t>> sphere_global_pos_Y;
        std::vector<int64_t, cudallocator<int64_t>> sphere_global_pos_Z;

        sphere_global_pos_X.resize(nSpheres);
        sphere_global_pos_Y.resize(nSpheres);
        sphere_global_pos_Z.resize(nSpheres);

        // Copy from array of structs to 3 arrays
        for (unsigned int i = 0; i < nSpheres; i++) {
            float3 vec = user_sphere_positions.at(i);
            // cast to double, convert to SU, then cast to int64_t
            sphere_global_pos_X.at(i) = (int64_t)((double)vec.x / LENGTH_SU2UU);
            sphere_global_pos_Y.at(i) = (int64_t)((double)vec.y / LENGTH_SU2UU);
            sphere_global_pos_Z.at(i) = (int64_t)((double)vec.z / LENGTH_SU2UU);

            // Convert to not_stupid_bool
            sphere_fixed.at(i) = (not_stupid_bool)((user_provided_fixed) ? user_sphere_fixed[i] : false);
            if (user_provided_vel) {
                auto vel = user_sphere_vel.at(i);
                pos_X_dt.at(i) = (float)(vel.x / VEL_SU2UU);
                pos_Y_dt.at(i) = (float)(vel.y / VEL_SU2UU);
                pos_Z_dt.at(i) = (float)(vel.z / VEL_SU2UU);
            }
        }

        packSphereDataPointers();
        // Figure our the number of blocks that need to be launched to cover the box
        unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
        initializeLocalPositions<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
            sphere_data, sphere_global_pos_X.data(), sphere_global_pos_Y.data(), sphere_global_pos_Z.data(), nSpheres,
            gran_params);

        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());
    }

    TRACK_VECTOR_RESIZE(sphere_acc_X, nSpheres, "sphere_acc_X", 0);
    TRACK_VECTOR_RESIZE(sphere_acc_Y, nSpheres, "sphere_acc_Y", 0);
    TRACK_VECTOR_RESIZE(sphere_acc_Z, nSpheres, "sphere_acc_Z", 0);

    // The buffer array that stores any quantity that the user wish to quarry. We resize it here once instead of
    // resizing on-the-call, to save time, in case that quarry function is called with a high frequency. The last
    // element in this array is to store the reduced value.
    TRACK_VECTOR_RESIZE(sphere_stats_buffer, nSpheres + 1, "sphere_stats_buffer", 0);

    // NOTE that this will get resized again later, this is just the first estimate
    TRACK_VECTOR_RESIZE(spheres_in_SD_composite, 2 * nSpheres, "spheres_in_SD_composite", NULL_CHGPU_ID);

    if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
        // add rotational DOFs
        TRACK_VECTOR_RESIZE(sphere_Omega_X, nSpheres, "sphere_Omega_X", 0);
        TRACK_VECTOR_RESIZE(sphere_Omega_Y, nSpheres, "sphere_Omega_Y", 0);
        TRACK_VECTOR_RESIZE(sphere_Omega_Z, nSpheres, "sphere_Omega_Z", 0);

        // add torques
        TRACK_VECTOR_RESIZE(sphere_ang_acc_X, nSpheres, "sphere_ang_acc_X", 0);
        TRACK_VECTOR_RESIZE(sphere_ang_acc_Y, nSpheres, "sphere_ang_acc_Y", 0);
        TRACK_VECTOR_RESIZE(sphere_ang_acc_Z, nSpheres, "sphere_ang_acc_Z", 0);

        {
            bool user_provided_ang_vel = user_sphere_ang_vel.size() != 0;
            if (user_provided_ang_vel && user_sphere_ang_vel.size() != nSpheres)
                CHGPU_ERROR("Provided angular velocity array has length %zu, but there are %u spheres!\n",
                            user_sphere_ang_vel.size(), nSpheres);
            if (user_provided_ang_vel) {
                for (unsigned int i = 0; i < nSpheres; i++) {
                    auto ang_vel = user_sphere_ang_vel.at(i);
                    sphere_Omega_X.at(i) = (float)(ang_vel.x * TIME_SU2UU);
                    sphere_Omega_Y.at(i) = (float)(ang_vel.y * TIME_SU2UU);
                    sphere_Omega_Z.at(i) = (float)(ang_vel.z * TIME_SU2UU);
                }
            }
        }
    }

    if (time_integrator == CHGPU_TIME_INTEGRATOR::CHUNG) {
        TRACK_VECTOR_RESIZE(sphere_acc_X_old, nSpheres, "sphere_acc_X_old", 0);
        TRACK_VECTOR_RESIZE(sphere_acc_Y_old, nSpheres, "sphere_acc_Y_old", 0);
        TRACK_VECTOR_RESIZE(sphere_acc_Z_old, nSpheres, "sphere_acc_Z_old", 0);

        // friction and multistep means keep old ang acc
        if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
            TRACK_VECTOR_RESIZE(sphere_ang_acc_X_old, nSpheres, "sphere_ang_acc_X_old", 0);
            TRACK_VECTOR_RESIZE(sphere_ang_acc_Y_old, nSpheres, "sphere_ang_acc_Y_old", 0);
            TRACK_VECTOR_RESIZE(sphere_ang_acc_Z_old, nSpheres, "sphere_ang_acc_Z_old", 0);
        }
    }

    // If this is a new-boot, we usually want to do this defragment.
    // But if this is a restart, then probably no. We do not want every time the simulation restarts,
    // we have the order of particles completely changed: it may be bad for visualization or debugging
	if (defragment_on_start) {
        defragment_initial_positions();
    }

    bool user_provided_internal_data = false;
    if (gran_params->friction_mode == CHGPU_FRICTION_MODE::MULTI_STEP ||
        gran_params->friction_mode == CHGPU_FRICTION_MODE::SINGLE_STEP) {
        TRACK_VECTOR_RESIZE(contact_partners_map, MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres, "contact_partners_map",
                            NULL_CHGPU_ID);
        TRACK_VECTOR_RESIZE(contact_active_map, MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres, "contact_active_map", false);

        // If the user provides a checkpointed history array, we load it here
        bool user_provided_partner_map = user_partner_map.size() != 0;
        if (user_provided_partner_map && user_partner_map.size() != MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres)
            CHGPU_ERROR("ERROR! The user provided contact partner map has size %zu. It needs to be %u * %u!\n",
                        user_partner_map.size(), MAX_SPHERES_TOUCHED_BY_SPHERE, nSpheres);

        // Hope that using .at (instead of []) gives better err msg when things go wrong,
        // at the cost of some speed which is not important in I/O
        if (user_provided_partner_map) {
            for (unsigned int i = 0; i < nSpheres; i++) {
                for (unsigned int j = 0; j < MAX_SPHERES_TOUCHED_BY_SPHERE; j++) {
                    contact_partners_map.at(MAX_SPHERES_TOUCHED_BY_SPHERE * i + j) =
                        user_partner_map.at(MAX_SPHERES_TOUCHED_BY_SPHERE * i + j);
                }
            }
        }

        user_provided_internal_data = user_provided_internal_data || user_provided_partner_map;
    }

    if (gran_params->friction_mode == CHGPU_FRICTION_MODE::MULTI_STEP) {
        float3 null_history = {0., 0., 0.};
        TRACK_VECTOR_RESIZE(contact_history_map, MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres, "contact_history_map",
                            null_history);

        // If the user provides a checkpointed history array, we load it here
        bool user_provided_friction_history = user_friction_history.size() != 0;
        if (user_provided_friction_history && user_friction_history.size() != MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres)
            CHGPU_ERROR("ERROR! The user provided contact friction history has size %zu. It needs to be %u * %u!\n",
                        user_friction_history.size(), MAX_SPHERES_TOUCHED_BY_SPHERE, nSpheres);

        if (user_provided_friction_history) {
            for (unsigned int i = 0; i < nSpheres; i++) {
                for (unsigned int j = 0; j < MAX_SPHERES_TOUCHED_BY_SPHERE; j++) {
                    float3 history_UU = user_friction_history[MAX_SPHERES_TOUCHED_BY_SPHERE * i + j];
                    float3 history_SU = make_float3(history_UU.x / LENGTH_SU2UU, history_UU.y / LENGTH_SU2UU,
                                                    history_UU.z / LENGTH_SU2UU);
                    contact_history_map.at(MAX_SPHERES_TOUCHED_BY_SPHERE * i + j) = history_SU;
                }
            }
        }

        user_provided_internal_data = user_provided_internal_data || user_provided_friction_history;
    }

    // This if content should be executed rarely, if at all.
    // If user gives Chrono::Gpu internal data from a file then it's a restart,
    // then defragment_on_start should be set to false. But I implemented it anyway.
    if (user_provided_internal_data && defragment_on_start) {
        defragment_friction_history(MAX_SPHERES_TOUCHED_BY_SPHERE);
    }

    // record normal contact force
    if (gran_params->recording_contactInfo == true) {
        float3 null_force = {0.0f, 0.0f, 0.0f};
        TRACK_VECTOR_RESIZE(normal_contact_force, MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres, "normal contact force",
                            null_force);
    }

    // record friction force
    if (gran_params->recording_contactInfo == true && gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
        float3 null_force = {0.0f, 0.0f, 0.0f};
        TRACK_VECTOR_RESIZE(tangential_friction_force, MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres,
                            "tangential contact force", null_force);
    }

    // record rolling friction torque
    if (gran_params->recording_contactInfo == true && gran_params->rolling_mode != CHGPU_ROLLING_MODE::NO_RESISTANCE) {
        float3 null_force = {0.0f, 0.0f, 0.0f};
        TRACK_VECTOR_RESIZE(rolling_friction_torque, MAX_SPHERES_TOUCHED_BY_SPHERE * nSpheres,
                            "rolling friction torque", null_force);
    }

    // make sure the right pointers are packed
    packSphereDataPointers();
}

/// <summary>
/// runSphereBroadphase goes through three stages. First, a kernel figures out for each SD, how many spheres touch it.
/// Then, there is a prefix scan done (which requires two CUB function calls) to figure out offsets into the big fat
/// array that contains, for SD after SD, which spheres touch the SD. This last thing is accomplished by a kernel call.
/// 
/// CAVEAT: in this approach, the outcome of the prefix scan operation will be canibalized during the kernel call that
/// updates the big fat composite array. As such, there is a "scratch-pad" version that is used along the way
/// </summary>
/// <returns></returns>
__host__ void ChSystemGpu_impl::runSphereBroadphase() {
    METRICS_PRINTF("Resetting broadphase info!\n");

    // reset the number of spheres per SD, the offsets in the big composite array, and the big fat composite array
    resetBroadphaseInformation();

    // Frist stage of the computation in this function: Figure out the how many spheres touch each SD. 
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
    getNumberOfSpheresTouchingEachSD<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nSpheres, gran_params);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    // Starting the second stage of this function call - the prefix scan operation
    unsigned int* out_ptr = SD_SphereCompositeOffsets.data();
    unsigned int* in_ptr = SD_NumSpheresTouching.data();
    gpuErrchk(hipMemcpy(out_ptr, in_ptr, nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));

    // cold run; CUB determines the amount of storage it needs (since first argument is NULL pointer)
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(NULL, temp_storage_bytes, in_ptr, out_ptr, nSDs);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
   
    // give CUB needed temporary storage on the device
    void* d_scratch_space = (void*)stateOfSolver_resources.pDeviceMemoryScratchSpace(temp_storage_bytes);
    // Run the actual exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_scratch_space, temp_storage_bytes, in_ptr, out_ptr, nSDs);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    // Beginning of the last stage of computation in this function: assembling the big composite array.
    // num_entries: total number of sphere entries to record in the big fat composite array
    unsigned int num_entries = out_ptr[nSDs - 1] + in_ptr[nSDs - 1];
    spheres_in_SD_composite.resize(num_entries, NULL_CHGPU_ID);
    sphere_data->spheres_in_SD_composite = spheres_in_SD_composite.data();

    // Copy the offesets in the scratch pad; the subsequent kernel call would step on the outcome of the prefix scan
    gpuErrchk(hipMemcpy(SD_SphereCompositeOffsets_ScratchPad.data(), SD_SphereCompositeOffsets.data(),
                         nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    // Populate the composite array; in the process, the content of the scratch pad will be modified
    // nBlocks = (MAX_SDs_TOUCHED_BY_SPHERE * nSpheres + 2*CUDA_THREADS_PER_BLOCK - 1) / (2*CUDA_THREADS_PER_BLOCK);
    // populateSpheresInEachSD<<<nBlocks, 2*CUDA_THREADS_PER_BLOCK>>>(sphere_data, nSpheres, gran_params);
    nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / (CUDA_THREADS_PER_BLOCK);
    populateSpheresInEachSD<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nSpheres, gran_params);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
}


__host__ void ChSystemGpu_impl::updateBCPositions() {
    for (unsigned int i = 0; i < BC_params_list_UU.size(); i++) {
        auto bc_type = BC_type_list.at(i);
        const BC_params_t<float, float3>& params_UU = BC_params_list_UU.at(i);
        BC_params_t<int64_t, int64_t3>& params_SU = BC_params_list_SU.at(i);
        auto offset_function = BC_offset_function_list.at(i);
        setBCOffset(bc_type, params_UU, params_SU, offset_function(elapsedSimTime));
    }

    if (!BD_is_fixed) {
        double3 new_BD_offset = BDOffsetFunction(elapsedSimTime);

        int64_t3 bd_offset_SU = {0, 0, 0};
        bd_offset_SU.x = (int64_t)(new_BD_offset.x / LENGTH_SU2UU);
        bd_offset_SU.y = (int64_t)(new_BD_offset.y / LENGTH_SU2UU);
        bd_offset_SU.z = (int64_t)(new_BD_offset.z / LENGTH_SU2UU);

        int64_t old_frame_X = gran_params->BD_frame_X;
        int64_t old_frame_Y = gran_params->BD_frame_Y;
        int64_t old_frame_Z = gran_params->BD_frame_Z;

        gran_params->BD_frame_X = bd_offset_SU.x + BD_rest_frame_SU.x;
        gran_params->BD_frame_Y = bd_offset_SU.y + BD_rest_frame_SU.y;
        gran_params->BD_frame_Z = bd_offset_SU.z + BD_rest_frame_SU.z;

        unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

        int64_t3 offset_delta = {0, 0, 0};

        // if the frame X increases, the local X should decrease
        offset_delta.x = old_frame_X - gran_params->BD_frame_X;
        offset_delta.y = old_frame_Y - gran_params->BD_frame_Y;
        offset_delta.z = old_frame_Z - gran_params->BD_frame_Z;

        // printf("offset is %lld, %lld, %lld\n", offset_delta.x, offset_delta.y, offset_delta.z);

        packSphereDataPointers();

        applyBDFrameChange<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(offset_delta, sphere_data, nSpheres, gran_params);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }
}

__host__ double ChSystemGpu_impl::AdvanceSimulation(float duration) {
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Settling simulation loop.
    float duration_SU = (float)(duration / TIME_SU2UU);
    unsigned int nsteps = (unsigned int)std::round(duration_SU / stepSize_SU);

    METRICS_PRINTF("advancing by %f at timestep %f, %u timesteps at approx user timestep %f\n", duration_SU,
                   stepSize_SU, nsteps, duration / nsteps);
    float time_elapsed_SU = 0;  // time elapsed in this advance call

    packSphereDataPointers();
    // Run the simulation, there are aggressive synchronizations because we want to have no race conditions
    for (unsigned int n = 0; n < nsteps; n++) {
        updateBCPositions();
        runSphereBroadphase();
        resetSphereAccelerations();
        resetBCForces();

        METRICS_PRINTF("Starting computeSphereForces!\n");

        if (gran_params->friction_mode == CHGPU_FRICTION_MODE::FRICTIONLESS) {
            // Compute sphere-sphere forces
            computeSphereForces_frictionless<<<nSDs, MAX_COUNT_OF_SPHERES_PER_SD>>>(
                sphere_data, gran_params, BC_type_list.data(), BC_params_list_SU.data(),
                (unsigned int)BC_params_list_SU.size());
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        } else if (gran_params->friction_mode == CHGPU_FRICTION_MODE::SINGLE_STEP ||
                   gran_params->friction_mode == CHGPU_FRICTION_MODE::MULTI_STEP) {
            // figure out who is contacting
            determineContactPairs<<<nSDs, MAX_COUNT_OF_SPHERES_PER_SD>>>(sphere_data, gran_params);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());

            computeSphereContactForces<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
                sphere_data, gran_params, BC_type_list.data(), BC_params_list_SU.data(),
                (unsigned int)BC_params_list_SU.size(), nSpheres);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }

        METRICS_PRINTF("Starting integrateSpheres!\n");
        integrateSpheres<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(stepSize_SU, sphere_data, nSpheres, gran_params);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        if (gran_params->friction_mode != CHGPU_FRICTION_MODE::FRICTIONLESS) {
            const unsigned int nThreadsUpdateHist = 2 * CUDA_THREADS_PER_BLOCK;
            unsigned int fricMapSize = nSpheres * MAX_SPHERES_TOUCHED_BY_SPHERE;
            unsigned int nBlocksFricHistoryPostProcess = (fricMapSize + nThreadsUpdateHist - 1) / nThreadsUpdateHist;
            updateFrictionData<<<nBlocksFricHistoryPostProcess, nThreadsUpdateHist>>>(fricMapSize, sphere_data,
                                                                                      gran_params);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
            updateAngVels<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(stepSize_SU, sphere_data, nSpheres, gran_params);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }

        elapsedSimTime += (float)(stepSize_SU * TIME_SU2UU);  // Advance current time
        time_elapsed_SU += stepSize_SU;
    }

    return time_elapsed_SU * TIME_SU2UU;  // return elapsed UU time
}
}  // namespace gpu
}  // namespace chrono
