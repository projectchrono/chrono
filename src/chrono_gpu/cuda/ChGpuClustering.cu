#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
//
// Contains functions that separated soil particles into clusters
// Reference: Andrade, Guilherme, et al. "G-dbscan: A gpu accelerated algorithm
// for density-based clustering." Procedia Computer Science 18 (2013): 369-378.
// =============================================================================
// Authors: Gabriel Taillon
// =============================================================================

#include "chrono_gpu/ChGpuDefines.h"
#include "chrono_gpu/physics/ChSystemGpu_impl.h"
#include "chrono_gpu/cuda/ChCudaMathUtils.cuh"
#include "chrono_gpu/cuda/ChGpuHelpers.cuh"
#include "chrono_gpu/cuda/ChGpu_SMC.cuh"
#include "chrono_gpu/cuda/ChGpuClustering.cuh"

/// Identifies all clusters by breadth first search and outputs
/// Returns h_clusters: array of pointers to arrays of variable length
/// h_clusters[0][0] -> number of pointers/cluster in h_clusters
/// h_clusters[M][0] -> size of the Mth cluster
/// h_clusters[M][N] -> Nth point in Mth cluster
/// at worst, there will be nSpheres h_clusters
static __host__ unsigned int ** ClusterSearchBFS(unsigned int nSpheres,
                                                 ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                                 unsigned int* adj_num,
                                                 unsigned int* adj_offset,
                                                 unsigned int* adj_list,
                                                 SPHERE_TYPE* sphere_type) {
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    hipMemset(sphere_data->sphere_cluster,
               static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::GROUND),
               sizeof(*sphere_data->sphere_cluster) * nSpheres);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    unsigned int ** h_clusters; /* all identified clusters */
    unsigned int * h_cluster;
    unsigned int h_cluster_num = 0;
    h_clusters = (unsigned int **)malloc(sizeof(*h_clusters) * (nSpheres+1));
    h_clusters[0] = (unsigned int *)malloc(sizeof(**h_clusters));
    h_clusters[0][0] = h_cluster_num;  // number of h_clusters

    // border_num: number of remaining border vertices to search in BFS_kernel
    unsigned int * d_border_num;
    gpuErrchk(hipMalloc((void**)&d_border_num, sizeof(*d_border_num)));
    unsigned int * h_border_num = (unsigned int *)malloc(sizeof(*h_border_num)); 
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    bool * d_borders;  // [mySphereID] -> is vertex a border?
    bool * d_visited;  // [mySphereID] -> was vertex d_visited in BFS_kernel?
    bool * h_visited;  // [mySphereID] -> host of d_visited
    bool * h_searched;  // [mySphereID] -> was vertex searched before?
    gpuErrchk(hipMalloc((void**)&d_borders, sizeof(*d_borders) * nSpheres));
    gpuErrchk(hipMalloc((void**)&d_visited, sizeof(*d_visited) * nSpheres));
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    h_visited = (bool *)calloc(sizeof(*h_visited), nSpheres);
    h_searched = (bool *)calloc(sizeof(*h_searched), nSpheres);
    SPHERE_TYPE h_current_type;

    // Search for clusters at every sphere.
    for (size_t i = 0; i < nSpheres; i++) {
        h_current_type = sphere_type[i];
        /// find the next h_cluster, at the first sphere not yet searched
        /// all spheres connected to it are part of this cluster
        if ((!h_searched[i]) && (h_current_type == SPHERE_TYPE::CORE)) {
            hipMemset(d_borders, false, sizeof(*d_borders) * nSpheres);
            hipMemset(d_visited, false, sizeof(*d_visited) * nSpheres);
            hipMemset(&d_borders[i], true, sizeof(*d_borders));
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
            h_cluster_num++;

            // visit all spheres connected to sphere i in parallel
            do {
                // find and visit border points, establishing the cluster
                ClusterSearchBFSKernel<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
                    nSpheres,
                    adj_num,
                    adj_offset,
                    adj_list,
                    d_borders,
                    d_visited);
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
                // Allocate temporary storage
                void *d_temp_storage = NULL;
                size_t temp_storage_bytes = 0;
                // Determine temporary device storage requirements
                hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                                       d_borders, d_border_num, nSpheres);
                gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
                // sum d_borders to find remainder borders to visit
                hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                                       d_borders, d_border_num, nSpheres);
                hipMemcpy(h_border_num, d_border_num,
                           sizeof(*d_border_num), hipMemcpyDeviceToHost);
                gpuErrchk(hipFree(d_temp_storage));
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());
            } while ((*h_border_num) > 0);

            hipMemcpy(h_visited, d_visited, sizeof(*d_visited) * nSpheres,
                       hipMemcpyDeviceToHost);
            h_cluster = (unsigned int *)calloc((nSpheres + 1), sizeof(*h_cluster));
            // h_cluster[0] is its size, so it length nSpheres + 1
            assert(h_cluster[0] == 0);

            // First pass of setting sphere_cluster
            // GROUND and VOLUME clusters come later
            unsigned int cluster_index = h_cluster_num + static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::START);

            for (size_t j = 0; j < nSpheres; j++) {
                if (h_visited[j]) {
                    if ((sphere_type[j] != chrono::gpu::SPHERE_TYPE::CORE) &&
                        (sphere_type[j] != chrono::gpu::SPHERE_TYPE::VOLUME)) {
                        sphere_type[j] = chrono::gpu::SPHERE_TYPE::BORDER;
                    }

                    h_searched[j] = true;
                    h_cluster[++h_cluster[0]] = j;
                    sphere_data->sphere_cluster[j] = cluster_index;
                }
            }
            h_clusters[h_cluster_num] = h_cluster;
            assert(h_cluster[0] <= nSpheres);
            assert(h_clusters[h_cluster_num][0] <= nSpheres);
            h_clusters[0][0] = h_cluster_num;
            h_searched[i] = true;
        }
    }
    size_t h_clusters_bytesize = sizeof(*h_clusters) * (h_clusters[0][0]+1);
    h_clusters = (unsigned int **)realloc(h_clusters, h_clusters_bytesize);

    free(h_visited);
    free(h_searched);
    free(h_border_num);
    gpuErrchk(hipFree(d_borders));
    gpuErrchk(hipFree(d_border_num));
    gpuErrchk(hipFree(d_visited));
    assert(h_clusters[0][0] == h_cluster_num);
    assert(h_clusters[0][0] <= nSpheres);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    return(h_clusters);
}

/// Uses sphere_contact_map to construct adjacency lists for clustering
__host__ void ConstructGraphByContact(ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                      ChSystemGpu_impl::GranParamsPtr gran_params,
                                      unsigned int nSpheres) {
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    ComputeAdjNumByContact<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                                gran_params,
                                                                nSpheres);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    ComputeAdjOffsetFromAdjNum(nSpheres,
                              sphere_data->adj_num,
                              sphere_data->adj_offset);
    
    ComputeAdjListByContact<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                                 gran_params,
                                                                 nSpheres);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

/// UNTESTED
/// G-DBSCAN; density-based h_clustering algorithm.
/// Identifies core, border and noise points in h_clusters.
/// min_pts: minimal number of points for a h_cluster
/// radius: proximity radius, points inside can form a h_cluster
__host__ void ConstructGraphByProximity(ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                        ChSystemGpu_impl::GranParamsPtr gran_params,
                                        unsigned int nSpheres,
                                        size_t min_pts,
                                        float radius) {
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    /// compute all adjacent spheres inside radius
    ComputeAdjNumByProximity<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                                  gran_params,
                                                                  nSpheres,
                                                                  radius);
    /// compute all adjacent spheres inside radius
    ComputeAdjOffsetFromAdjNum(nSpheres,
                              sphere_data->adj_num,
                              sphere_data->adj_offset);
    /// compute adjacency list
    ComputeAdjListByProximity<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                                   gran_params,
                                                                   nSpheres,
                                                                   radius);
}

/// identify ground cluster by spheres lower than a certain plane
__host__ void IdentifyGroundClusterByLowest(ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                            ChSystemGpu_impl::GranParamsPtr gran_params,
                                            unsigned int ** h_clusters,
                                            unsigned int nSpheres) {
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
    unsigned int cluster_num = h_clusters[0][0];
    unsigned int cluster_index;

    unsigned int * h_below_num = (unsigned int *)malloc(sizeof(*h_below_num));
    bool * d_below;
    gpuErrchk(hipMalloc((void**)&d_below, sizeof(*d_below) * nSpheres));
    unsigned int * d_below_num;
    gpuErrchk(hipMalloc((void**)&d_below_num, sizeof(*d_below_num)));
    unsigned int ground_cluster = static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::GROUND);
    ///　Find ALL clusters with any sphere center below plane box_z + 1*sphere_radius
    for (size_t i = 1; i < (cluster_num + 1); i++) {
        hipMemset(d_below, false, sizeof(*d_below) * nSpheres);

        // if any sphere is in VOLUME cluster, cluster index should be VOLUME
        if (sphere_data->sphere_cluster[h_clusters[i][1]] == static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::VOLUME)) {
            cluster_index = static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::VOLUME);
        } else {
            cluster_index = i + static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::START);
        }
        AreSpheresBelowZLim<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                                 gran_params,
                                                                 nSpheres,
                                                                 d_below,
                                                                 cluster_index,
                                                                 gran_params->ground_z_lim);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // Sum number of particles in below z_lim
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        // Determine temporary device storage requirements
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               d_below, d_below_num, nSpheres);
        // find and visit border points, establishing the cluster
        gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               d_below, d_below_num, nSpheres);
        hipMemcpy(h_below_num, d_below_num,
                   sizeof(*d_below_num), hipMemcpyDeviceToHost);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipFree(d_temp_storage));

        if ((*h_below_num) > 0) {
            SwitchClusterIndex<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                                    gran_params,
                                                                    nSpheres,
                                                                    cluster_index,
                                                                    ground_cluster);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    gpuErrchk(hipFree(d_below));
    gpuErrchk(hipFree(d_below_num));
    free(h_below_num);
}

__host__ void FreeClusters(unsigned int ** h_clusters) {
    unsigned int cluster_num = h_clusters[0][0];
    for (size_t i = 0; i < (cluster_num + 1); i++) {
        free(h_clusters[i]);
    }
    free(h_clusters);
}

/// Finds cluster in h_clusters with most sphres (biggest cluster)
/// sets sphere_cluster of all those spheres to GROUND
__host__ void IdentifyGroundClusterByBiggest(ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                             ChSystemGpu_impl::GranParamsPtr gran_params,
                                             unsigned int ** h_clusters,
                                             unsigned int nSpheres) {
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
    unsigned int ground_cluster = static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::GROUND);
    unsigned int cluster_num = h_clusters[0][0];
    unsigned int sphere_num_in_cluster = 1;
    unsigned int biggest_cluster_size = 1;
    unsigned int biggest_cluster;

    // find which cluster is the biggest
    for (size_t i = 1; i < (cluster_num + 1); i++) {
        sphere_num_in_cluster = h_clusters[i][0];
        assert(sphere_num_in_cluster <= nSpheres);
        if (sphere_num_in_cluster > biggest_cluster_size) {
            biggest_cluster_size = sphere_num_in_cluster;
            if (sphere_data->sphere_cluster[h_clusters[i][1]] == static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::VOLUME)) {
                biggest_cluster = static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::VOLUME);
            } else {
                biggest_cluster = i + static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::START);
            }
        }
    }

    SwitchClusterIndex<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                            gran_params,
                                                            nSpheres,
                                                            biggest_cluster,
                                                            ground_cluster);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

__host__ void IdentifyVolumeCluster(ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                    ChSystemGpu_impl::GranParamsPtr gran_params,
                                    unsigned int nSpheres,
                                    unsigned int ** h_clusters) {
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    /// sets sphere_type to VOLUME if inside a mesh
    /// must be set AFTER GdbscanInitSphereType,
    /// and AFTER interactionGranMat_TriangleSoup,
    /// which is AFTER AdvanceSimulation
    SetVolumeSphereType<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                             nSpheres);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // in_volume_num: number of spheres inside the volume mesh
    unsigned int * d_in_volume_num;
    unsigned int * h_in_volume_num = (unsigned int *)malloc(sizeof(*h_in_volume_num));
    gpuErrchk(hipMalloc((void**)&d_in_volume_num, sizeof(*d_in_volume_num)));
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    bool * d_in_volume;  // [mySphereID] -> is particle inside the volume?
    gpuErrchk(hipMalloc((void**)&d_in_volume, sizeof(*d_in_volume) * nSpheres));
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    unsigned int cluster_num = h_clusters[0][0];

    for (size_t i = 1; i < (cluster_num + 1); i++) {
        hipMemset(d_in_volume, false, sizeof(*d_in_volume) * nSpheres);
        unsigned int * h_cluster = h_clusters[i];
        unsigned int cluster_index = i + static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::START);

        // find if any sphere in cluster was tagged in the VOLUME type
        FindVolumeTypeInCluster<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data,
                                                               nSpheres,
                                                               d_in_volume,
                                                               cluster_index);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // Sum number of particles in d_in_volume into h_in_volume_num
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        // Determine temporary device storage requirements
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               d_in_volume, d_in_volume_num, nSpheres);
        // find and visit border points, establishing the cluster
        gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes,
                               d_in_volume, d_in_volume_num, nSpheres);
        hipMemcpy(h_in_volume_num, d_in_volume_num,
                   sizeof(*d_in_volume_num), hipMemcpyDeviceToHost);
        gpuErrchk(hipFree(d_temp_storage));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // any sphere of cluster is type  VOLUME -> sphere_cluster becomes VOLUME
        // UNLESS it is GROUND
        if (*h_in_volume_num > 0) {
            unsigned int volume_cluster = static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::VOLUME);
            unsigned int sphere_num_in_cluster = h_cluster[0];
            for (size_t j = 1; j < (sphere_num_in_cluster + 1); j++) {
                if (sphere_data->sphere_cluster[h_cluster[j]] != static_cast<unsigned int>(chrono::gpu::CLUSTER_INDEX::GROUND)) {
                    sphere_data->sphere_cluster[h_cluster[j]] = volume_cluster;
                }
            }
        }
    }
    gpuErrchk(hipFree(d_in_volume_num));
    gpuErrchk(hipFree(d_in_volume));
    free(h_in_volume_num);
}


/// Finds the GROUND cluster using cluster_ground_method in gran_params
__host__ void IdentifyGroundCluster(ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                    ChSystemGpu_impl::GranParamsPtr gran_params,
                                    unsigned int nSpheres,
                                    unsigned int ** h_clusters) {
    unsigned int cluster_num = h_clusters[0][0];

    if (cluster_num > 0) {
        switch (gran_params->cluster_ground_method) {
            case chrono::gpu::CLUSTER_GROUND_METHOD::NONE: {
                break;
            }
            case chrono::gpu::CLUSTER_GROUND_METHOD::BIGGEST: {
                IdentifyGroundClusterByBiggest(sphere_data, gran_params, h_clusters, nSpheres);
                break;
            }
            case chrono::gpu::CLUSTER_GROUND_METHOD::LOWEST: {
                IdentifyGroundClusterByLowest(sphere_data, gran_params, h_clusters, nSpheres);
                break;
            }
            default: {
                break;
            }
        }
    }
}

/// Search part of G-DBSCAN; density-based h_clustering algorithm.
/// Identifies core, border and noise points in h_clusters.
/// Searches using a parallel Breadth-First search
/// min_pts: minimal number of points for a cluster
__host__ unsigned int ** GdbscanSearchGraphByBFS(ChSystemGpu_impl::GranSphereDataPtr sphere_data,
                                 ChSystemGpu_impl::GranParamsPtr gran_params,
                                 unsigned int nSpheres,
                                 size_t min_pts) {
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    /// sphere_type is CORE if neighbors_num > min_pts else NOISE
    GdbscanInitSphereType<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(nSpheres,
                                                               sphere_data->adj_num,
                                                               sphere_data->sphere_type,
                                                               gran_params->gdbscan_min_pts);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    unsigned int ** h_clusters = ClusterSearchBFS(nSpheres, sphere_data,
                                                  sphere_data->adj_num,
                                                  sphere_data->adj_offset,
                                                  sphere_data->adj_list,
                                                  sphere_data->sphere_type);
    unsigned int cluster_num = h_clusters[0][0];

    GdbscanFinalClusterFromType<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(nSpheres,
                                                                     sphere_data->sphere_cluster,
                                                                     sphere_data->sphere_type);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    return(h_clusters);
}


