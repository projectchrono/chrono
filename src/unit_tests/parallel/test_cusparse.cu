// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All right reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Hammad Mazhar
// =============================================================================
//
// ChronoParallel unit test for MPR collision detection
// =============================================================================

#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>
#include "test_matvec.h"
#include <hipsparse.h>
#include <thrust/device_vector.h>
#define THRUSTCASTI(x) (int*) thrust::raw_pointer_cast(&x[0])
#define THRUSTCASTF(x) (real*) thrust::raw_pointer_cast(&x[0])

void mat_vec_cusparse(thrust::host_vector<int>& h_row,
                      thrust::host_vector<int>& h_col,
                      thrust::host_vector<real>& h_val,
                      thrust::host_vector<real>& h_rhs,
                      thrust::host_vector<real>& h_x,
                      int M,
                      int N,
                      int NNZ) {
  thrust::device_vector<int> d_row = h_row;
  thrust::device_vector<int> d_col = h_col;
  thrust::device_vector<real> d_val = h_val;
  thrust::device_vector<real> d_rhs = h_rhs;
  thrust::device_vector<real> d_x = h_x;

  real alpha = 1.0;
  real alpham1 = -1.0;
  real beta = 0.0;
  real r0 = 0.;

  hipsparseHandle_t cusparseHandle = 0;
  hipsparseStatus_t cusparseStatus;
  cusparseStatus = hipsparseCreate(&cusparseHandle);

  hipsparseMatDescr_t descr = 0;
  cusparseStatus = hipsparseCreateMatDescr(&descr);

  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipsparseStatus_t status;
  thrust::host_vector<int> h_csr(M + 1, 0);

  thrust::device_vector<int> d_csr = h_csr;
  hipsparseXcoo2csr(cusparseHandle, THRUSTCASTI(d_row), NNZ, M, THRUSTCASTI(d_csr), HIPSPARSE_INDEX_BASE_ZERO);

  hipEventRecord(start, 0);
  for (int i = 0; i < 100; i++) {
    hipsparseScsrmv(cusparseHandle,
                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   M,
                   N,
                   NNZ,
                   &alpha,
                   descr,
                   THRUSTCASTF(d_val),
                   THRUSTCASTI(d_csr),
                   THRUSTCASTI(d_col),
                   THRUSTCASTF(d_rhs),
                   &beta,
                   THRUSTCASTF(d_x));
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  h_x = d_x;

  std::cout << time / 1000.0 << std::endl;
}
