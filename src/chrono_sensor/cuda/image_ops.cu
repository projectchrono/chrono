#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Eric Brandt, Asher Elmquist
// =============================================================================
//
// =============================================================================

#include <hip/hip_runtime.h>
#include "image_ops.cuh"
#include "chrono_sensor/optix/shaders/device_utils.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

namespace chrono {
namespace sensor {

__global__ void image_gauss_kernel_vert(unsigned char* buf, int w, int h, int c, int f_width, float* dweights) {
    int index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    // only run for each output pixel
    if (index < w * h * c) {
        // float f_std = (float)f / 2.f;
        // int f_width = (int)(2.f * 3.14f * f_std);

        int channel = index % c;
        int col = index / c % w;
        int row = index / c / w;

        float sum = 0;
        for (int i = -f_width; i <= f_width; i++) {
            int index_in = channel + col * c + abs(row + i) * w * c;
            if (row + i >= h)
                index_in = channel + col * c + (2 * h - (row + i + 1)) * w * c;

            // float weight = exp(-i * i / (2 * f_std * f_std)) / sqrtf(2.f * 3.14f * f_std * f_std);
            sum += dweights[i + f_width] * ((float)buf[index_in]);
            // sum += ((float)buf[index_in]);
        }
        sum = fminf(255.f,fmaxf(0.f,sum));
        buf[index] = (unsigned char)(sum);
    }
}

__global__ void image_gauss_kernel_horiz(unsigned char* buf, int w, int h, int c, int f_width, float* dweights) {
    int index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    // only run for each output pixel
    if (index < w * h * c) {
        int channel = index % c;
        int col = index / c % w;
        int row = index / c / w;
        float sum = 0;
        for (int i = -f_width; i <= f_width; i++) {
            int index_in = channel + abs(col + i) * c + row * w * c;
            if (col + i >= w)
                index_in = channel + (2 * w - (col + i + 1)) * c + row * w * c;
            sum += dweights[i + f_width] * ((float)buf[index_in]);
        }
        sum = fminf(255.f,fmaxf(0.f,sum));
        buf[index] = (unsigned char)(sum);
    }
}

// merge pixels by the factor
__global__ void image_alias_kernel(unsigned char* bufIn,
                                   unsigned char* bufOut,
                                   int w_out,
                                   int h_out,
                                   int factor,
                                   int pix_size) {
    int out_index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    int w_in = w_out * factor;
    int h_in = h_out * factor;
    //
    // only run for each output pixel
    if (out_index < w_out * h_out * pix_size) {
        int idc_out = out_index % pix_size;
        int idx_out = (out_index / pix_size) % w_out;
        int idy_out = (out_index / pix_size) / w_out;

        float mean = 0.0;

        for (int i = -1; i < factor + 1; i++) {
            for (int j = -1; j < factor + 1; j++) {
                int idc_in = idc_out;
                int idx_in = idx_out * factor + j;
                int idy_in = idy_out * factor + i;

                // reflect when out of range

                if (idx_in < 0)
                    idx_in = -idx_in - 1;
                else if (idx_in >= w_in)
                    idx_in = 2 * w_in - (idx_in + 1);
                if (idy_in < 0)
                    idy_in = -idy_in - 1;
                else if (idy_in >= h_in)
                    idy_in = 2 * h_in - (idy_in + 1);

                int in_index = idy_in * w_in * pix_size + idx_in * pix_size + idc_in;
                mean += (float)bufIn[in_index];
            }
        }
        // bufOut[out_index] = (unsigned char)(mean / (factor * factor));
        bufOut[out_index] = (unsigned char)(mean / ((factor + 2) * (factor + 2)));
        if (idc_out == 3) {
            bufOut[out_index] = 255;
        }
        // bufOut[out_index] = (unsigned char)(25 * idc_out);
    }
}

// merge pixels by the factor
__global__ void image_alias_float_kernel(float* bufIn, float* bufOut, int w_out, int h_out, int factor, int pix_size) {
    int out_index = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer

    int w_in = w_out * factor;
    //
    // only run for each output pixel
    if (out_index < w_out * h_out * pix_size) {
        int idc_out = out_index % pix_size;
        int idx_out = (out_index / pix_size) % w_out;
        int idy_out = (out_index / pix_size) / w_out;

        float mean = 0.f;

        for (int i = 0; i < factor; i++) {
            for (int j = 0; j < factor; j++) {
                int idc_in = idc_out;
                int idx_in = idx_out * factor + j;
                int idy_in = idy_out * factor + i;

                int in_index = idy_in * w_in * pix_size + idx_in * pix_size + idc_in;
                mean += bufIn[in_index];
            }
        }
        bufOut[out_index] = mean / (factor * factor);
    }
}
// merge pixels by the factor
__global__ void image_half4_to_uchar4_kernel(__half* bufIn, unsigned char* bufOut, int N) {
    int idx = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer
    if (idx < N) {
        bufOut[idx] = (unsigned char)(clamp(__half2float(bufIn[idx]), 0.f, 1.f) * 255.f);
    }
}


//__global__ void minmax_kernel_2d(float* input,
//                                 float* min_output,
//                                 float* max_output,
//                                 const int width,
//                                 const int height) {
//    extern __shared__ float sdata[];
//
//     int tid = threadIdx.x;
//    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
//
//    float min_val = (i < width * height) ? input[i] : FLT_MAX;
//    float max_val = (i < width * height) ? input[i] : -FLT_MAX;
//
//    if (i + blockDim.x < width * height) {
//        float val = input[i + blockDim.x];
//        min_val = fminf(min_val, val);
//        max_val = fmaxf(max_val, val);
//    }
//
//    sdata[tid * 2] = min_val;
//    sdata[tid * 2 + 1] = max_val;
//    __syncthreads();
//
//    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
//        if (tid < s) {
//            sdata[tid * 2] = min_val = fminf(min_val, sdata[(tid + s) * 2]);
//            sdata[tid * 2 + 1] = max_val = fmaxf(max_val, sdata[(tid + s) * 2 + 1]);
//        }
//        __syncthreads();
//    }
//
//    if (tid == 0) {
//        atomicMin(min_output, min_val);
//        atomicMax(max_output, max_val);
//    }
//}


__global__ void depth_to_uchar4_kernel(float* bufIn, unsigned char* bufOut, float d_min, float d_max, int N) {
    int idx = (blockDim.x * blockIdx.x + threadIdx.x);  // index into output buffer
    if (idx < N) {
        float normalized_depth = clamp((bufIn[idx] - d_min) / (d_max - d_min), 0.f, 1.f);
        unsigned char intensity = (unsigned char)(normalized_depth * 255.f);

        // Gray scale colormap
        bufOut[idx * 4 + 0] = intensity;
        bufOut[idx * 4 + 1] = intensity;
        bufOut[idx * 4 + 2] = intensity;
        bufOut[idx * 4 + 3] = (unsigned char)255;

    }
       
}



void cuda_image_gauss_blur_char(void* buf, int w, int h, int c, int factor, hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w * h * c + nThreads - 1) / nThreads;

    float f_std = (float)factor / 4.f;
    int f_width = (int)(3.14f * f_std);

    int entries = 2 * f_width + 1;

    float* weights = new float[entries];

    for (int i = 0; i <= 2 * f_width; i++) {
        int offset = i - f_width;
        weights[i] = exp(-offset * offset / (2 * f_std * f_std)) / sqrtf(2.f * 3.14f * f_std * f_std);
    }
    float* dweights;
    hipMalloc(&dweights, entries * sizeof(float));
    hipMemcpy(dweights, weights, entries * sizeof(float), hipMemcpyHostToDevice);

    image_gauss_kernel_vert<<<nBlocks, nThreads, 0, stream>>>((unsigned char*)buf, w, h, c, f_width, dweights);
    image_gauss_kernel_horiz<<<nBlocks, nThreads, 0, stream>>>((unsigned char*)buf, w, h, c, f_width, dweights);
    hipFree(dweights);
    delete[] weights;
}

void cuda_image_alias(void* bufIn, void* bufOut, int w_out, int h_out, int factor, int pix_size, hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w_out * h_out * pix_size + nThreads - 1) / nThreads;

    image_alias_kernel<<<nBlocks, nThreads, 0, stream>>>((unsigned char*)bufIn, (unsigned char*)bufOut, w_out, h_out,
                                                         factor, pix_size);
}

void cuda_image_alias_float(void* bufIn,
                            void* bufOut,
                            int w_out,
                            int h_out,
                            int factor,
                            int pix_size,
                            hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w_out * h_out * pix_size + nThreads - 1) / nThreads;

    image_alias_float_kernel<<<nBlocks, nThreads, 0, stream>>>((float*)bufIn, (float*)bufOut, w_out, h_out, factor,
                                                               pix_size);
}

void cuda_image_half4_to_uchar4(void* bufIn, void* bufOut, int w, int h, hipStream_t& stream) {
    const int nThreads = 512;
    int nBlocks = (w * h * 4 + nThreads - 1) / nThreads;
    image_half4_to_uchar4_kernel<<<nBlocks, nThreads, 0, stream>>>((__half*)bufIn, (unsigned char*)bufOut, w * h * 4);
}

void cuda_depth_to_uchar4(void* bufIn, void* bufOut, int w, int h, hipStream_t& stream) {
    // Set up kernel launch configuration
    // int blockSize = 256;
    // int gridSize = (w * h + blockSize * 2 - 1) / (blockSize * 2);

    /*float *d_min, *d_max;
    hipMalloc(&d_min, sizeof(float));
    hipMalloc(&d_max, sizeof(float));

    hipMemcpy(d_min, &MIN, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_max, &MAX, sizeof(float), hipMemcpyHostToDevice);*/

    thrust::device_vector<float> bufIn_thrust((float*)bufIn, (float*)bufIn + w * h);
    thrust::device_ptr<float> buffIn_ptr = thrust::device_pointer_cast((float*)bufIn);
    // thrust::pair<float*, float*> result = thrust::minmax_element(thrust::device, (float*)bufIn, (float*)bufIn + w *
    // h);

    thrust::pair<thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator> result =
        thrust::minmax_element(bufIn_thrust.begin(), bufIn_thrust.end());

    // Launch the kernel
    // minmax_kernel_2d<<<gridSize, blockSize, blockSize * 2 * sizeof(float)>>>((float*)bufIn, d_min, d_max, w, h);

    // hipDeviceSynchronize();

    const int nThreads = 512;
    int nBlocks = (w * h + nThreads - 1) / nThreads;

    depth_to_uchar4_kernel<<<nBlocks, nThreads, 0, stream>>>((float*)bufIn, (unsigned char*)bufOut, *(result.first),
                                                             *(result.second), w * h);
}

}  // namespace sensor
}  // namespace chrono
