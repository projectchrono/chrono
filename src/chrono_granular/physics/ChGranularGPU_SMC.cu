#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2019 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Authors: Conlain Kelly, Nic Olsen, Dan Negrut
// =============================================================================

#include <cmath>
#include <numeric>

#include "chrono_granular/physics/ChGranularGPU_SMC.cuh"
#include "chrono_granular/utils/ChGranularUtilities.h"
#include "chrono/core/ChTimer.h"

namespace chrono {
namespace granular {

__host__ double ChSystemGranularSMC::get_max_z() const {
    size_t nSpheres = sphere_local_pos_Z.size();
    std::vector<int64_t> sphere_pos_global_Z;
    sphere_pos_global_Z.resize(nSpheres);
    for (size_t index = 0; index < nSpheres; index++) {
        unsigned int ownerSD = sphere_data->sphere_owner_SDs[index];
        int3 sphere_pos_local =
            make_int3(sphere_data->sphere_local_pos_X[index], sphere_data->sphere_local_pos_Y[index],
                      sphere_data->sphere_local_pos_Z[index]);
        sphere_pos_global_Z[index] = convertPosLocalToGlobal(ownerSD, sphere_pos_local, gran_params).z;
    }

    double max_z_SU = *(std::max_element(sphere_pos_global_Z.begin(), sphere_pos_global_Z.end()));
    double max_z_UU = max_z_SU * LENGTH_SU2UU;

    return max_z_UU;
}

// Reset broadphase data structures
void ChSystemGranularSMC::resetBroadphaseInformation() {
    // Set all the offsets to zero
    gpuErrchk(hipMemset(SD_NumSpheresTouching.data(), 0, SD_NumSpheresTouching.size() * sizeof(unsigned int)));
    gpuErrchk(hipMemset(SD_SphereCompositeOffsets.data(), 0, SD_SphereCompositeOffsets.size() * sizeof(unsigned int)));
    // For each SD, all the spheres touching that SD should have their ID be NULL_GRANULAR_ID
    gpuErrchk(hipMemset(spheres_in_SD_composite.data(), NULL_GRANULAR_ID,
                         spheres_in_SD_composite.size() * sizeof(unsigned int)));
    gpuErrchk(hipDeviceSynchronize());
}

// Reset sphere acceleration data structures
void ChSystemGranularSMC::resetSphereAccelerations() {
    // cache past acceleration data
    if (time_integrator == GRAN_TIME_INTEGRATOR::CHUNG) {
        gpuErrchk(hipMemcpy(sphere_acc_X_old.data(), sphere_acc_X.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_acc_Y_old.data(), sphere_acc_Y.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(sphere_acc_Z_old.data(), sphere_acc_Z.data(), nSpheres * sizeof(float),
                             hipMemcpyDeviceToDevice));
        // if we have multistep AND friction, cache old alphas
        if (gran_params->friction_mode != FRICTIONLESS) {
            gpuErrchk(hipMemcpy(sphere_ang_acc_X_old.data(), sphere_ang_acc_X.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
            gpuErrchk(hipMemcpy(sphere_ang_acc_Y_old.data(), sphere_ang_acc_Y.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
            gpuErrchk(hipMemcpy(sphere_ang_acc_Z_old.data(), sphere_ang_acc_Z.data(), nSpheres * sizeof(float),
                                 hipMemcpyDeviceToDevice));
        }
        gpuErrchk(hipDeviceSynchronize());
    }

    // reset current accelerations to zero to zero
    gpuErrchk(hipMemset(sphere_acc_X.data(), 0, nSpheres * sizeof(float)));
    gpuErrchk(hipMemset(sphere_acc_Y.data(), 0, nSpheres * sizeof(float)));
    gpuErrchk(hipMemset(sphere_acc_Z.data(), 0, nSpheres * sizeof(float)));

    // reset torques to zero, if applicable
    if (gran_params->friction_mode != FRICTIONLESS) {
        gpuErrchk(hipMemset(sphere_ang_acc_X.data(), 0, nSpheres * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Y.data(), 0, nSpheres * sizeof(float)));
        gpuErrchk(hipMemset(sphere_ang_acc_Z.data(), 0, nSpheres * sizeof(float)));
    }
}

__global__ void compute_absv(const unsigned int nSpheres,
                             const float* velX,
                             const float* velY,
                             const float* velZ,
                             float* d_absv) {
    unsigned int my_sphere = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_sphere < nSpheres) {
        float v[3] = {velX[my_sphere], velY[my_sphere], velZ[my_sphere]};
        d_absv[my_sphere] = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
    }
}

__host__ float ChSystemGranularSMC::get_max_vel() const {
    float* d_absv;
    float* d_max_vel;
    float h_max_vel;
    gpuErrchk(hipMalloc(&d_absv, nSpheres * sizeof(float)));
    gpuErrchk(hipMalloc(&d_max_vel, sizeof(float)));

    compute_absv<<<(nSpheres + 255) / 256, 256>>>(nSpheres, pos_X_dt.data(), pos_Y_dt.data(), pos_Z_dt.data(), d_absv);

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nSpheres);
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_absv, d_max_vel, nSpheres);
    gpuErrchk(hipMemcpy(&h_max_vel, d_max_vel, sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_absv));
    gpuErrchk(hipFree(d_max_vel));

    return h_max_vel;
}

__host__ int3 ChSystemGranularSMC::getSDTripletFromID(unsigned int SD_ID) const {
    return SDIDTriplet(SD_ID, gran_params);
}
/// Sort sphere positions by subdomain id
/// Occurs entirely on host, not intended to be efficient
/// ONLY DO AT BEGINNING OF SIMULATION
__host__ void ChSystemGranularSMC::defragment_initial_positions() {
    INFO_PRINTF("Starting defrag run!\n");
    ChTimer<> timer;
    timer.start();

    // key and value pointers
    std::vector<unsigned int, cudallocator<unsigned int>> sphere_ids;

    // load sphere indices
    sphere_ids.resize(nSpheres);
    std::iota(sphere_ids.begin(), sphere_ids.end(), 0);

    // sort sphere ids by owner SD
    std::sort(sphere_ids.begin(), sphere_ids.end(),
              [&](std::size_t i, std::size_t j) { return sphere_owner_SDs.at(i) < sphere_owner_SDs.at(j); });

    std::vector<int, cudallocator<int>> sphere_pos_x_tmp;
    std::vector<int, cudallocator<int>> sphere_pos_y_tmp;
    std::vector<int, cudallocator<int>> sphere_pos_z_tmp;
    std::vector<not_stupid_bool, cudallocator<not_stupid_bool>> sphere_fixed_tmp;
    std::vector<unsigned int, cudallocator<unsigned int>> sphere_owner_SDs_tmp;

    sphere_pos_x_tmp.resize(nSpheres);
    sphere_pos_y_tmp.resize(nSpheres);
    sphere_pos_z_tmp.resize(nSpheres);
    sphere_fixed_tmp.resize(nSpheres);
    sphere_owner_SDs_tmp.resize(nSpheres);

    // reorder values into new sorted
    for (unsigned int i = 0; i < nSpheres; i++) {
        sphere_pos_x_tmp.at(i) = sphere_local_pos_X.at(sphere_ids.at(i));
        sphere_pos_y_tmp.at(i) = sphere_local_pos_Y.at(sphere_ids.at(i));
        sphere_pos_z_tmp.at(i) = sphere_local_pos_Z.at(sphere_ids.at(i));
        sphere_fixed_tmp.at(i) = sphere_fixed.at(sphere_ids.at(i));
        sphere_owner_SDs_tmp.at(i) = sphere_owner_SDs.at(sphere_ids.at(i));
    }

    // swap into the correct data structures
    sphere_local_pos_X.swap(sphere_pos_x_tmp);
    sphere_local_pos_Y.swap(sphere_pos_y_tmp);
    sphere_local_pos_Z.swap(sphere_pos_z_tmp);
    sphere_fixed.swap(sphere_fixed_tmp);
    sphere_owner_SDs.swap(sphere_owner_SDs_tmp);

    timer.stop();
    INFO_PRINTF("finished defrag run in %f seconds!\n", timer.GetTimeSeconds());
}
__host__ void ChSystemGranularSMC::setupSphereDataStructures() {
    // Each fills user_sphere_positions with positions to be copied
    if (user_sphere_positions.size() == 0) {
        printf("ERROR: no sphere positions given!\n");
        exit(1);
    }

    nSpheres = (unsigned int)user_sphere_positions.size();
    INFO_PRINTF("%u balls added!\n", nSpheres);
    gran_params->nSpheres = nSpheres;

    TRACK_VECTOR_RESIZE(sphere_owner_SDs, nSpheres, "sphere_owner_SDs", NULL_GRANULAR_ID);

    // Allocate space for new bodies
    TRACK_VECTOR_RESIZE(sphere_local_pos_X, nSpheres, "sphere_local_pos_X", 0);
    TRACK_VECTOR_RESIZE(sphere_local_pos_Y, nSpheres, "sphere_local_pos_Y", 0);
    TRACK_VECTOR_RESIZE(sphere_local_pos_Z, nSpheres, "sphere_local_pos_Z", 0);

    TRACK_VECTOR_RESIZE(sphere_fixed, nSpheres, "sphere_fixed", 0);

    // temporarily store global positions as 64-bit, discard as soon as local positions are loaded
    {
        bool user_provided_fixed = user_sphere_fixed.size() != 0;
        if (user_provided_fixed && user_sphere_fixed.size() != nSpheres) {
            printf("Provided fixity array does not match provided particle positions\n");
            exit(1);
        }

        std::vector<int64_t, cudallocator<int64_t>> sphere_global_pos_X;
        std::vector<int64_t, cudallocator<int64_t>> sphere_global_pos_Y;
        std::vector<int64_t, cudallocator<int64_t>> sphere_global_pos_Z;

        sphere_global_pos_X.resize(nSpheres);
        sphere_global_pos_Y.resize(nSpheres);
        sphere_global_pos_Z.resize(nSpheres);

        // Copy from array of structs to 3 arrays
        for (unsigned int i = 0; i < nSpheres; i++) {
            auto vec = user_sphere_positions.at(i);
            // cast to double, convert to SU, then cast to int64_t
            sphere_global_pos_X.at(i) = (int64_t)((double)vec.x() / LENGTH_SU2UU);
            sphere_global_pos_Y.at(i) = (int64_t)((double)vec.y() / LENGTH_SU2UU);
            sphere_global_pos_Z.at(i) = (int64_t)((double)vec.z() / LENGTH_SU2UU);

            // Convert to not_stupid_bool
            sphere_fixed.at(i) = (not_stupid_bool)((user_provided_fixed) ? user_sphere_fixed[i] : false);
        }

        packSphereDataPointers();
        // Figure our the number of blocks that need to be launched to cover the box
        unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;
        initializeLocalPositions<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
            sphere_data, sphere_global_pos_X.data(), sphere_global_pos_Y.data(), sphere_global_pos_Z.data(), nSpheres,
            gran_params);

        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipPeekAtLastError());
        defragment_initial_positions();
    }

    TRACK_VECTOR_RESIZE(pos_X_dt, nSpheres, "pos_X_dt", 0);
    TRACK_VECTOR_RESIZE(pos_Y_dt, nSpheres, "pos_Y_dt", 0);
    TRACK_VECTOR_RESIZE(pos_Z_dt, nSpheres, "pos_Z_dt", 0);
    TRACK_VECTOR_RESIZE(sphere_acc_X, nSpheres, "sphere_acc_X", 0);
    TRACK_VECTOR_RESIZE(sphere_acc_Y, nSpheres, "sphere_acc_Y", 0);
    TRACK_VECTOR_RESIZE(sphere_acc_Z, nSpheres, "sphere_acc_Z", 0);

    // NOTE that this will get resized again later, this is just the first estimate
    TRACK_VECTOR_RESIZE(spheres_in_SD_composite, 2 * nSpheres, "spheres_in_SD_composite", NULL_GRANULAR_ID);

    if (gran_params->friction_mode != GRAN_FRICTION_MODE::FRICTIONLESS) {
        // add rotational DOFs
        TRACK_VECTOR_RESIZE(sphere_Omega_X, nSpheres, "sphere_Omega_X", 0);
        TRACK_VECTOR_RESIZE(sphere_Omega_Y, nSpheres, "sphere_Omega_Y", 0);
        TRACK_VECTOR_RESIZE(sphere_Omega_Z, nSpheres, "sphere_Omega_Z", 0);

        // add torques
        TRACK_VECTOR_RESIZE(sphere_ang_acc_X, nSpheres, "sphere_ang_acc_X", 0);
        TRACK_VECTOR_RESIZE(sphere_ang_acc_Y, nSpheres, "sphere_ang_acc_Y", 0);
        TRACK_VECTOR_RESIZE(sphere_ang_acc_Z, nSpheres, "sphere_ang_acc_Z", 0);
    }

    if (gran_params->friction_mode == GRAN_FRICTION_MODE::MULTI_STEP ||
        gran_params->friction_mode == GRAN_FRICTION_MODE::SINGLE_STEP) {
        TRACK_VECTOR_RESIZE(contact_partners_map, 12 * nSpheres, "contact_partners_map", NULL_GRANULAR_ID);
        TRACK_VECTOR_RESIZE(contact_active_map, 12 * nSpheres, "contact_active_map", false);
    }
    if (gran_params->friction_mode == GRAN_FRICTION_MODE::MULTI_STEP) {
        float3 null_history = {0., 0., 0.};
        TRACK_VECTOR_RESIZE(contact_history_map, 12 * nSpheres, "contact_history_map", null_history);
    }

    if (time_integrator == GRAN_TIME_INTEGRATOR::CHUNG) {
        TRACK_VECTOR_RESIZE(sphere_acc_X_old, nSpheres, "sphere_acc_X_old", 0);
        TRACK_VECTOR_RESIZE(sphere_acc_Y_old, nSpheres, "sphere_acc_Y_old", 0);
        TRACK_VECTOR_RESIZE(sphere_acc_Z_old, nSpheres, "sphere_acc_Z_old", 0);

        // friction and multistep means keep old ang acc
        if (gran_params->friction_mode != GRAN_FRICTION_MODE::FRICTIONLESS) {
            TRACK_VECTOR_RESIZE(sphere_ang_acc_X_old, nSpheres, "sphere_ang_acc_X_old", 0);
            TRACK_VECTOR_RESIZE(sphere_ang_acc_Y_old, nSpheres, "sphere_ang_acc_Y_old", 0);
            TRACK_VECTOR_RESIZE(sphere_ang_acc_Z_old, nSpheres, "sphere_ang_acc_Z_old", 0);
        }
    }
    // make sure the right pointers are packed
    packSphereDataPointers();
}

__host__ void ChSystemGranularSMC::runSphereBroadphase() {
    METRICS_PRINTF("Resetting broadphase info!\n");

    resetBroadphaseInformation();
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    packSphereDataPointers();

    sphereBroadphase_dryrun<CUDA_THREADS_PER_BLOCK>
        <<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nSpheres, gran_params);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    // num spheres in last SD
    unsigned int last_SD_num_spheres = SD_NumSpheresTouching.at(nSDs - 1);

    unsigned int* out_ptr = SD_SphereCompositeOffsets.data();
    unsigned int* in_ptr = SD_NumSpheresTouching.data();

    // copy data into the tmp array
    gpuErrchk(hipMemcpy(out_ptr, in_ptr, nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, in_ptr, out_ptr, nSDs);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    // Allocate temporary storage
    gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, in_ptr, out_ptr, nSDs);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    // total number of sphere entries to record
    unsigned int num_entries = out_ptr[nSDs - 1] + in_ptr[nSDs - 1];
    spheres_in_SD_composite.resize(num_entries, NULL_GRANULAR_ID);

    // make sure the DEs pointer is updated
    packSphereDataPointers();

    // printf("first run: num entries is %u, theoretical max is %u\n", num_entries, nSDs * MAX_COUNT_OF_SPHERES_PER_SD);

    // for (unsigned int i = 0; i < nSDs; i++) {
    //     printf("SD %d has offset %u, N %u \n", i, out_ptr[i], in_ptr[i]);
    // }

    // back up the offsets
    // TODO use a cached allocator, CUB provides one
    std::vector<unsigned int, cudallocator<unsigned int>> SD_SphereCompositeOffsets_bak;
    SD_SphereCompositeOffsets_bak.resize(SD_SphereCompositeOffsets.size());
    gpuErrchk(hipMemcpy(SD_SphereCompositeOffsets_bak.data(), SD_SphereCompositeOffsets.data(),
                         nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    sphereBroadphase<CUDA_THREADS_PER_BLOCK><<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(sphere_data, nSpheres, gran_params);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    //
    // for (unsigned int i = 0; i < nSDs; i++) {
    //     printf("SD %d has offset %u, N %u \n", i, out_ptr[i], in_ptr[i]);
    // }
    //
    // for (unsigned int i = 0; i < num_entries; i++) {
    //     printf("entry %u is %u\n", i, spheres_in_SD_composite[i]);
    // }

    // restore the old offsets
    gpuErrchk(hipMemcpy(SD_SphereCompositeOffsets.data(), SD_SphereCompositeOffsets_bak.data(),
                         nSDs * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    gpuErrchk(hipFree(d_temp_storage));
}

__host__ void ChSystemGranularSMC::updateBCPositions() {
    for (unsigned int i = 0; i < BC_params_list_UU.size(); i++) {
        auto bc_type = BC_type_list.at(i);
        const BC_params_t<float, float3>& params_UU = BC_params_list_UU.at(i);
        BC_params_t<int64_t, int64_t3>& params_SU = BC_params_list_SU.at(i);
        auto offset_function = BC_offset_function_list.at(i);
        setBCOffset(bc_type, params_UU, params_SU, offset_function(elapsedSimTime));
    }

    if (!BD_is_fixed) {
        double3 new_BD_offset = BDOffsetFunction(elapsedSimTime);

        int64_t3 bd_offset_SU = {0, 0, 0};
        bd_offset_SU.x = new_BD_offset.x / LENGTH_SU2UU;
        bd_offset_SU.y = new_BD_offset.y / LENGTH_SU2UU;
        bd_offset_SU.z = new_BD_offset.z / LENGTH_SU2UU;

        int64_t old_frame_X = gran_params->BD_frame_X;
        int64_t old_frame_Y = gran_params->BD_frame_Y;
        int64_t old_frame_Z = gran_params->BD_frame_Z;

        gran_params->BD_frame_X = bd_offset_SU.x + BD_rest_frame_SU.x;
        gran_params->BD_frame_Y = bd_offset_SU.y + BD_rest_frame_SU.y;
        gran_params->BD_frame_Z = bd_offset_SU.z + BD_rest_frame_SU.z;

        unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

        int64_t3 offset_delta = {0, 0, 0};

        // if the frame X increases, the local X should decrease
        offset_delta.x = old_frame_X - gran_params->BD_frame_X;
        offset_delta.y = old_frame_Y - gran_params->BD_frame_Y;
        offset_delta.z = old_frame_Z - gran_params->BD_frame_Z;

        // printf("offset is %lld, %lld, %lld\n", offset_delta.x, offset_delta.y, offset_delta.z);

        packSphereDataPointers();

        applyBDFrameChange<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(offset_delta, sphere_data, nSpheres, gran_params);

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }
}

__host__ double ChSystemGranularSMC::advance_simulation(float duration) {
    // Figure our the number of blocks that need to be launched to cover the box
    unsigned int nBlocks = (nSpheres + CUDA_THREADS_PER_BLOCK - 1) / CUDA_THREADS_PER_BLOCK;

    // Settling simulation loop.
    float duration_SU = duration / TIME_SU2UU;
    unsigned int nsteps = std::round(duration_SU / stepSize_SU);

    METRICS_PRINTF("advancing by %f at timestep %f, %u timesteps at approx user timestep %f\n", duration_SU,
                   stepSize_SU, nsteps, duration / nsteps);
    float time_elapsed_SU = 0;  // time elapsed in this advance call

    // Run the simulation, there are aggressive synchronizations because we want to have no race conditions
    for (; time_elapsed_SU < stepSize_SU * nsteps; time_elapsed_SU += stepSize_SU) {
        updateBCPositions();

        runSphereBroadphase();
        packSphereDataPointers();

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        resetSphereAccelerations();
        resetBCForces();

        METRICS_PRINTF("Starting computeSphereForces!\n");

        if (gran_params->friction_mode == FRICTIONLESS) {
            // Compute sphere-sphere forces
            computeSphereForces_frictionless<<<nSDs, MAX_COUNT_OF_SPHERES_PER_SD>>>(
                sphere_data, gran_params, BC_type_list.data(), BC_params_list_SU.data(), BC_params_list_SU.size());
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        } else if (gran_params->friction_mode == SINGLE_STEP || gran_params->friction_mode == MULTI_STEP) {
            // figure out who is contacting
            determineContactPairs<<<nSDs, MAX_COUNT_OF_SPHERES_PER_SD>>>(sphere_data, gran_params);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());

            computeSphereContactForces<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(
                sphere_data, gran_params, BC_type_list.data(), BC_params_list_SU.data(), BC_params_list_SU.size(),
                nSpheres);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }

        METRICS_PRINTF("Starting integrateSpheres!\n");
        integrateSpheres<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(stepSize_SU, sphere_data, nSpheres, gran_params);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        if (gran_params->friction_mode != GRAN_FRICTION_MODE::FRICTIONLESS) {
            updateFrictionData<<<nBlocks, CUDA_THREADS_PER_BLOCK>>>(stepSize_SU, sphere_data, nSpheres, gran_params);
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }

        elapsedSimTime += stepSize_SU * TIME_SU2UU;  // Advance current time
    }

    return time_elapsed_SU * TIME_SU2UU;  // return elapsed UU time
}
}  // namespace granular
}  // namespace chrono
