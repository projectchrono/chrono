// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================
//
// Base class for processing sph force in fsi system.
// =============================================================================

#include <thrust/extrema.h>
#include <thrust/sort.h>
#include "chrono_fsi/physics/ChFsiForce.cuh"
#include "chrono_fsi/utils/ChUtilsDevice.cuh"
#include "chrono_fsi/physics/ChSphGeneral.cuh"

namespace chrono {
namespace fsi {

ChFsiForce::ChFsiForce(std::shared_ptr<ChBce> otherBceWorker,
                       std::shared_ptr<SphMarkerDataD> otherSortedSphMarkersD,
                       std::shared_ptr<ProximityDataD> otherMarkersProximityD,
                       std::shared_ptr<FsiData> otherFsiData,
                       std::shared_ptr<SimParams> params,
                       std::shared_ptr<ChCounters> numObjects,
                       bool verb)
    : ChFsiBase(params, numObjects),
      bceWorker(otherBceWorker),
      sortedSphMarkers_D(otherSortedSphMarkersD),
      markersProximity_D(otherMarkersProximityD),
      fsiData(otherFsiData),
      verbose(verb) {
    fsiCollisionSystem = chrono_types::make_shared<ChCollisionSystemFsi>(
        otherSortedSphMarkersD, otherMarkersProximityD, otherFsiData, paramsH, numObjectsH);
    sphMarkersD = NULL;
}

void ChFsiForce::Initialize() {
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH.get(), sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH.get(), sizeof(ChCounters));

    vel_XSPH_Sorted_D.resize(numObjectsH->numAllMarkers);
    vel_vis_Sorted_D.resize(numObjectsH->numAllMarkers);
    derivVelRhoD_Sorted_D.resize(numObjectsH->numAllMarkers);
    fsiCollisionSystem->Initialize();
}

ChFsiForce::~ChFsiForce() {}

// Use invasive to avoid one extra copy.
// However, keep in mind that sorted is changed.
void ChFsiForce::CopySortedToOriginal_Invasive_R3(thrust::device_vector<Real3>& original,
                                                  thrust::device_vector<Real3>& sorted,
                                                  const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
    thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(), sorted.begin());
    dummyMarkerIndex.clear();
    thrust::copy(sorted.begin(), sorted.end(), original.begin());
}

void ChFsiForce::CopySortedToOriginal_NonInvasive_R3(thrust::device_vector<Real3>& original,
                                                     const thrust::device_vector<Real3>& sorted,
                                                     const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<Real3> dummySorted = sorted;
    CopySortedToOriginal_Invasive_R3(original, dummySorted, gridMarkerIndex);
}

// Use invasive to avoid one extra copy.
// However, keep in mind that sorted is changed.
void ChFsiForce::CopySortedToOriginal_Invasive_R4(thrust::device_vector<Real4>& original,
                                                  thrust::device_vector<Real4>& sorted,
                                                  const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
    thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(), sorted.begin());
    dummyMarkerIndex.clear();
    thrust::copy(sorted.begin(), sorted.end(), original.begin());
}

void ChFsiForce::CopySortedToOriginal_NonInvasive_R4(thrust::device_vector<Real4>& original,
                                                     thrust::device_vector<Real4>& sorted,
                                                     const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<Real4> dummySorted = sorted;
    CopySortedToOriginal_Invasive_R4(original, dummySorted, gridMarkerIndex);
}

}  // namespace fsi
}  // namespace chrono
