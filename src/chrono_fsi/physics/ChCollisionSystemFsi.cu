#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Arman Pazouki, Milad Rakhsha, Wei Hu
// =============================================================================
//
// Base class for processing proximity in fsi system.
// =============================================================================

#include <thrust/sort.h>
#include "chrono_fsi/physics/ChCollisionSystemFsi.cuh"
#include "chrono_fsi/physics/ChSphGeneral.cuh"
#include "chrono_fsi/utils/ChUtilsDevice.cuh"

namespace chrono {
namespace fsi {

// calcHashD :
// 1. Get particle index determined by the block and thread we are in.
// 2. From x, y, z position, determine which bin it is in.
// 3. Calculate hash from bin index.
// 4. Store hash and particle index associated with it.
__global__ void calcHashD(
    uint* gridMarkerHashD,   // gridMarkerHash Store particle hash here
    uint* gridMarkerIndexD,  // gridMarkerIndex Store particle index here
    Real4* posRad,           // posRad Vector containing the positions of all particles (SPH and BCE)
    const size_t numAllMarkers,  // Total number of particles (fluid + boundary)
    volatile bool* isErrorD) {
    /* Calculate the index of where the particle is stored in posRad. */
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numAllMarkers)
        return;

    Real3 p = mR3(posRad[index]);

    if (!(isfinite(p.x) && isfinite(p.y) && isfinite(p.z))) {
        printf(
            "Error! particle position is NAN: thrown from "
            "ChCollisionSystemFsi.cu, calcHashD !\n");
        *isErrorD = true;
        return;
    }

    /* Check particle is inside the domain. */
    Real3 boxCorner = paramsD.worldOrigin - mR3(40 * paramsD.HSML);
    if (p.x < boxCorner.x || p.y < boxCorner.y || p.z < boxCorner.z) {
        printf(
            "Out of Min Boundary, point %f %f %f, boundary min: %f %f %f. "
            "Thrown from ChCollisionSystemFsi.cu, calcHashD !\n",
            p.x, p.y, p.z, boxCorner.x, boxCorner.y, boxCorner.z);
        *isErrorD = true;
        return;
    }
    boxCorner = paramsD.worldOrigin + paramsD.boxDims + mR3(40 * paramsD.HSML);
    if (p.x > boxCorner.x || p.y > boxCorner.y || p.z > boxCorner.z) {
        printf(
            "Out of max Boundary, point %f %f %f, boundary max: %f %f %f. "
            "Thrown from ChCollisionSystemFsi.cu, calcHashD !\n",
            p.x, p.y, p.z, boxCorner.x, boxCorner.y, boxCorner.z);
        *isErrorD = true;
        return;
    }

    /* Get x,y,z bin index in grid */
    int3 gridPos = calcGridPos(p);
    /* Calculate a hash from the bin index */
    uint hash = calcGridHash(gridPos);

    /* Store grid hash */
    gridMarkerHashD[index] = hash;
    /* Store particle index associated to the hash we stored in gridMarkerHashD */
    gridMarkerIndexD[index] = index;
}

/**
 * @brief reorderDataAndFindCellStartD
 * @details See ChCollisionSystemFsi.cuh for more info
 */
__global__ void reorderDataAndFindCellStartD(uint* cellStartD,          // output: cell start index
                                             uint* cellEndD,            // output: cell end index
                                             Real4* sortedPosRadD,      // output: sorted positions
                                             Real3* sortedVelMasD,      // output: sorted velocities
                                             Real4* sortedRhoPreMuD,    // output: sorted density pressure
                                             Real3* sortedTauXxYyZzD,   // output: sorted total stress xxyyzz
                                             Real3* sortedTauXyXzYzD,   // output: sorted total stress xyzxyz
                                             Real3* tauXxYyZzD,         // input: original total stress xxyyzz
                                             Real3* tauXyXzYzD,         // input: original total stress xyzxyz
                                             uint* gridMarkerHashD,     // input: sorted grid hashes
                                             uint* gridMarkerIndexD,    // input: sorted particle indices
                                             Real4* posRadD,            // input: original position array
                                             Real3* velMasD,            // input: original velocity array
                                             Real4* rhoPresMuD,         // input: original density pressure
                                             const size_t numAllMarkers) {
    extern __shared__ uint sharedHash[];  // blockSize + 1 elements
    /* Get the particle index the current thread is supposed to be looking at. */
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    uint hash;
    /* handle case when no. of particles not multiple of block size */
    if (index < numAllMarkers) {
        hash = gridMarkerHashD[index];
        /* Load hash data into shared memory so that we can look at neighboring
         * particle's hash
         * value without loading two hash values per thread
         */
        sharedHash[threadIdx.x + 1] = hash;

        if (index > 0 && threadIdx.x == 0) {
            /* first thread in block must load neighbor particle hash */
            sharedHash[0] = gridMarkerHashD[index - 1];
        }
    }

    __syncthreads();

    if (index < numAllMarkers) {
        /* If this particle has a different cell index to the previous particle then
         * it must be
         * the first particle in the cell, so store the index of this particle in
         * the cell. As it
         * isn't the first particle, it must also be the cell end of the previous
         * particle's cell
         */
        if (index == 0 || hash != sharedHash[threadIdx.x]) {
            cellStartD[hash] = index;
            if (index > 0)
                cellEndD[sharedHash[threadIdx.x]] = index;
        }

        if (index == numAllMarkers - 1) {
            cellEndD[hash] = index + 1;
        }

        /* Now use the sorted index to reorder the pos and vel data */
        uint originalIndex = gridMarkerIndexD[index];  // map sorted to original
        Real3 posRad = mR3(posRadD[originalIndex]);  // macro does either global read or
                                                     // texture fetch
        Real3 velMas = velMasD[originalIndex];       // see particles_kernel.cuh
        Real4 rhoPreMu = rhoPresMuD[originalIndex];

        if (!(isfinite(posRad.x) && isfinite(posRad.y) && isfinite(posRad.z))) {
            printf(
                "Error! particle position is NAN: thrown from "
                "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
        }
        if (!(isfinite(velMas.x) && isfinite(velMas.y) && isfinite(velMas.z))) {
            printf(
                "Error! particle velocity is NAN: thrown from "
                "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
        }
        if (!(isfinite(rhoPreMu.x) && isfinite(rhoPreMu.y) && isfinite(rhoPreMu.z) && isfinite(rhoPreMu.w))) {
            printf(
                "Error! particle rhoPreMu is NAN: thrown from "
                "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
        }

        sortedPosRadD[index] = mR4(posRad, posRadD[originalIndex].w);
        sortedVelMasD[index] = velMas;
        sortedRhoPreMuD[index] = rhoPreMu;

        // For granular material
        if( paramsD.elastic_SPH ) {
            Real3 tauXxYyZz = tauXxYyZzD[originalIndex];  
            Real3 tauXyXzYz = tauXyXzYzD[originalIndex]; 
            if (!(isfinite(tauXxYyZz.x) && isfinite(tauXxYyZz.y) && isfinite(tauXxYyZz.z))) {  
                printf(
                    "Error! particle tauXxYyZz is NAN: thrown from "
                    "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
            }
            if (!(isfinite(tauXyXzYz.x) && isfinite(tauXyXzYz.y) && isfinite(tauXyXzYz.z))) { 
                printf(
                    "Error! particle tauXyXzYz is NAN: thrown from "
                    "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
            }
            sortedTauXxYyZzD[index] = tauXxYyZz;  
            sortedTauXyXzYzD[index] = tauXyXzYz; 
        }
    }
}

__global__ void findCellStartEndD(uint* cellStartD,         // output: cell start index
                                  uint* cellEndD,           // output: cell end index
                                  uint* gridMarkerHashD,    // input: sorted grid hashes
                                  uint* gridMarkerIndexD,   // input: sorted particle indices
                                  const size_t numAllMarkers) {
    extern __shared__ uint sharedHash[];  // blockSize + 1 elements
    /* Get the particle index the current thread is supposed to be looking at. */
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    uint hash;
    /* handle case when no. of particles not multiple of block size */
    if (index < numAllMarkers) {
        hash = gridMarkerHashD[index];
        /* Load hash data into shared memory so that we can look at neighboring
         * particle's hash
         * value without loading two hash values per thread
         */
        sharedHash[threadIdx.x + 1] = hash;

        if (index > 0 && threadIdx.x == 0) {
            /* first thread in block must load neighbor particle hash */
            sharedHash[0] = gridMarkerHashD[index - 1];
        }
    }

    __syncthreads();

    if (index < numAllMarkers) {
        /* If this particle has a different cell index to the previous particle then
         * it must be
         * the first particle in the cell, so store the index of this particle in
         * the cell. As it
         * isn't the first particle, it must also be the cell end of the previous
         * particle's cell
         */
        if (index == 0 || hash != sharedHash[threadIdx.x]) {
            cellStartD[hash] = index;
            if (index > 0)
                cellEndD[sharedHash[threadIdx.x]] = index;
        }

        if (index == numAllMarkers - 1) {
            cellEndD[hash] = index + 1;
        }
    }
}

__global__ void reorderDataD(uint* gridMarkerIndexD,     // input: sorted particle indices
                             uint* extendedActivityIdD,  // input: particles in an extended active sub-domain
                             uint* mapOriginalToSorted,  // input: original index to sorted index
                             Real4* sortedPosRadD,       // output: sorted positions
                             Real3* sortedVelMasD,       // output: sorted velocities
                             Real4* sortedRhoPreMuD,     // output: sorted density pressure
                             Real3* sortedTauXxYyZzD,    // output: sorted total stress xxyyzz
                             Real3* sortedTauXyXzYzD,    // output: sorted total stress xyzxyz
                             Real4* posRadD,             // input: original position array
                             Real3* velMasD,             // input: original velocity array
                             Real4* rhoPresMuD,          // input: original density pressure
                             Real3* tauXxYyZzD,          // input: original total stress xxyyzz
                             Real3* tauXyXzYzD,          // input: original total stress xyzxyz
                             const size_t numAllMarkers) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numAllMarkers)
        return;

    // Now use the sorted index to reorder the pos and vel data
    uint originalIndex = id;

    // no need to do anything if it is not an active particle
    uint activity = extendedActivityIdD[originalIndex];
    if(activity == 0)
        return; 

    // map original to sorted
    uint index = mapOriginalToSorted[originalIndex]; 

    Real3 posRad = mR3(posRadD[originalIndex]);
    Real3 velMas = velMasD[originalIndex];
    Real4 rhoPreMu = rhoPresMuD[originalIndex];

    if (!(isfinite(posRad.x) && isfinite(posRad.y) && isfinite(posRad.z))) {
        printf(
            "Error! particle position is NAN: thrown from "
            "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
    }
    if (!(isfinite(velMas.x) && isfinite(velMas.y) && isfinite(velMas.z))) {
        printf(
            "Error! particle velocity is NAN: thrown from "
            "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
    }
    if (!(isfinite(rhoPreMu.x) && isfinite(rhoPreMu.y) && isfinite(rhoPreMu.z) && isfinite(rhoPreMu.w))) {
        printf(
            "Error! particle rhoPreMu is NAN: thrown from "
            "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
    }

    sortedPosRadD[index] = mR4(posRad, posRadD[originalIndex].w);
    sortedVelMasD[index] = velMas;
    sortedRhoPreMuD[index] = rhoPreMu;

    // For granular material
    if( paramsD.elastic_SPH ) {
        Real3 tauXxYyZz = tauXxYyZzD[originalIndex];  
        Real3 tauXyXzYz = tauXyXzYzD[originalIndex]; 
        if (!(isfinite(tauXxYyZz.x) && isfinite(tauXxYyZz.y) && isfinite(tauXxYyZz.z))) {  
            printf(
                "Error! particle tauXxYyZz is NAN: thrown from "
                "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
        }
        if (!(isfinite(tauXyXzYz.x) && isfinite(tauXyXzYz.y) && isfinite(tauXyXzYz.z))) { 
            printf(
                "Error! particle tauXyXzYz is NAN: thrown from "
                "ChCollisionSystemFsi.cu, reorderDataAndFindCellStartD !\n");
        }
        sortedTauXxYyZzD[index] = tauXxYyZz;  
        sortedTauXyXzYzD[index] = tauXyXzYz; 
    }
}

__global__ void OriginalToSortedD(uint* mapOriginalToSorted,
                                  uint* gridMarkerIndex,
                                  const size_t numAllMarkers) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numAllMarkers)
        return;

    uint index = gridMarkerIndex[id];

    mapOriginalToSorted[index] = id;
}

//--------------------------------------------------------------------------------------------------------------------------------
ChCollisionSystemFsi::ChCollisionSystemFsi(std::shared_ptr<SphMarkerDataD> otherSortedSphMarkersD,
                                           std::shared_ptr<ProximityDataD> otherMarkersProximityD,
                                           std::shared_ptr<FsiGeneralData> otherFsiGeneralData,
                                           std::shared_ptr<SimParams> otherParamsH,
                                           std::shared_ptr<NumberOfObjects> otherNumObjects)
    : sortedSphMarkersD(otherSortedSphMarkersD),
      markersProximityD(otherMarkersProximityD),
      fsiGeneralData(otherFsiGeneralData),
      paramsH(otherParamsH),
      numObjectsH(otherNumObjects) {
    sphMarkersD = NULL;
}
ChCollisionSystemFsi::~ChCollisionSystemFsi() {}

//--------------------------------------------------------------------------------------------------------------------------------
void ChCollisionSystemFsi::Finalize() {
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH.get(), sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH.get(), sizeof(NumberOfObjects));
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChCollisionSystemFsi::calcHash() {
    if (!(markersProximityD->gridMarkerHashD.size() == numObjectsH->numAllMarkers &&
          markersProximityD->gridMarkerIndexD.size() == numObjectsH->numAllMarkers)) {
        printf(
            "mError! calcHash!, gridMarkerHashD.size() %zu "
            "gridMarkerIndexD.size() %zu numObjectsH->numAllMarkers %zu \n",
            markersProximityD->gridMarkerHashD.size(), markersProximityD->gridMarkerIndexD.size(),
            numObjectsH->numAllMarkers);
        throw std::runtime_error("Error! size error, calcHash!");
    }

    bool *isErrorH, *isErrorD;
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    //------------------------------------------------------------------------
    /* Is there a need to optimize the number of threads used at once? */
    uint numThreads, numBlocks;
    computeGridSize((int)numObjectsH->numAllMarkers, 256, numBlocks, numThreads);

    /* Execute Kernel */
    calcHashD<<<numBlocks, numThreads>>>(U1CAST(markersProximityD->gridMarkerHashD),
                                         U1CAST(markersProximityD->gridMarkerIndexD), 
                                         mR4CAST(sphMarkersD->posRadD),
                                         numObjectsH->numAllMarkers, isErrorD);

    /* Check for errors in kernel execution */
    hipDeviceSynchronize();
    cudaCheckError();
    //------------------------------------------------------------------------
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed in  calcHashD!\n");
    }
    hipFree(isErrorD);
    free(isErrorH);
}

void ChCollisionSystemFsi::ResetCellSize(int s) {
    markersProximityD->cellStartD.resize(s);
    markersProximityD->cellEndD.resize(s);
}

void ChCollisionSystemFsi::reorderDataAndFindCellStart() {
    int3 cellsDim = paramsH->gridSize;
    int numCells = cellsDim.x * cellsDim.y * cellsDim.z;
    if (!(markersProximityD->cellStartD.size() == numCells && markersProximityD->cellEndD.size() == numCells)) {
        throw std::runtime_error("Error! size error, reorderDataAndFindCellStart!\n");
    }

    thrust::fill(markersProximityD->cellStartD.begin(), markersProximityD->cellStartD.end(), 0);
    thrust::fill(markersProximityD->cellEndD.begin(), markersProximityD->cellEndD.end(), 0);

    uint numThreads, numBlocks;
    computeGridSize((uint)numObjectsH->numAllMarkers, 256, numBlocks, numThreads);  // 256 is blockSize

    uint smemSize = sizeof(uint) * (numThreads + 1);
    // Find the start index and the end index of the sorted array in each cell
    findCellStartEndD<<<numBlocks, numThreads, smemSize>>>(
        U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),          
        U1CAST(markersProximityD->gridMarkerHashD), U1CAST(markersProximityD->gridMarkerIndexD),
        numObjectsH->numAllMarkers);
    hipDeviceSynchronize();
    cudaCheckError();

    // Launch a kernel to find the location of original particles in the sorted arrays.
    // This is faster than using thrust::sort_by_key()
    OriginalToSortedD<<<numBlocks, numThreads>>>(
        U1CAST(markersProximityD->mapOriginalToSorted),
        U1CAST(markersProximityD->gridMarkerIndexD), numObjectsH->numAllMarkers);

    // Reorder the arrays according to the sorted index of all particles
    reorderDataD<<<numBlocks, numThreads>>>(
        U1CAST(markersProximityD->gridMarkerIndexD),
        U1CAST(fsiGeneralData->extendedActivityIdD),
        U1CAST(markersProximityD->mapOriginalToSorted),
        mR4CAST(sortedSphMarkersD->posRadD), mR3CAST(sortedSphMarkersD->velMasD), 
        mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(sortedSphMarkersD->tauXxYyZzD), 
        mR3CAST(sortedSphMarkersD->tauXyXzYzD), mR4CAST(sphMarkersD->posRadD), 
        mR3CAST(sphMarkersD->velMasD), mR4CAST(sphMarkersD->rhoPresMuD), 
        mR3CAST(sphMarkersD->tauXxYyZzD), mR3CAST(sphMarkersD->tauXyXzYzD), 
        numObjectsH->numAllMarkers);
    hipDeviceSynchronize();
    cudaCheckError();
}

void ChCollisionSystemFsi::ArrangeData(std::shared_ptr<SphMarkerDataD> otherSphMarkersD) {
    sphMarkersD = otherSphMarkersD;
    int3 cellsDim = paramsH->gridSize;
    int numCells = cellsDim.x * cellsDim.y * cellsDim.z;
    ResetCellSize(numCells);
    calcHash();
    thrust::sort_by_key(markersProximityD->gridMarkerHashD.begin(), markersProximityD->gridMarkerHashD.end(),
                        markersProximityD->gridMarkerIndexD.begin());
    reorderDataAndFindCellStart();
}

}  // end namespace fsi
}  // end namespace chrono
