#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Arman Pazouki, Wei Hu
// =============================================================================
#include <thrust/extrema.h>
#include <thrust/sort.h>
#include "chrono_fsi/physics/ChFsiForceExplicitSPH.cuh"

//================================================================================================================================
namespace chrono {
namespace fsi {


__device__ __inline__ void calc_G_Matrix(Real4* sortedPosRad,
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real* G_i,
                                         uint* cellStart,
                                         uint* cellEnd,
                                         const size_t numAllMarkers) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    // This is the elements of inverse of G
    Real mGi[9] = {0.0};
    // examine neighbouring cells
    for (int z = -1; z <= 1; z++)
        for (int y = -1; y <= 1; y++)
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell50
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 rij = Distance(posRadA, posRadB);
                        Real d = length(rij);
                        if (d > SuppRadii || sortedRhoPreMu[j].w <= -2)
                            continue;
                        Real3 grad_i_wij = GradWh(rij, h_i);
                        Real3 grw_vj = grad_i_wij * paramsD.volume0;
                        mGi[0] -= rij.x * grw_vj.x;
                        mGi[1] -= rij.x * grw_vj.y;
                        mGi[2] -= rij.x * grw_vj.z;
                        mGi[3] -= rij.y * grw_vj.x;
                        mGi[4] -= rij.y * grw_vj.y;
                        mGi[5] -= rij.y * grw_vj.z;
                        mGi[6] -= rij.z * grw_vj.x;
                        mGi[7] -= rij.z * grw_vj.y;
                        mGi[8] -= rij.z * grw_vj.z;
                    }
                }
            }
    Real Det = (mGi[0] * mGi[4] * mGi[8] - 
                mGi[0] * mGi[5] * mGi[7] - 
                mGi[1] * mGi[3] * mGi[8] +
                mGi[1] * mGi[5] * mGi[6] + 
                mGi[2] * mGi[3] * mGi[7] - 
                mGi[2] * mGi[4] * mGi[6]);
    if (abs(Det) > 0.01) {
        Real OneOverDet = 1.0/Det;
        G_i[0] =  (mGi[4] * mGi[8] - mGi[5] * mGi[7]) * OneOverDet;
        G_i[1] = -(mGi[1] * mGi[8] - mGi[2] * mGi[7]) * OneOverDet;
        G_i[2] =  (mGi[1] * mGi[5] - mGi[2] * mGi[4]) * OneOverDet;
        G_i[3] = -(mGi[3] * mGi[8] - mGi[5] * mGi[6]) * OneOverDet;
        G_i[4] =  (mGi[0] * mGi[8] - mGi[2] * mGi[6]) * OneOverDet;
        G_i[5] = -(mGi[0] * mGi[5] - mGi[2] * mGi[3]) * OneOverDet;
        G_i[6] =  (mGi[3] * mGi[7] - mGi[4] * mGi[6]) * OneOverDet;
        G_i[7] = -(mGi[0] * mGi[7] - mGi[1] * mGi[6]) * OneOverDet;
        G_i[8] =  (mGi[0] * mGi[4] - mGi[1] * mGi[3]) * OneOverDet;
    } else {
        for (int i = 0; i < 9; i++) {
            G_i[i] = 0.0;
        }
        G_i[0] = 1;
        G_i[4] = 1;
        G_i[8] = 1;
    }
}

__device__ __inline__ void calc_A_Matrix(Real4* sortedPosRad,
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real* A_i,
                                         Real* G_i,
                                         uint* cellStart,
                                         uint* cellEnd,
                                         const size_t numAllMarkers) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    // examine neighbouring cells
    for (int z = -1; z <= 1; z++)
        for (int y = -1; y <= 1; y++)
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell50
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 rij = Distance(posRadA, posRadB);
                        Real d = length(rij);
                        if (d > SuppRadii || sortedRhoPreMu[j].w <= -2)
                            continue;
                        Real h_j = sortedPosRad[j].w;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_ij = GradWh(rij, h_ij);
                        Real V_j = paramsD.markerMass/paramsD.rho0;
                        Real com_part = 0;
                        com_part = (G_i[0] * grad_ij.x + G_i[1] * grad_ij.y + G_i[2] * grad_ij.z) * V_j;
                        A_i[0] += rij.x * rij.x * com_part;  // 111
                        A_i[1] += rij.x * rij.y * com_part;  // 112
                        A_i[2] += rij.x * rij.z * com_part;  // 113
                        A_i[3] += rij.y * rij.x * com_part;  // 121
                        A_i[4] += rij.y * rij.y * com_part;  // 122
                        A_i[5] += rij.y * rij.z * com_part;  // 123
                        A_i[6] += rij.z * rij.x * com_part;  // 131
                        A_i[7] += rij.z * rij.y * com_part;  // 132
                        A_i[8] += rij.z * rij.z * com_part;  // 133
                        com_part = (G_i[3] * grad_ij.x + G_i[4] * grad_ij.y + G_i[5] * grad_ij.z) * V_j;
                        A_i[9] += rij.x * rij.x * com_part;   // 211
                        A_i[10] += rij.x * rij.y * com_part;  // 212
                        A_i[11] += rij.x * rij.z * com_part;  // 213
                        A_i[12] += rij.y * rij.x * com_part;  // 221
                        A_i[13] += rij.y * rij.y * com_part;  // 222
                        A_i[14] += rij.y * rij.z * com_part;  // 223
                        A_i[15] += rij.z * rij.x * com_part;  // 231
                        A_i[16] += rij.z * rij.y * com_part;  // 232
                        A_i[17] += rij.z * rij.z * com_part;  // 233
                        com_part = (G_i[6] * grad_ij.x + G_i[7] * grad_ij.y + G_i[8] * grad_ij.z) * V_j;
                        A_i[18] += rij.x * rij.x * com_part;  // 311
                        A_i[19] += rij.x * rij.y * com_part;  // 312
                        A_i[20] += rij.x * rij.z * com_part;  // 313
                        A_i[21] += rij.y * rij.x * com_part;  // 321
                        A_i[22] += rij.y * rij.y * com_part;  // 322
                        A_i[23] += rij.y * rij.z * com_part;  // 323
                        A_i[24] += rij.z * rij.x * com_part;  // 331
                        A_i[25] += rij.z * rij.y * com_part;  // 332
                        A_i[26] += rij.z * rij.z * com_part;  // 333
                    }
                }
            }

}

__device__ __inline__ void calc_L_Matrix(Real4* sortedPosRad,
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real* A_i,
                                         Real* L_i,
                                         Real* G_i,
                                         uint* cellStart,
                                         uint* cellEnd,
                                         const size_t numAllMarkers) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    Real B[36] = {0.0};
    Real L[6] = {0.0};

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    // examine neighbouring cells
    for (int z = -1; z <= 1; z++)
        for (int y = -1; y <= 1; y++)
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                // get start of bucket for this cell50
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {  // cell is not empty
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 rij = Distance(posRadA, posRadB);
                        Real d = length(rij);
                        if (d > SuppRadii || sortedRhoPreMu[j].w <= -2)
                            continue;
                        Real3 eij = rij / d;

                        Real h_j = sortedPosRad[j].w;
                        // Real m_j = paramsD.markerMass;
                        Real h_ij = 0.5 * (h_j + h_i);
                        Real3 grad_ij = GradWh(rij, h_ij);
                        Real V_j = paramsD.markerMass/paramsD.rho0;
                        Real com_part = 0;
                        // mn=11

                        Real XX = (eij.x * grad_ij.x);
                        Real XY = (eij.x * grad_ij.y + eij.y * grad_ij.x);
                        Real XZ = (eij.x * grad_ij.z + eij.z * grad_ij.x);
                        Real YY = (eij.y * grad_ij.y);
                        Real YZ = (eij.y * grad_ij.z + eij.z * grad_ij.y);
                        Real ZZ = (eij.z * grad_ij.z);

                        com_part = (A_i[0] * eij.x + A_i[9] * eij.y + A_i[18] * eij.z + rij.x * eij.x) * V_j;
                        B[6 * 0 + 0] += com_part * XX;  // 11
                        B[6 * 0 + 1] += com_part * XY;  // 12
                        B[6 * 0 + 2] += com_part * XZ;  // 13
                        B[6 * 0 + 3] += com_part * YY;  // 14
                        B[6 * 0 + 4] += com_part * YZ;  // 15
                        B[6 * 0 + 5] += com_part * ZZ;  // 15
                        // mn=12
                        com_part = (A_i[1] * eij.x + A_i[10] * eij.y + A_i[19] * eij.z + rij.x * eij.y) * V_j;
                        B[6 * 1 + 0] += com_part * XX;  // 21
                        B[6 * 1 + 1] += com_part * XY;  // 22
                        B[6 * 1 + 2] += com_part * XZ;  // 23
                        B[6 * 1 + 3] += com_part * YY;  // 24
                        B[6 * 1 + 4] += com_part * YZ;  // 25
                        B[6 * 1 + 5] += com_part * ZZ;  // 25

                        // mn=13
                        com_part = (A_i[2] * eij.x + A_i[11] * eij.y + A_i[20] * eij.z + rij.x * eij.z) * V_j;
                        B[6 * 2 + 0] += com_part * XX;  // 31
                        B[6 * 2 + 1] += com_part * XY;  // 32
                        B[6 * 2 + 2] += com_part * XZ;  // 33
                        B[6 * 2 + 3] += com_part * YY;  // 34
                        B[6 * 2 + 4] += com_part * YZ;  // 35
                        B[6 * 2 + 5] += com_part * ZZ;  // 36

                        // Note that we skip mn=21 since it is similar to mn=12
                        // mn=22
                        com_part = (A_i[4] * eij.x + A_i[13] * eij.y + A_i[22] * eij.z + rij.y * eij.y) * V_j;
                        B[6 * 3 + 0] += com_part * XX;  // 41
                        B[6 * 3 + 1] += com_part * XY;  // 42
                        B[6 * 3 + 2] += com_part * XZ;  // 43
                        B[6 * 3 + 3] += com_part * YY;  // 44
                        B[6 * 3 + 4] += com_part * YZ;  // 45
                        B[6 * 3 + 5] += com_part * ZZ;  // 46

                        // mn=23
                        com_part = (A_i[5] * eij.x + A_i[14] * eij.y + A_i[23] * eij.z + rij.y * eij.z) * V_j;
                        B[6 * 4 + 0] += com_part * XX;  // 51
                        B[6 * 4 + 1] += com_part * XY;  // 52
                        B[6 * 4 + 2] += com_part * XZ;  // 53
                        B[6 * 4 + 3] += com_part * YY;  // 54
                        B[6 * 4 + 4] += com_part * YZ;  // 55
                        B[6 * 4 + 5] += com_part * ZZ;  // 56
                        // mn=33
                        com_part = (A_i[8] * eij.x + A_i[17] * eij.y + A_i[26] * eij.z + rij.z * eij.z) * V_j;
                        B[6 * 5 + 0] += com_part * XX;  // 61
                        B[6 * 5 + 1] += com_part * XY;  // 62
                        B[6 * 5 + 2] += com_part * XZ;  // 63
                        B[6 * 5 + 3] += com_part * YY;  // 64
                        B[6 * 5 + 4] += com_part * YZ;  // 65
                        B[6 * 5 + 5] += com_part * ZZ;  // 66
                    }
                }
            }

    inv6xdelta_mn(B, L);
    L_i[0] = L[0];
    L_i[1] = L[1];
    L_i[2] = L[2];
    L_i[3] = L[1];
    L_i[4] = L[3];
    L_i[5] = L[4];
    L_i[6] = L[2];
    L_i[7] = L[4];
    L_i[8] = L[5];

    // Real Det = (L_i[0] * L_i[4] * L_i[8] - L_i[0] * L_i[5] * L_i[7] - L_i[1] * L_i[3] * L_i[8] +
    //             L_i[1] * L_i[5] * L_i[6] + L_i[2] * L_i[3] * L_i[7] - L_i[2] * L_i[4] * L_i[6]);
    // if (abs(Det) < 0.01) {
    //     for (int i = 0; i < 9; i++) {
    //         L_i[0 * 9 + i] = 0.0;
    //         L_i[0 * 9 + 0] = 1;
    //         L_i[0 * 9 + 4] = 1;
    //         L_i[0 * 9 + 8] = 1;
    //     }
    // }
    // printf("L Det %f\n", Det);

}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Shear_Stress_Rate(Real4* sortedPosRad,
                                  Real4* sortedRhoPreMu,
                                  Real3* sortedVelMas,
                                  Real3* velMas_ModifiedBCE,
                                  Real4* rhoPreMu_ModifiedBCE,
                                  Real3* sortedTauXxYyZz,
                                  Real3* sortedTauXyXzYz,
                                  Real3* sortedDerivTauXxYyZz,
                                  Real3* sortedDerivTauXyXzYz,
                                  uint* gridMarkerIndex,
                                  uint* cellStart,
                                  uint* cellEnd,
                                  const size_t numAllMarkers) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numAllMarkers) {
        return;
    }
    if (sortedRhoPreMu[index].w > -0.5) {
        return;
    }

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real3 velMasA = sortedVelMas[index];
    Real hA = sortedPosRad[index].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    Real tauxx = sortedTauXxYyZz[index].x;
    Real tauyy = sortedTauXxYyZz[index].y;
    Real tauzz = sortedTauXxYyZz[index].z;
    Real tauxy = sortedTauXyXzYz[index].x;
    Real tauxz = sortedTauXyXzYz[index].y;
    Real tauyz = sortedTauXyXzYz[index].z;
    Real tauzx = tauxz;
    Real tauzy = tauyz;
    Real tauyx = tauxy;
    Real dTauxx = 0.0;
    Real dTauyy = 0.0;
    Real dTauzz = 0.0;
    Real dTauxy = 0.0;
    Real dTauxz = 0.0;
    Real dTauyz = 0.0;

    Real G_i[9] = {0.0};
    calc_G_Matrix(sortedPosRad,sortedVelMas,sortedRhoPreMu,G_i,cellStart,cellEnd,numAllMarkers);

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    if (j != index) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real d = length(dist3);

                        if (d > SuppRadii)
                            continue;
                        Real3 velMasB = sortedVelMas[j];
                        Real4 rhoPresMuB = sortedRhoPreMu[j];
                        if (rhoPresMuB.w > -1.0) {
                            int bceIndexB = gridMarkerIndex[j] - (numObjectsD.numFluidMarkers);
                            if (!(bceIndexB >= 0 &&
                                  bceIndexB < numObjectsD.numBoundaryMarkers + numObjectsD.numRigid_SphMarkers)) {
                                printf("Error! bceIndex out of bound, collideCell !\n");
                            }
                            rhoPresMuB = rhoPreMu_ModifiedBCE[bceIndexB]; // to check
                            velMasB = velMas_ModifiedBCE[bceIndexB]; // to check
                            velMasB = 2.0*velMasB - velMasA; // noslip BC
                        }
                        Real rhoB = rhoPresMuB.x;
                        Real hB = sortedPosRad[j].w;
                        Real mB = paramsD.markerMass;
                        Real3 gradW = GradWh(dist3, (hA + hB) * 0.5);

                        Real3 gradW_new;
                        gradW_new.x = G_i[0]*gradW.x + G_i[1]*gradW.y + G_i[2]*gradW.z;
                        gradW_new.y = G_i[3]*gradW.x + G_i[4]*gradW.y + G_i[5]*gradW.z;
                        gradW_new.z = G_i[6]*gradW.x + G_i[7]*gradW.y + G_i[8]*gradW.z;
                        gradW = gradW_new;

                        // start to calculate the rate
                        Real Gm = paramsD.G_shear;  // shear modulus of the material
                        Real half_mB_over_rhoB = 0.5 * (mB / rhoB);
                        Real3 vAB = velMasA - velMasB;
                        Real3 vAB_h = (velMasA - velMasB) * half_mB_over_rhoB;
                        // entries of strain rate tensor
                        Real exx = -2.0 * vAB_h.x * gradW.x;
                        Real eyy = -2.0 * vAB_h.y * gradW.y;
                        Real ezz = -2.0 * vAB_h.z * gradW.z;
                        Real exy = -vAB_h.x * gradW.y - vAB_h.y * gradW.x;
                        Real exz = -vAB_h.x * gradW.z - vAB_h.z * gradW.x;
                        Real eyz = -vAB_h.y * gradW.z - vAB_h.z * gradW.y;
                        // entries of rotation rate (spin) tensor
                        // Real wxx = 0.0;
                        // Real wyy = 0.0;
                        // Real wzz = 0.0;
                        Real wxy = -vAB_h.x * gradW.y + vAB_h.y * gradW.x;
                        Real wxz = -vAB_h.x * gradW.z + vAB_h.z * gradW.x;
                        Real wyz = -vAB_h.y * gradW.z + vAB_h.z * gradW.y;
                        Real wyx = -wxy;
                        // Real wzx = -wxz;
                        Real wzy = -wyz;

                        Real edia = 1.0 / 3.0 * (exx + eyy + ezz);
                        Real twoGm = 2.0 * Gm;
                        Real K_edia = paramsD.K_bulk*1.0*edia;
                        dTauxx += twoGm * (exx - edia) + 2.0 * (tauxy * wxy + tauxz * wxz) + K_edia;
                        dTauyy += twoGm * (eyy - edia) - 2.0 * (tauyx * wxy - tauyz * wyz) + K_edia;
                        dTauzz += twoGm * (ezz - edia) - 2.0 * (tauzx * wxz + tauzy * wyz) + K_edia;
                        dTauxy += twoGm * exy - (tauxx * wxy + tauxz * wzy) + (wxy * tauyy + wxz * tauzy);
                        dTauxz += twoGm * exz - (tauxx * wxz + tauxy * wyz) + (wxy * tauyz + wxz * tauzz);
                        dTauyz += twoGm * eyz - (tauyx * wxz + tauyy * wyz) + (wyx * tauxz + wyz * tauzz);
                    }
                }
            }
        }
    }
    sortedDerivTauXxYyZz[index] = mR3(dTauxx, dTauyy, dTauzz);
    sortedDerivTauXyXzYz[index] = mR3(dTauxy, dTauxz, dTauyz);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calcRho_kernel(Real4* sortedPosRad,
                               Real4* sortedRhoPreMu,
                               Real4* sortedRhoPreMu_old,
                               Real* _sumWij_rhoi,
                               uint* cellStart,
                               uint* cellEnd,
                               const size_t numAllMarkers,
                               int density_reinit,
                               volatile bool* isErrorD) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    sortedRhoPreMu_old[i_idx].y = Eos(sortedRhoPreMu_old[i_idx].x, sortedRhoPreMu_old[i_idx].w);

    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real h_i = sortedPosRad[i_idx].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    Real sum_mW = 0;
    Real sum_mW_rho = 0.0000001;

    Real sum_W = 0.0;
    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                uint startIndex = cellStart[gridHash];
                if (startIndex != 0xffffffff) {
                    uint endIndex = cellEnd[gridHash];
                    for (uint j = startIndex; j < endIndex; j++) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real d = length(dist3);
                        if (d > SuppRadii)
                            continue;
                        if (sortedRhoPreMu_old[j].w == -1) {  //
                            Real h_j = sortedPosRad[j].w;
                            Real m_j = paramsD.markerMass;  // cube(h_j * paramsD.MULT_INITSPACE) * paramsD.rho0;
                            Real W3 = W3h_GPU(d, 0.5 * (h_j + h_i));
                            sum_mW += m_j * W3;
                            sum_W += W3;
                            sum_mW_rho += m_j * W3 / sortedRhoPreMu_old[j].x;
                        }
                    }
                }
            }
        }
    }
    //    sumWij_inv[i_idx] = paramsD.markerMass / sum_mW;

    // sortedRhoPreMu[i_idx].x = sum_mW;
    if ((density_reinit == 0) && (sortedRhoPreMu[i_idx].w == -1))
        sortedRhoPreMu[i_idx].x = sum_mW / sum_mW_rho;

    if ((sortedRhoPreMu[i_idx].x > 3 * paramsD.rho0 || sortedRhoPreMu[i_idx].x < 0.01 * paramsD.rho0) &&
        sortedRhoPreMu[i_idx].w == -1)
        printf("(calcRho_kernel)density marker %d, sum_mW=%f, sum_W=%f, h_i=%f\n", i_idx, sum_mW, sum_W, h_i);
}

//--------------------------------------------------------------------------------------------------------------------------------
// modify pressure for body force
__device__ __inline__ void modifyPressure(Real4& rhoPresMuB, const Real3& dist3Alpha) {
    // body force in x direction
    rhoPresMuB.y = (dist3Alpha.x > 0.5 * paramsD.boxDims.x) ? (rhoPresMuB.y - paramsD.deltaPress.x) : rhoPresMuB.y;
    rhoPresMuB.y = (dist3Alpha.x < -0.5 * paramsD.boxDims.x) ? (rhoPresMuB.y + paramsD.deltaPress.x) : rhoPresMuB.y;
    // body force in x direction
    rhoPresMuB.y = (dist3Alpha.y > 0.5 * paramsD.boxDims.y) ? (rhoPresMuB.y - paramsD.deltaPress.y) : rhoPresMuB.y;
    rhoPresMuB.y = (dist3Alpha.y < -0.5 * paramsD.boxDims.y) ? (rhoPresMuB.y + paramsD.deltaPress.y) : rhoPresMuB.y;
    // body force in x direction
    rhoPresMuB.y = (dist3Alpha.z > 0.5 * paramsD.boxDims.z) ? (rhoPresMuB.y - paramsD.deltaPress.z) : rhoPresMuB.y;
    rhoPresMuB.y = (dist3Alpha.z < -0.5 * paramsD.boxDims.z) ? (rhoPresMuB.y + paramsD.deltaPress.z) : rhoPresMuB.y;
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 CubicSolve(Real aa, Real bb, Real cc, Real dd) {
    Real disc, q, r, dum1, dum2, term1, r13;
    bb /= aa;
    cc /= aa;
    dd /= aa;
    if (aa == 0) {
        return mR3(0, 0, 0);
    }
    if (abs(bb) < 1e-9) {
        return mR3(0, 0, 0);
    }
    if (abs(cc) < 1e-9) {
        return mR3(0, 0, 0);
    }
    if (abs(dd) < 1e-9) {
        return mR3(0, 0, 0);
    }
    q = (3.0 * cc - (bb * bb)) / 9.0;
    r = -(27.0 * dd) + bb * (9.0 * cc - 2.0 * (bb * bb));
    r /= 54.0;
    disc = q * q * q + r * r;
    term1 = (bb / 3.0);

    /*     dataForm.x1Im.value = 0; //The first root is always real.
        if (disc > 0) { // one root real, two are complex
            s = r + Math.sqrt(disc);
            s = ((s < 0) ? -Math.pow(-s, (1.0/3.0)) : Math.pow(s, (1.0/3.0)));
            t = r - Math.sqrt(disc);
            t = ((t < 0) ? -Math.pow(-t, (1.0/3.0)) : Math.pow(t, (1.0/3.0)));
            dataForm.x1Re.value = -term1 + s + t;
            term1 += (s + t)/2.0;
            dataForm.x3Re.value = dataForm.x2Re.value = -term1;
            term1 = Math.sqrt(3.0)*(-t + s)/2;
            dataForm.x2Im.value = term1;
            dataForm.x3Im.value = -term1;
            return;
        }
        // End if (disc > 0)
        // The remaining options are all real
        dataForm.x3Im.value = dataForm.x2Im.value = 0;
        if (disc == 0){ // All roots real, at least two are equal.
            r13 = ((r < 0) ? -Math.pow(-r,(1.0/3.0)) : Math.pow(r,(1.0/3.0)));
            dataForm.x1Re.value = -term1 + 2.0*r13;
            dataForm.x3Re.value = dataForm.x2Re.value = -(r13 + term1);
            return;
        } // End if (disc == 0)
    */

    Real xRex, xRey, xRez;
    // have complex root
    if (disc > 0) {
        xRex = 0.0;
        xRey = 0.0;
        xRez = 0.0;
        return mR3(xRex, xRey, xRez);
    }
    // All roots real, at least two are equal.
    if (disc == 0) {
        if (r < 0) {
            r13 = pow(-r, (1.0 / 3.0));
        } else {
            r13 = pow(r, (1.0 / 3.0));
        }
        xRex = -term1 + 2.0 * r13;
        xRey = -(r13 + term1);
        xRez = xRey;
        return mR3(xRex, xRey, xRez);
    }
    // All roots are real and unequal (to get here, q < 0)
    q = -q;
    dum1 = q * q * q;
    dum2 = r / (sqrt(dum1 + 1.0e-9));
    if ((dum2 >= 0) && (dum2 <= 1)) {
        dum1 = acos(dum2);
    } else {
        xRex = 0.0;
        xRey = 0.0;
        xRez = 0.0;
        return mR3(xRex, xRey, xRez);
    }
    r13 = 2.0 * sqrt(q);
    xRex = -term1 + r13 * cos(dum1 / 3.0);
    xRey = -term1 + r13 * cos((dum1 + 2.0 * 3.1415926) / 3.0);
    xRez = -term1 + r13 * cos((dum1 + 4.0 * 3.1415926) / 3.0);

    return mR3(xRex, xRey, xRez);
}
__device__ inline Real3 CubicEigen(Real4 c1, Real4 c2, Real4 c3) {
    Real a = c1.x;
    Real b = c1.y;
    Real c = c1.z;
    Real d = c1.w;

    Real l = c2.x;
    Real m = c2.y;
    Real n = c2.z;
    Real k = c2.w;

    Real p = c3.x;
    Real q = c3.y;
    Real r = c3.z;
    Real s = c3.w;

    Real D = (a * m * r + b * p * n + c * l * q) - (a * n * q + b * l * r + c * m * p) + 1.0e-9;
    Real x = ((b * r * k + c * m * s + d * n * q) - (b * n * s + c * q * k + d * m * r)) / D;
    Real y = ((a * n * s + c * p * k + d * l * r) - (a * r * k + c * l * s + d * n * p)) / D;
    Real z = ((a * q * k + b * l * s + d * m * p) - (a * m * s + b * p * k + d * l * q)) / D;

    b = b + 1.0e-9;
    x = 1.0e0;
    z = (-l + a * m / b) / (n - c * m / b);
    y = (-a - c * z) / b;
    Real R = sqrt(x * x + y * y + z * z);
    x = x / R;
    y = y / R;
    z = z / R;

    // if(abs(D) < 1){
    //     return mR3(0,0,0);
    // }

    // if(abs(m) < 0.1){
    //     x=0;
    //     y=1;
    //     z=0;
    //     return mR3(x,y,z);
    // }
    // else{
    //     y=0;
    //     if(abs(c) > 0.1){
    //         x=1;
    //         z=-a/c;
    //         return mR3(x,y,z);
    //     }
    //     if(abs(a) > 0.1){
    //         z=1;
    //         x=-c/a;
    //         return mR3(x,y,z);
    //     }
    // }

    return mR3(x, y, z);
}

//--------------------------------------------------------------------------------------------------------------------------------
/**	
 * @brief DifVelocityRho
 * @details  See SDKCollisionSystem.cuh
 */
__device__ inline Real4 DifVelocityRho(float G_i[9],
                                       Real3 dist3,
                                       Real d,
                                       Real4 posRadA,
                                       Real4 posRadB,
                                       Real3 velMasA,
                                       Real3 vel_XSPH_A,
                                       Real3 velMasB,
                                       Real3 vel_XSPH_B,
                                       Real4 rhoPresMuA,
                                       Real4 rhoPresMuB,
                                       Real multViscosity) {
    Real3 gradW = GradWh(dist3, (posRadA.w + posRadB.w) * 0.5);

    // Real3 gradW_new;
    // gradW_new.x = G_i[0]*gradW.x + G_i[1]*gradW.y + G_i[2]*gradW.z;
    // gradW_new.y = G_i[3]*gradW.x + G_i[4]*gradW.y + G_i[5]*gradW.z;
    // gradW_new.z = G_i[6]*gradW.x + G_i[7]*gradW.y + G_i[8]*gradW.z;
    // gradW = gradW_new;

    //    Real vAB_Dot_rAB = dot(velMasA - velMasB, dist3);
    //
    //    //	//*** Artificial viscosity type 1.1
    //    Real alpha = .001;
    //    Real c_ab = 10 * paramsD.v_Max;  // Ma = .1;//sqrt(7.0f * 10000 /
    //                                     //    ((rhoPresMuA.x + rhoPresMuB.x) / 2.0f));
    //                                     // Real h = paramsD.HSML;
    //    Real rho = .5f * (rhoPresMuA.x + rhoPresMuB.x);
    //    Real nu = alpha * paramsD.HSML * c_ab / rho;
    //
    //    //*** Artificial viscosity type 1.2
    //    //    Real nu = 22.8f * paramsD.mu0 / 2.0f / (rhoPresMuA.x * rhoPresMuB.x);
    //    Real3 derivV = -paramsD.markerMass *
    //                   (rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x) -
    //                    nu * vAB_Dot_rAB / (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML)) *
    //                   gradW;
    //    return mR4(derivV, rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x * dot(vel_XSPH_A - vel_XSPH_B, gradW));

    //*** Artificial viscosity type 2
    if (rhoPresMuA.w > -1 && rhoPresMuB.w > -1)
        return mR4(0.0);

    Real rAB_Dot_GradWh = dot(dist3, gradW);
    Real rAB_Dot_GradWh_OverDist = rAB_Dot_GradWh / (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML);
    Real3 derivV = - paramsD.markerMass *(rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)) * gradW
                   + paramsD.markerMass * (8.0f * multViscosity) * paramsD.mu0 
                   * rAB_Dot_GradWh_OverDist * (velMasA - velMasB) / square(rhoPresMuA.x + rhoPresMuB.x);

    //    Real derivRho = rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x * dot(vel_XSPH_A - vel_XSPH_B, gradW);
    //	Real zeta = 0;//.05;//.1;
    //	Real derivRho = rhoPresMuA.x * paramsD.markerMass * invrhoPresMuBx *
    //(dot(vel_XSPH_A - vel_XSPH_B, gradW)
    //			+ zeta * paramsD.HSML * (10 * paramsD.v_Max) * 2 * (rhoPresMuB.x
    /// rhoPresMuA.x - 1) *
    // rAB_Dot_GradWh_OverDist
    //			);

    //--------------------------------
    // Ferrari Modification
    Real derivRho = paramsD.markerMass * dot(vel_XSPH_A - vel_XSPH_B, gradW);
    //    Real cA = FerrariCi(rhoPresMuA.x);
    //    Real cB = FerrariCi(rhoPresMuB.x);
    //    derivRho += rAB_Dot_GradWh / (d + paramsD.epsMinMarkersDis * paramsD.HSML) * max(cA, cB) / rhoPresMuB.x *
    //                (rhoPresMuB.x - rhoPresMuA.x);

    //*** Artificial viscosity
    Real vAB_Dot_rAB = dot(velMasA - velMasB, dist3);
    // if (vAB_Dot_rAB < 0.0) {
    //     if ((rhoPresMuA.w < - 0.5)  && (rhoPresMuB.w < - 0.5)){ // only for fluid particles
            Real alpha = 0.0;//paramsD.Ar_vis_alpha;
            Real c_ab = paramsD.Cs;
            Real rho = 0.5f * (rhoPresMuA.x * rhoPresMuB.x);
            Real nu = -alpha * paramsD.HSML * c_ab / rho;
            Real derivM1 = -paramsD.markerMass * (nu * vAB_Dot_rAB / (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML));
            derivV.x += derivM1 * gradW.x;
            derivV.y += derivM1 * gradW.y;
            derivV.z += derivM1 * gradW.z;
    //     }
    // }

    //    --------------------------------
    return mR4(derivV, derivRho);

    //	//*** Artificial viscosity type 1.3
    //    Real rAB_Dot_GradWh = dot(dist3, gradW);
    //    Real3 derivV = -paramsD.markerMass *
    //                       (rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x *
    //                       rhoPresMuB.x)) * gradW +
    //                   paramsD.markerMass / (rhoPresMuA.x * rhoPresMuB.x) * 2.0f * paramsD.mu0 * rAB_Dot_GradWh /
    //                       (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML) * (velMasA - velMasB);
    //    return mR4(derivV, rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x * dot(vel_XSPH_A - vel_XSPH_B, gradW));
}

/// Only for modelling elastic and granular problems
__device__ inline Real4 DifVelocityRho_ElasticSPH(Real3 gradW,
                                                  Real3 dist3,
                                                  Real d,
                                                  Real invd,
                                                  Real4 posRadA,
                                                  Real4 posRadB,
                                                  Real3 velMasA_in,
                                                  Real3 vel_XSPH_A_in,
                                                  Real3 velMasB_in,
                                                  Real3 vel_XSPH_B_in,
                                                  Real4 rhoPresMuA,
                                                  Real4 rhoPresMuB,
                                                  Real multViscosity,
                                                  Real3 tauXxYyZz_A_in,
                                                  Real3 tauXyXzYz_A_in,
                                                  Real3 tauXxYyZz_B_in,
                                                  Real3 tauXyXzYz_B_in) {
    // if (rhoPresMuA.w > -1 )
    //     return mR4(0.0);
    // if (rhoPresMuB.w > -1 )
    //     return mR4(0.0);
    Real3 velMasA        = velMasA_in;
    Real3 velMasB        = velMasB_in;
    Real3 vel_XSPH_A     = vel_XSPH_A_in;
    Real3 vel_XSPH_B     = vel_XSPH_B_in;
    Real3 tauXxYyZz_A    = tauXxYyZz_A_in;
    Real3 tauXxYyZz_B    = tauXxYyZz_B_in;
    Real3 tauXyXzYz_A    = tauXyXzYz_A_in;
    Real3 tauXyXzYz_B    = tauXyXzYz_B_in;
    if (rhoPresMuA.w > -0.5 && rhoPresMuB.w > -0.5)
        return mR4(0.0);
    if (rhoPresMuA.w < -0.5 && rhoPresMuB.w > -0.5){
        tauXxYyZz_B = tauXxYyZz_A;
        tauXyXzYz_B = tauXyXzYz_A;
        vel_XSPH_B = 2.0*vel_XSPH_B - vel_XSPH_A; // noslip BC
        // velMasB = 2.0*velMasB - velMasA; // noslip BC
    }
    if (rhoPresMuA.w > -0.5 && rhoPresMuB.w < -0.5){
        tauXxYyZz_A = tauXxYyZz_B;
        tauXyXzYz_A = tauXyXzYz_B;
    }

    Real txxA = tauXxYyZz_A.x;
    Real tyyA = tauXxYyZz_A.y;
    Real tzzA = tauXxYyZz_A.z;
    Real txyA = tauXyXzYz_A.x;
    Real txzA = tauXyXzYz_A.y;
    Real tyzA = tauXyXzYz_A.z;

    Real txxB = tauXxYyZz_B.x;
    Real tyyB = tauXxYyZz_B.y;
    Real tzzB = tauXxYyZz_B.z;
    Real txyB = tauXyXzYz_B.x;
    Real txzB = tauXyXzYz_B.y;
    Real tyzB = tauXyXzYz_B.z;

    // Real PA = rhoPresMuA.y;
    // Real PB = rhoPresMuB.y;
    // Real rhoA = rhoPresMuA.x;
    // Real rhoB = rhoPresMuB.x;
    // Real rhoA2 = rhoA * rhoA;
    // Real rhoB2 = rhoB * rhoB;

    Real Mass = paramsD.markerMass;
    Real MassOverRhoA2 = Mass * paramsD.invrho0 * paramsD.invrho0;//Mass/rhoA2;
    Real MassOverRhoB2 = MassOverRhoA2;//Mass/rhoB2;

    Real3 MA_gradW = gradW * MassOverRhoA2;
    Real3 MB_gradW = gradW * MassOverRhoB2;

    Real derivVx = //-Mass * (PA / (rhoA * rhoA) + PB / (rhoB * rhoB)) * gradW.x +
                    (txxA * MA_gradW.x + txyA * MA_gradW.y + txzA * MA_gradW.z) +
                    (txxB * MB_gradW.x + txyB * MB_gradW.y + txzB * MB_gradW.z) ;
    Real derivVy = //-Mass * (PA / (rhoA * rhoA) + PB / (rhoB * rhoB)) * gradW.y +
                    (txyA * MA_gradW.x + tyyA * MA_gradW.y + tyzA * MA_gradW.z) +
                    (txyB * MB_gradW.x + tyyB * MB_gradW.y + tyzB * MB_gradW.z) ;
    Real derivVz = //-Mass * (PA / (rhoA * rhoA) + PB / (rhoB * rhoB)) * gradW.z +
                    (txzA * MA_gradW.x + tyzA * MA_gradW.y + tzzA * MA_gradW.z) +
                    (txzB * MB_gradW.x + tyzB * MB_gradW.y + tzzB * MB_gradW.z) ;

    // TODO: visco-plastic model 
    // Real vel = length(velMasA);
    // if(vel > 0.3){
    //     Real rAB_Dot_GradWh = dot(dist3, gradW);
    //     Real rAB_Dot_GradWh_OverDist = rAB_Dot_GradWh / (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML);
    //     Real3 derivV = - paramsD.markerMass *(rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)) * gradW
    //                    + paramsD.markerMass * (8.0f * multViscosity) * paramsD.mu_fric_s 
    //                    * pow(rhoPresMuA.x + rhoPresMuB.x, Real(-2)) * rAB_Dot_GradWh_OverDist * (velMasA - velMasB);
    //     derivVx = derivV.x;
    //     derivVy = derivV.y;
    //     derivVz = derivV.z;
    // }

    //*** Artificial viscosity
    Real vAB_Dot_rAB = dot(velMasA - velMasB, dist3);
    // if (vAB_Dot_rAB < 0.0) {
    //     if ((rhoPresMuA.w < - 0.5)  && (rhoPresMuB.w < - 0.5)){ // only for fluid particles
            Real alpha = paramsD.Ar_vis_alpha;
            Real c_ab = paramsD.Cs;
            // Real rho = 0.5f * (rhoA + rhoB);
            Real nu = -alpha * paramsD.HSML * c_ab * paramsD.invrho0;
            Real derivM1 = -Mass * (nu * vAB_Dot_rAB * (invd * invd));//+ paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML
            derivVx += derivM1 * gradW.x;
            derivVy += derivM1 * gradW.y;
            derivVz += derivM1 * gradW.z;
    //     }
    // }

    // damping force
    /*if (1 == 0) {
        Real xi0 = paramsD.Vis_Dam;
        Real E0 = paramsD.E_young;
        Real h0 = paramsD.HSML;
        Real Cd = xi0 * sqrt(E0 / (rhoA * h0 * h0));
        derivVx -= Cd * velMasA.x;
        derivVy -= Cd * velMasA.y;
        derivVz -= Cd * velMasA.z;
    }*/

    // Real derivRho = Mass * dot(vel_XSPH_A - vel_XSPH_B, gradW);
    return mR4(derivVx, derivVy, derivVz, 0.0);
}

//--------------------------------------------------------------------------------------------------------------------------------
 __device__ inline Real3 GradientOperator(  float G_i[9],
                                            Real3 dist3,
                                            Real4 posRadA,
                                            Real4 posRadB,
                                            Real fA,
                                            Real fB,
                                            Real4 rhoPresMuA,
                                            Real4 rhoPresMuB) {
    Real3 gradW = GradWh(dist3, (posRadA.w + posRadB.w) * 0.5);
    Real3 gradW_new;
    gradW_new.x = G_i[0]*gradW.x + G_i[1]*gradW.y + G_i[2]*gradW.z;
    gradW_new.y = G_i[3]*gradW.x + G_i[4]*gradW.y + G_i[5]*gradW.z;
    gradW_new.z = G_i[6]*gradW.x + G_i[7]*gradW.y + G_i[8]*gradW.z;

    Real Vol = paramsD.markerMass/rhoPresMuB.x;
    Real fji = fB - fA;
    Real Gra_ij_x = fji*gradW_new.x * Vol;
    Real Gra_ij_y = fji*gradW_new.y * Vol;
    Real Gra_ij_z = fji*gradW_new.z * Vol;

    return mR3(Gra_ij_x, Gra_ij_y, Gra_ij_z);
}

//--------------------------------------------------------------------------------------------------------------------------------
 __device__ inline Real4 LaplacianOperator( float G_i[9],
                                            float L_i[9],
                                            Real3 dist3,
                                            Real4 posRadA,
                                            Real4 posRadB,
                                            Real fA,
                                            Real fB,
                                            Real4 rhoPresMuA,
                                            Real4 rhoPresMuB) {
    Real3 gradW = GradWh(dist3, (posRadA.w + posRadB.w) * 0.5);
    Real d = length(dist3);
    Real3 eij = dist3/d;

    Real Vol = paramsD.markerMass/rhoPresMuB.x;
    Real fij = fA - fB;

    Real ex_Gwx = eij.x*gradW.x;
    Real ex_Gwy = eij.x*gradW.y;
    Real ex_Gwz = eij.x*gradW.z;
    Real ey_Gwx = eij.y*gradW.x;
    Real ey_Gwy = eij.y*gradW.y;
    Real ey_Gwz = eij.y*gradW.z;
    Real ez_Gwx = eij.z*gradW.x;
    Real ez_Gwy = eij.z*gradW.y;
    Real ez_Gwz = eij.z*gradW.z;

    Real Part1 = L_i[0]*ex_Gwx + L_i[1]*ex_Gwy + L_i[2]*ex_Gwz
               + L_i[3]*ey_Gwx + L_i[4]*ey_Gwy + L_i[5]*ey_Gwz
               + L_i[6]*ez_Gwx + L_i[7]*ez_Gwy + L_i[8]*ez_Gwz;
    Real Part2 = fij/d * Vol;
    Real3 Part3 = mR3(-eij.x, -eij.y, -eij.z) * Vol;

    return mR4(2.0*Part1*Part2, Part3.x*(2.0*Part1), Part3.y*(2.0*Part1), Part3.z*(2.0*Part1));
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void EOS(Real4* sortedRhoPreMu, uint numAllMarkers, volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numAllMarkers)
        return;
    sortedRhoPreMu[index].y = Eos(sortedRhoPreMu[index].x, sortedRhoPreMu[index].w);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Navier_Stokes(Real4* sortedDerivVelRho,
                              Real3* shift_r,
                              Real4* sortedPosRad,
                              Real3* sortedVelMas,
                              Real4* sortedRhoPreMu,
                              Real3* velMas_ModifiedBCE,
                              Real4* rhoPreMu_ModifiedBCE,
                              Real3* sortedTauXxYyZz,
                              Real3* sortedTauXyXzYz,
                              uint* gridMarkerIndex,
                              uint* cellStart,
                              uint* cellEnd,
                              const size_t numAllMarkers,
                              Real MaxVel,
                              volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numAllMarkers)
        return;
    if (sortedRhoPreMu[index].w > -0.5 && sortedRhoPreMu[index].w < 0.5){
        sortedDerivVelRho[index] = mR4(0.0);
        return;
    }
        
    Real3 posRadA = mR3(sortedPosRad[index]);
    Real3 velMasA = sortedVelMas[index];
    Real4 rhoPresMuA = sortedRhoPreMu[index];
    Real4 derivVelRho = mR4(0.0);
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    Real G_i[9] = {0.0};
    Real A_i[27] = {0.0};
    Real L_i[9] = {0.0};
    calc_G_Matrix(sortedPosRad,sortedVelMas,sortedRhoPreMu,G_i,cellStart,cellEnd,numAllMarkers);
    if(!paramsD.elastic_SPH){
        calc_A_Matrix(sortedPosRad,sortedVelMas,sortedRhoPreMu,A_i,G_i,cellStart,cellEnd,numAllMarkers);
        calc_L_Matrix(sortedPosRad,sortedVelMas,sortedRhoPreMu,A_i,L_i,G_i,cellStart,cellEnd,numAllMarkers);
    }
    float Gi[9] = {1.0,0.0,0.0, 0.0,1.0,0.0, 0.0,0.0,1.0};
    float Li[9] = {1.0,0.0,0.0, 0.0,1.0,0.0, 0.0,0.0,1.0};
    Gi[0] = G_i[0];
    Gi[1] = G_i[1];
    Gi[2] = G_i[2];
    Gi[3] = G_i[3];
    Gi[4] = G_i[4];
    Gi[5] = G_i[5];
    Gi[6] = G_i[6];
    Gi[7] = G_i[7];
    Gi[8] = G_i[8];
    Li[0] = L_i[0];
    Li[1] = L_i[1];
    Li[2] = L_i[2];
    Li[3] = L_i[3];
    Li[4] = L_i[4];
    Li[5] = L_i[5];
    Li[6] = L_i[6];
    Li[7] = L_i[7];
    Li[8] = L_i[8];
    // Real3 posGra = mR3(0.0);
    // Real4 posLap = mR4(0.0);
    Real3 preGra = mR3(0.0);
    Real3 velxGra = mR3(0.0);
    Real3 velyGra = mR3(0.0);
    Real3 velzGra = mR3(0.0);
    Real4 velxLap = mR4(0.0);
    Real4 velyLap = mR4(0.0);
    Real4 velzLap = mR4(0.0);

    Real radii = paramsD.MULT_INITSPACE * paramsD.HSML*1.241;//1.129;//1.241
    Real invRadii = 1.0/radii;
    Real3 v_ab = (velMasA + velMasA)*0.5;
    Real v_ab_m = length(v_ab);
    Real bsvdT = paramsD.beta_shifting * v_ab_m * paramsD.dT ;

    // Real3 numeratorXxYyZz = mR3(0.0);
    // Real3 numeratorXyXzYz = mR3(0.0);
    // Real denominator = 1e-9;

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    Real3 inner_sum = mR3(0.0);
    // Real mi_bar = 0.0, r0 = 0.0;
    Real sum_w_i = W3h_GPU(0.0, sortedPosRad[index].w) * cube(sortedPosRad[index].w * paramsD.MULT_INITSPACE);
    int N_ = 1;
    int N_s = 0;
    for (int x = -1; x <= 1; x++) {
        for (int y = -1; y <= 1; y++) {
            for (int z = -1; z <= 1; z++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    if (j != index) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        // Real3 dist3Alpha = posRadA - posRadB;
                        Real3 dist3 = Distance(posRadA, posRadB);  // change from B-A to A-B
                        Real d = length(dist3);
                        if (d > SuppRadii)
                            continue;
                        Real4 rhoPresMuB = sortedRhoPreMu[j];
                        if (rhoPresMuA.w > -.1 && rhoPresMuB.w > -.1) {  // no rigid-rigid force
                            continue;
                        }
                        Real invd = 1.0 / d;
                        // modifyPressure(rhoPresMuB, dist3Alpha);
                        // if (!(isfinite(rhoPresMuB.x) && isfinite(rhoPresMuB.y) && isfinite(rhoPresMuB.z))) {
                        //     printf("Error! particle rhoPresMuB is NAN: thrown from modifyPressure !\n");
                        // }
                        Real3 velMasB = sortedVelMas[j];
                        if (rhoPresMuB.w > -1.0) {
                            int bceIndexB = gridMarkerIndex[j] - (numObjectsD.numFluidMarkers);
                            // if (!(bceIndexB >= 0 &&
                            //       bceIndexB < numObjectsD.numBoundaryMarkers + numObjectsD.numRigid_SphMarkers)) {
                            //     printf("Error! bceIndex out of bound, collideCell !\n");
                            // }
                            rhoPresMuB = rhoPreMu_ModifiedBCE[bceIndexB];
                            velMasB = velMas_ModifiedBCE[bceIndexB];
                        }
                        Real multViscosit = 1;
                        // if (!(isfinite(rhoPresMuB.x) && isfinite(rhoPresMuB.y) && isfinite(rhoPresMuB.z))) {
                        //     printf("Error! particle rhoPresMuB is NAN: thrown from collideCell ! type=%f\n",
                        //            rhoPresMuB.w);
                        // }
                        // change from "-=" to "+="
                        if(paramsD.elastic_SPH){
                            Real3 gradW = GradWh(dist3, paramsD.HSML);
                            Real3 gradW_new;
                            gradW_new.x = G_i[0]*gradW.x + G_i[1]*gradW.y + G_i[2]*gradW.z;
                            gradW_new.y = G_i[3]*gradW.x + G_i[4]*gradW.y + G_i[5]*gradW.z;
                            gradW_new.z = G_i[6]*gradW.x + G_i[7]*gradW.y + G_i[8]*gradW.z;
                            gradW = gradW_new;
                            derivVelRho += DifVelocityRho_ElasticSPH(gradW, dist3, d, invd, 
                                                      sortedPosRad[index], sortedPosRad[j], velMasA, velMasA,
                                                      velMasB, velMasB, rhoPresMuA, rhoPresMuB, multViscosit,
                                                      sortedTauXxYyZz[index], sortedTauXyXzYz[index],
                                                      sortedTauXxYyZz[j], sortedTauXyXzYz[j]);
                        }
                        else{
                            derivVelRho += DifVelocityRho(Gi, dist3, d, sortedPosRad[index], sortedPosRad[j], velMasA, velMasA,
                                                          velMasB, velMasB, rhoPresMuA, rhoPresMuB, multViscosit);
                            preGra  += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], 
                                                        -rhoPresMuA.y, rhoPresMuB.y, rhoPresMuA, rhoPresMuB);
                            velxGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], 
                                                        velMasA.x, velMasB.x, rhoPresMuA, rhoPresMuB);
                            velyGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], 
                                                        velMasA.y, velMasB.y, rhoPresMuA, rhoPresMuB);
                            velzGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], 
                                                        velMasA.z, velMasB.z, rhoPresMuA, rhoPresMuB);
                            velxLap += LaplacianOperator(Gi, Li, dist3, sortedPosRad[index], sortedPosRad[j], 
                                                         velMasA.x, velMasB.x, rhoPresMuA, rhoPresMuB);
                            velyLap += LaplacianOperator(Gi, Li, dist3, sortedPosRad[index], sortedPosRad[j], 
                                                         velMasA.y, velMasB.y, rhoPresMuA, rhoPresMuB);
                            velzLap += LaplacianOperator(Gi, Li, dist3, sortedPosRad[index], sortedPosRad[j], 
                                                         velMasA.z, velMasB.z, rhoPresMuA, rhoPresMuB);
                        }

                        if (d > paramsD.HSML*1.0e-9 && sum_w_i < paramsD.C_Wi) {
                            // Real m_j = cube(sortedPosRad[j].w * paramsD.MULT_INITSPACE) * paramsD.rho0;
                            // mi_bar += m_j;
                            // r0 += d;
                            // inner_sum += m_j * dist3 / (d * d * d); 
                            sum_w_i = sum_w_i + W3h_GPU(d, sortedPosRad[index].w) * cube(sortedPosRad[index].w * paramsD.MULT_INITSPACE);
                            N_ = N_ + 1;
                        }

                        // find particles that have contact with this particle
                        if(N_s < 12 && d < 2.0*radii){
                            Real Pen = (radii - d) * invRadii;
                            Real3 r_0 = bsvdT * invd * dist3 ;
                            Real3 r_s = r_0 * Pen;
                            if (d < 1.0*radii) {
                                inner_sum += 3.0*r_s;
                                N_s = N_s + 1;
                            }
                            else if (d < 1.1*radii) {
                                inner_sum += 1.0*r_s;
                                N_s = N_s + 1;
                            }
                            else {
                                inner_sum += 0.1 * 1.0 * (-r_0);
                                N_s = N_s + 1;
                            }
                        }
                        // posGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], 
                        //                            square(posRadA.x), square(posRadB.x), rhoPresMuA, rhoPresMuB);
                        // posLap += LaplacianOperator(Gi, Li, dist3, sortedPosRad[index], sortedPosRad[j], 
                        //                             square(posRadA.x), square(posRadB.x), rhoPresMuA, rhoPresMuB);
                    }
                }
            }
        }
    }

    if(paramsD.elastic_SPH){
        if(sum_w_i < paramsD.C_Wi){
            derivVelRho.w = -1.0;
        }
        else{
            derivVelRho.w =  1.0;
        }
    }

    if(!paramsD.elastic_SPH){
        Real nu = paramsD.mu0/paramsD.rho0;
        Real dvxdt = -preGra.x/rhoPresMuA.x + (velxLap.x + velxGra.x*velxLap.y + velxGra.y*velxLap.z + velxGra.z*velxLap.w) * nu;
        Real dvydt = -preGra.y/rhoPresMuA.x + (velyLap.x + velyGra.x*velyLap.y + velyGra.y*velyLap.z + velyGra.z*velyLap.w) * nu;
        Real dvzdt = -preGra.z/rhoPresMuA.x + (velzLap.x + velzGra.x*velzLap.y + velzGra.y*velzLap.z + velzGra.z*velzLap.w) * nu;
        Real drhodt = -paramsD.rho0*(velxGra.x + velyGra.y + velzGra.z);

        Real Det_G = (Gi[0] * Gi[4] * Gi[8] - Gi[0] * Gi[5] * Gi[7] - Gi[1] * Gi[3] * Gi[8] +
                      Gi[1] * Gi[5] * Gi[6] + Gi[2] * Gi[3] * Gi[7] - Gi[2] * Gi[4] * Gi[6]);
        Real Det_L = (Li[0] * Li[4] * Li[8] - Li[0] * Li[5] * Li[7] - Li[1] * Li[3] * Li[8] +
                      Li[1] * Li[5] * Li[6] + Li[2] * Li[3] * Li[7] - Li[2] * Li[4] * Li[6]);
        if(rhoPresMuA.w == -1){
            if( Det_G > 0.9 && Det_G < 1.1 && Det_L > 0.9 && Det_L < 1.1 && sum_w_i > 0.9){
                // printf("Det_G, Det_L %f %f %f %f %f %d\n", Det_G, Det_L, posRadA.x, posRadA.y, posRadA.z, N_);
                derivVelRho = mR4(dvxdt, dvydt, dvzdt, drhodt);
            }
            // Real dvdt =length(mR3(derivVelRho));
            // Real coeff = 1000.0/dvdt;
            // if(dvdt > 1000.0){
                // derivVelRho = mR4(dvxdt * coeff, dvydt * coeff, dvzdt * coeff, 0.0);
                // derivVelRho = mR4(0.0);
            // }
        }
    }

    if (!(isfinite(derivVelRho.x) && isfinite(derivVelRho.y) && isfinite(derivVelRho.z))) {
        printf("Error! particle derivVel is NAN: thrown from ChFsiForceExplicitSPH.cu, collideD !\n");
        *isErrorD = true;
    }
    if (!(isfinite(derivVelRho.w))) {
        printf("Error! particle derivRho is NAN: thrown from ChFsiForceExplicitSPH.cu, collideD !\n");
        *isErrorD = true;
    }

    // add gravity and other body force to fluid markers
    if (rhoPresMuA.w > -1.5 && rhoPresMuA.w < -0.5){
        Real3 totalFluidBodyForce3 = paramsD.bodyForce3 + paramsD.gravity;
        derivVelRho += mR4(totalFluidBodyForce3);    
    }

    sortedDerivVelRho[index] = derivVelRho;

    // r0 /= N_;
    // mi_bar /= N_;
    // if (sum_w_i > 0.95 && sortedRhoPreMu[index].w < -0.5 )
        // shift_r[index] = paramsD.beta_shifting * r0 * r0 * MaxVel * paramsD.dT * inner_sum / (mi_bar+1e-9);
    // else
    //     shift_r[index] = mR3(0.0);

    Real det_r_max = length(0.05*velMasA*paramsD.dT);
    Real det_r_A = length(inner_sum);
    if(det_r_A < det_r_max){
        shift_r[index] = inner_sum;
    }
    else{
        shift_r[index] = inner_sum * det_r_max/(det_r_A + 1e-9);
    }
    // shift_r[index] = mR3(0.0);
    
    // shift_r[index].y = 0.0;
    // if (sum_w_i < 0.95 && sortedRhoPreMu[index].w < -0.5)
    //     printf("Finished in %f %f %f %f %f\n", sum_w_i, sortedPosRad[index].x, sortedPosRad[index].y, sortedPosRad[index].z, sortedRhoPreMu[index].w);
}

__global__ void NS_SSR( Real4* sortedDerivVelRho,
                        Real3* sortedDerivTauXxYyZz,
                        Real3* sortedDerivTauXyXzYz,
                        Real3* shift_r,
                        Real4* sortedPosRad,
                        Real3* sortedVelMas,
                        Real4* sortedRhoPreMu,
                        Real3* velMas_ModifiedBCE,
                        Real4* rhoPreMu_ModifiedBCE,
                        Real3* sortedTauXxYyZz,
                        Real3* sortedTauXyXzYz,
                        uint* gridMarkerIndex,
                        uint* cellStart,
                        uint* cellEnd,
                        const size_t numAllMarkers,
                        volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numAllMarkers)
        return;
    if (sortedRhoPreMu[index].w > -0.5 && sortedRhoPreMu[index].w < 0.5){
        sortedDerivVelRho[index] = mR4(0.0);
        sortedDerivTauXxYyZz[index] = mR3(0.0);
        sortedDerivTauXyXzYz[index] = mR3(0.0);
        return;
    }
        
    Real3 posRadA = mR3(sortedPosRad[index]);
    Real3 velMasA = sortedVelMas[index];
    Real4 rhoPresMuA = sortedRhoPreMu[index];
    Real hA = sortedPosRad[index].w;
    Real4 derivVelRho = mR4(0.0);
    Real3 deltaV = mR3(0);
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    uint j_list[150];
    uint j_num = 0;

    // Get address in grid
    int3 gridPos = calcGridPos(posRadA);
    for (int x = -1; x <= 1; x++) {
        for (int y = -1; y <= 1; y++) {
            for (int z = -1; z <= 1; z++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    if (j != index) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB); 
                        Real d = length(dist3);
                        if (d < SuppRadii){
                            j_list[j_num] = j;
                            j_num++;
                        }
                    }
                }
            }
        }
    }

    Real tauxx = sortedTauXxYyZz[index].x;
    Real tauyy = sortedTauXxYyZz[index].y;
    Real tauzz = sortedTauXxYyZz[index].z;
    Real tauxy = sortedTauXyXzYz[index].x;
    Real tauxz = sortedTauXyXzYz[index].y;
    Real tauyz = sortedTauXyXzYz[index].z;
    Real tauzx = tauxz;
    Real tauzy = tauyz;
    Real tauyx = tauxy;
    Real dTauxx = 0.0;
    Real dTauyy = 0.0;
    Real dTauzz = 0.0;
    Real dTauxy = 0.0;
    Real dTauxz = 0.0;
    Real dTauyz = 0.0;

    Real G_i[9] = {1.0,0.0,0.0, 0.0,1.0,0.0, 0.0,0.0,1.0};
    {
        Real mGi[9] = {0.0};
        for(uint n = 0; n < j_num; n++){
            uint j =  j_list[n];
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real d = length(rij);
            if (d > SuppRadii || sortedRhoPreMu[j].w <= -2)
                continue;
            Real3 grad_i_wij = GradWh(rij, hA);
            Real3 grw_vj = grad_i_wij * paramsD.volume0;
            mGi[0] -= rij.x * grw_vj.x;
            mGi[1] -= rij.x * grw_vj.y;
            mGi[2] -= rij.x * grw_vj.z;
            mGi[3] -= rij.y * grw_vj.x;
            mGi[4] -= rij.y * grw_vj.y;
            mGi[5] -= rij.y * grw_vj.z;
            mGi[6] -= rij.z * grw_vj.x;
            mGi[7] -= rij.z * grw_vj.y;
            mGi[8] -= rij.z * grw_vj.z;
        }
        Real Det = (mGi[0] * mGi[4] * mGi[8] - 
                    mGi[0] * mGi[5] * mGi[7] - 
                    mGi[1] * mGi[3] * mGi[8] +
                    mGi[1] * mGi[5] * mGi[6] + 
                    mGi[2] * mGi[3] * mGi[7] - 
                    mGi[2] * mGi[4] * mGi[6]);
        if (abs(Det) > 0.01) {
            Real OneOverDet = 1.0 / Det;
            G_i[0] =  (mGi[4] * mGi[8] - mGi[5] * mGi[7]) * OneOverDet;
            G_i[1] = -(mGi[1] * mGi[8] - mGi[2] * mGi[7]) * OneOverDet;
            G_i[2] =  (mGi[1] * mGi[5] - mGi[2] * mGi[4]) * OneOverDet;
            G_i[3] = -(mGi[3] * mGi[8] - mGi[5] * mGi[6]) * OneOverDet;
            G_i[4] =  (mGi[0] * mGi[8] - mGi[2] * mGi[6]) * OneOverDet;
            G_i[5] = -(mGi[0] * mGi[5] - mGi[2] * mGi[3]) * OneOverDet;
            G_i[6] =  (mGi[3] * mGi[7] - mGi[4] * mGi[6]) * OneOverDet;
            G_i[7] = -(mGi[0] * mGi[7] - mGi[1] * mGi[6]) * OneOverDet;
            G_i[8] =  (mGi[0] * mGi[4] - mGi[1] * mGi[3]) * OneOverDet;
        } 
    }

    Real radii = paramsD.MULT_INITSPACE * paramsD.HSML*1.241;//1.129;//1.241
    Real invRadii = 1.0/radii;
    Real3 v_ab = (velMasA + velMasA)*0.5;
    Real v_ab_m = length(v_ab);
    Real bsvdT = paramsD.beta_shifting * v_ab_m * paramsD.dT ;

    Real3 inner_sum = mR3(0.0);
    Real sum_w_i = W3h_GPU(0.0, sortedPosRad[index].w) * cube(sortedPosRad[index].w * paramsD.MULT_INITSPACE);
    int N_ = 1;
    int N_s = 0;
    for(uint n = 0; n < j_num; n++){
        uint j =  j_list[n];
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 dist3 = Distance(posRadA, posRadB); 
        Real d = length(dist3);
        if (d > SuppRadii)
            continue;
        Real4 rhoPresMuB = sortedRhoPreMu[j];
        if (rhoPresMuA.w > -.1 && rhoPresMuB.w > -.1) {  // no rigid-rigid force
            continue;
        }
        Real invd = 1.0 / d;
        Real3 velMasB = sortedVelMas[j];
        if (rhoPresMuB.w > -1.0) {
            int bceIndexB = gridMarkerIndex[j] - (numObjectsD.numFluidMarkers);
            rhoPresMuB = rhoPreMu_ModifiedBCE[bceIndexB];
            velMasB = velMas_ModifiedBCE[bceIndexB];
        }
        Real multViscosit = 1;
        // For granular material dynamics
        // Real rhoB = rhoPresMuB.x;
        Real hB = sortedPosRad[j].w;
        // Real mB = paramsD.markerMass;
        Real3 gradW = GradWh(dist3, (hA + hB) * 0.5);
        Real3 gradW_new;
        gradW_new.x = G_i[0]*gradW.x + G_i[1]*gradW.y + G_i[2]*gradW.z;
        gradW_new.y = G_i[3]*gradW.x + G_i[4]*gradW.y + G_i[5]*gradW.z;
        gradW_new.z = G_i[6]*gradW.x + G_i[7]*gradW.y + G_i[8]*gradW.z;
        gradW = gradW_new;
        derivVelRho += DifVelocityRho_ElasticSPH(gradW, dist3, d, invd, 
                                    sortedPosRad[index], sortedPosRad[j], velMasA, velMasA,
                                    velMasB, velMasB, rhoPresMuA, rhoPresMuB, multViscosit,
                                    sortedTauXxYyZz[index], sortedTauXyXzYz[index],
                                    sortedTauXxYyZz[j], sortedTauXyXzYz[j]);
        if(sortedRhoPreMu[index].w < -0.5){
            // start to calculate the stress rate
            Real Gm = paramsD.G_shear;  // shear modulus of the material
            Real half_mB_over_rhoB = 0.5 * paramsD.volume0; //(mB / rhoB);
            Real3 velMasB_new = velMasB;
            if (rhoPresMuB.w > -1.0) 
                velMasB_new = 2.0*velMasB - velMasA; // noslip BC
            Real3 vAB = velMasA - velMasB_new;
            Real3 vAB_h = vAB * half_mB_over_rhoB;
            // entries of strain rate tensor
            Real exx = -2.0 * vAB_h.x * gradW.x;
            Real eyy = -2.0 * vAB_h.y * gradW.y;
            Real ezz = -2.0 * vAB_h.z * gradW.z;
            Real exy = -vAB_h.x * gradW.y - vAB_h.y * gradW.x;
            Real exz = -vAB_h.x * gradW.z - vAB_h.z * gradW.x;
            Real eyz = -vAB_h.y * gradW.z - vAB_h.z * gradW.y;
            // entries of rotation rate (spin) tensor
            // Real wxx = 0.0;
            // Real wyy = 0.0;
            // Real wzz = 0.0;
            Real wxy = -vAB_h.x * gradW.y + vAB_h.y * gradW.x;
            Real wxz = -vAB_h.x * gradW.z + vAB_h.z * gradW.x;
            Real wyz = -vAB_h.y * gradW.z + vAB_h.z * gradW.y;
            Real wyx = -wxy;
            // Real wzx = -wxz;
            Real wzy = -wyz;

            Real edia = 1.0 / 3.0 * (exx + eyy + ezz);
            Real twoGm = 2.0 * Gm;
            Real K_edia = paramsD.K_bulk*1.0*edia;
            dTauxx += twoGm * (exx - edia) + 2.0 * (tauxy * wxy + tauxz * wxz) + K_edia;
            dTauyy += twoGm * (eyy - edia) - 2.0 * (tauyx * wxy - tauyz * wyz) + K_edia;
            dTauzz += twoGm * (ezz - edia) - 2.0 * (tauzx * wxz + tauzy * wyz) + K_edia;
            dTauxy += twoGm * exy - (tauxx * wxy + tauxz * wzy) + (wxy * tauyy + wxz * tauzy);
            dTauxz += twoGm * exz - (tauxx * wxz + tauxy * wyz) + (wxy * tauyz + wxz * tauzz);
            dTauyz += twoGm * eyz - (tauyx * wxz + tauyy * wyz) + (wyx * tauxz + wyz * tauzz);
        }
        // Do integration for the kernel function
        if (d > paramsD.HSML*1.0e-9) {
            Real Wab = W3h_GPU(d, sortedPosRad[index].w);
            sum_w_i = sum_w_i +  Wab * cube(sortedPosRad[index].w * paramsD.MULT_INITSPACE);
            // XSPH
            if (rhoPresMuB.w > -1.5 && rhoPresMuB.w < -0.5){
                deltaV += paramsD.volume0 * (velMasB - velMasA) * Wab;
            }
            N_ = N_ + 1;
        }
        // Find particles that have contact with this particle
        if(N_s < 12 && d < 2.0*radii){
            Real Pen = (radii - d) * invRadii;
            Real3 r_0 = bsvdT * invd * dist3 ;
            Real3 r_s = r_0 * Pen;
            if (d < 1.0*radii) {
                inner_sum += 3.0*r_s;
                N_s = N_s + 1;
            }
            else if (d < 1.1*radii) {
                inner_sum += 1.0*r_s;
                N_s = N_s + 1;
            }
            else {
                inner_sum += 0.1 * 1.0 * (-r_0);
                N_s = N_s + 1;
            }
        }
    }

    // Check particles who have not enough neighbor particles (only for granular now)
    if(sum_w_i < paramsD.C_Wi){
        derivVelRho.w = -1.0;
    }
    else{
        derivVelRho.w =  1.0;
    }

    // Calculate the shifting vector
    Real det_r_max = length(0.05*velMasA*paramsD.dT);
    Real det_r_A = length(inner_sum);
    if(det_r_A < det_r_max){
        shift_r[index] = inner_sum;
    }
    else{
        shift_r[index] = inner_sum * det_r_max/(det_r_A + 1e-9);
    }
    shift_r[index] += deltaV * paramsD.dT;
    shift_r[index] = shift_r[index] * (1.0 / paramsD.dT);

    // add gravity other body force to fluid markers
    if (rhoPresMuA.w > -1.5 && rhoPresMuA.w < -0.5){
        Real3 totalFluidBodyForce3 = paramsD.bodyForce3 + paramsD.gravity;
        derivVelRho += mR4(totalFluidBodyForce3);    
    }

    sortedDerivVelRho[index] = derivVelRho;
    sortedDerivTauXxYyZz[index] = mR3(dTauxx, dTauyy, dTauzz);
    sortedDerivTauXyXzYz[index] = mR3(dTauxy, dTauxz, dTauyz);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void CalcVel_XSPH_D(Real3* vel_XSPH_Sorted_D,  // output: new velocity
                               Real4* sortedPosRad_old,   // input: sorted positions
                               Real4* sortedPosRad,       // input: sorted positions
                               Real3* sortedVelMas,       // input: sorted velocities
                               Real4* sortedRhoPreMu,
                               Real3* shift_r,
                               uint* gridMarkerIndex,  // input: sorted particle indices
                               uint* cellStart,
                               uint* cellEnd,
                               const size_t numAllMarkers,
                               volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numAllMarkers)
        return;

    Real4 rhoPreMuA = sortedRhoPreMu[index];
    Real3 velMasA = sortedVelMas[index];
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;

    Real3 posRadA = mR3(sortedPosRad_old[index]);
    Real3 deltaV = mR3(0);

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    Real3 inner_sum = mR3(0.0);
    //    Real mi_bar = 0.0, r0 = 0.0;
    Real3 dV = mR3(0.0f);
    // examine neighbouring cells
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    if (j != index) {  // check not colliding with self
                        Real3 posRadB = mR3(sortedPosRad_old[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real d = length(dist3);
                        if (d > SuppRadii)
                            continue;
                        Real4 rhoPresMuB = sortedRhoPreMu[j];

                        if (rhoPresMuB.w != -1.0)
                            continue;
                        Real3 velMasB = sortedVelMas[j];
                        Real rho_bar = 0.5 * (rhoPreMuA.x + rhoPresMuB.x);
                        deltaV += paramsD.markerMass * (velMasB - velMasA) *
                                  W3h_GPU(d, (sortedPosRad_old[index].w + sortedPosRad_old[j].w) * 0.5) / rho_bar;
                    }
                }
            }
        }
    }

    vel_XSPH_Sorted_D[index] = deltaV + shift_r[index]*(1.0/paramsD.dT);

    // sortedPosRad[index] += mR4(shift_r[index], 0.0); //

    if (!(isfinite(vel_XSPH_Sorted_D[index].x) && isfinite(vel_XSPH_Sorted_D[index].y) &&
          isfinite(vel_XSPH_Sorted_D[index].z))) {
        printf("Error! particle vXSPH is NAN: thrown from ChFsiForceExplicitSPH.cu, newVel_XSPH_D !\n");
        *isErrorD = true;
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
ChFsiForceExplicitSPH::ChFsiForceExplicitSPH(std::shared_ptr<ChBce> otherBceWorker,
                                             std::shared_ptr<SphMarkerDataD> otherSortedSphMarkersD,
                                             std::shared_ptr<ProximityDataD> otherMarkersProximityD,
                                             std::shared_ptr<FsiGeneralData> otherFsiGeneralData,
                                             std::shared_ptr<SimParams> otherParamsH,
                                             std::shared_ptr<NumberOfObjects> otherNumObjects)
    : ChFsiForce(otherBceWorker,
                 otherSortedSphMarkersD,
                 otherMarkersProximityD,
                 otherFsiGeneralData,
                 otherParamsH,
                 otherNumObjects) {
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
    density_initialization = 0;
}

//--------------------------------------------------------------------------------------------------------------------------------
ChFsiForceExplicitSPH::~ChFsiForceExplicitSPH() {}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::Finalize() {
    ChFsiForce::Finalize();
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH.get(), sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH.get(), sizeof(NumberOfObjects));
    hipMemcpyFromSymbol(paramsH.get(), paramsD, sizeof(SimParams));
    hipDeviceSynchronize();
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::ForceSPH(std::shared_ptr<SphMarkerDataD> otherSphMarkersD,
                                     std::shared_ptr<FsiBodiesDataD> otherFsiBodiesD,
                                     std::shared_ptr<FsiMeshDataD> fsiMeshD) {
    sphMarkersD = otherSphMarkersD;
    fsiCollisionSystem->ArrangeData(sphMarkersD);
    bceWorker->ModifyBceVelocity(sphMarkersD, otherFsiBodiesD);
    CollideWrapper();
    CalculateXSPH_velocity();
    // AddGravityToFluid();
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::CollideWrapper() {
    bool *isErrorH, *isErrorD;
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    //------------------------------------------------------------------------
    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize((int)numObjectsH->numAllMarkers, 128, numBlocks, numThreads);
    /* Execute the kernel */
    // thrust::device_vector<Real> _sumWij_rhoi(numObjectsH->numAllMarkers);
    thrust::device_vector<Real4> sortedDerivVelRho(numObjectsH->numAllMarkers);
    thrust::device_vector<Real3> sortedDerivTauXxYyZz(numObjectsH->numAllMarkers);
    thrust::device_vector<Real3> sortedDerivTauXyXzYz(numObjectsH->numAllMarkers);
    shift_r.resize(numObjectsH->numAllMarkers);
    // thrust::fill(_sumWij_rhoi.begin(), _sumWij_rhoi.end(), 0.);
    // thrust::fill(shift_r.begin(), shift_r.end(), mR3(0.0));
    // thrust::fill(sortedDerivVelRho.begin(), sortedDerivVelRho.end(), mR4(0.0));
    // thrust::fill(sortedDerivTauXxYyZz.begin(), sortedDerivTauXxYyZz.end(), mR3(0.0));
    // thrust::fill(sortedDerivTauXyXzYz.begin(), sortedDerivTauXyXzYz.end(), mR3(0.0));

    // thrust::device_vector<Real4> rhoPresMuD_old = sortedSphMarkersD->rhoPresMuD;

    if (density_initialization == 0){
        thrust::device_vector<Real> _sumWij_rhoi(numObjectsH->numAllMarkers);
        thrust::device_vector<Real4> rhoPresMuD_old = sortedSphMarkersD->rhoPresMuD;
        printf("Re-initializing density after %d steps.\n", paramsH->densityReinit);
        calcRho_kernel<<<numBlocks, numThreads>>>(
            mR4CAST(sortedSphMarkersD->posRadD), mR4CAST(sortedSphMarkersD->rhoPresMuD), mR4CAST(rhoPresMuD_old),
            R1CAST(_sumWij_rhoi), U1CAST(markersProximityD->cellStartD), U1CAST(markersProximityD->cellEndD),
            numObjectsH->numAllMarkers, density_initialization, isErrorD);
            ChUtilsDevice::Sync_CheckError(isErrorH, isErrorD, "calcRho_kernel");
    }
    
    if(paramsH->elastic_SPH){
        // execute the kernel Navier_Stokes and Shear_Stress_Rate in one kernel
        *isErrorH = false;
        hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
        // execute the kernel
        NS_SSR<<<numBlocks, numThreads>>>(
            mR4CAST(sortedDerivVelRho),mR3CAST(sortedDerivTauXxYyZz), mR3CAST(sortedDerivTauXyXzYz),
            mR3CAST(shift_r), mR4CAST(sortedSphMarkersD->posRadD),
            mR3CAST(sortedSphMarkersD->velMasD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
            mR3CAST(bceWorker->velMas_ModifiedBCE), mR4CAST(bceWorker->rhoPreMu_ModifiedBCE),
            mR3CAST(sortedSphMarkersD->tauXxYyZzD), mR3CAST(sortedSphMarkersD->tauXyXzYzD),
            U1CAST(markersProximityD->gridMarkerIndexD), U1CAST(markersProximityD->cellStartD),
            U1CAST(markersProximityD->cellEndD), numObjectsH->numAllMarkers, isErrorD);
        ChUtilsDevice::Sync_CheckError(isErrorH, isErrorD, "Navier_Stokes and Shear_Stress_Rate");
    }
    else{
        // EOS<<<numBlocks, numThreads>>>(mR4CAST(sortedSphMarkersD->rhoPresMuD),
        //    numObjectsH->numAllMarkers, isErrorD);
        // ChUtilsDevice::Sync_CheckError(isErrorH, isErrorD, "EOS");
        thrust::device_vector<Real3>::iterator iter =
            thrust::max_element(sortedSphMarkersD->velMasD.begin(), 
            sortedSphMarkersD->velMasD.end(), compare_Real3_mag());
        ////unsigned int position = iter - sortedSphMarkersD->velMasD.begin();
        Real MaxVel = length(*iter);
        *isErrorH = false;
        hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
        // execute the kernel
        Navier_Stokes<<<numBlocks, numThreads>>>(
            mR4CAST(sortedDerivVelRho), mR3CAST(shift_r), mR4CAST(sortedSphMarkersD->posRadD),
            mR3CAST(sortedSphMarkersD->velMasD), mR4CAST(sortedSphMarkersD->rhoPresMuD),
            mR3CAST(bceWorker->velMas_ModifiedBCE), mR4CAST(bceWorker->rhoPreMu_ModifiedBCE),
            mR3CAST(sortedSphMarkersD->tauXxYyZzD), mR3CAST(sortedSphMarkersD->tauXyXzYzD),  //
            U1CAST(markersProximityD->gridMarkerIndexD), U1CAST(markersProximityD->cellStartD),
            U1CAST(markersProximityD->cellEndD), numObjectsH->numAllMarkers, MaxVel, isErrorD);
        ChUtilsDevice::Sync_CheckError(isErrorH, isErrorD, "Navier_Stokes");
    }

    CopySortedToOriginal_Invasive_R4(fsiGeneralData->derivVelRhoD_old, sortedDerivVelRho,
                                        markersProximityD->gridMarkerIndexD);
    if(paramsH->elastic_SPH){
        CopySortedToOriginal_Invasive_R3(fsiGeneralData->derivTauXxYyZzD, sortedDerivTauXxYyZz,
                                            markersProximityD->gridMarkerIndexD);
        CopySortedToOriginal_Invasive_R3(fsiGeneralData->derivTauXyXzYzD, sortedDerivTauXyXzYz,
                                            markersProximityD->gridMarkerIndexD);
    }
    sortedDerivVelRho.clear();
    sortedDerivTauXxYyZz.clear(); 
    sortedDerivTauXyXzYz.clear(); 
    hipFree(isErrorD);
    free(isErrorH);
    density_initialization++;
    if (density_initialization >= paramsH->densityReinit)
        density_initialization = 0;
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::CalculateXSPH_velocity() {
    /* Calculate vel_XSPH */
    if (vel_XSPH_Sorted_D.size() != numObjectsH->numAllMarkers) {
        printf("vel_XSPH_Sorted_D.size() %zd numObjectsH->numAllMarkers %zd \n", vel_XSPH_Sorted_D.size(),
               numObjectsH->numAllMarkers);
        throw std::runtime_error(
            "Error! size error vel_XSPH_Sorted_D Thrown from "
            "CalculateXSPH_velocity!\n");
    }

    bool *isErrorH, *isErrorD;
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);

    //------------------------------------------------------------------------
    if(paramsH->elastic_SPH){
        // The XSPH vector already included in the shifting vector
        CopySortedToOriginal_Invasive_R3(fsiGeneralData->vel_XSPH_D, shift_r, markersProximityD->gridMarkerIndexD);
    }
    else{
        /* thread per particle */
        uint numThreads, numBlocks;
        computeGridSize((uint)numObjectsH->numAllMarkers, 128, numBlocks, numThreads);
        thrust::device_vector<Real4> sortedPosRad_old = sortedSphMarkersD->posRadD;
        thrust::fill(vel_XSPH_Sorted_D.begin(), vel_XSPH_Sorted_D.end(), mR3(0.0));

        /* Execute the kernel */
        CalcVel_XSPH_D<<<numBlocks, numThreads>>>(
            mR3CAST(vel_XSPH_Sorted_D), mR4CAST(sortedPosRad_old), mR4CAST(sortedSphMarkersD->posRadD),
            mR3CAST(sortedSphMarkersD->velMasD), mR4CAST(sortedSphMarkersD->rhoPresMuD), mR3CAST(shift_r),
            U1CAST(markersProximityD->gridMarkerIndexD), U1CAST(markersProximityD->cellStartD),
            U1CAST(markersProximityD->cellEndD), numObjectsH->numAllMarkers, isErrorD);
        ChUtilsDevice::Sync_CheckError(isErrorH, isErrorD, "CalcVel_XSPH_D");

        CopySortedToOriginal_NonInvasive_R3(fsiGeneralData->vel_XSPH_D, vel_XSPH_Sorted_D, markersProximityD->gridMarkerIndexD);
        // CopySortedToOriginal_NonInvasive_R4(sphMarkersD->posRadD, sortedSphMarkersD->posRadD, markersProximityD->gridMarkerIndexD);
    }

    if (density_initialization % paramsH->densityReinit == 0)
        CopySortedToOriginal_NonInvasive_R4(sphMarkersD->rhoPresMuD, sortedSphMarkersD->rhoPresMuD,
                                            markersProximityD->gridMarkerIndexD);
    hipFree(isErrorD);
    free(isErrorH);
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::AddGravityToFluid() {
    // add gravity to fluid markers
    /* Add outside forces. Don't add gravity to rigids, BCE, and boundaries, it is
     * added in ChSystem */
    Real3 totalFluidBodyForce3 = paramsH->bodyForce3 + paramsH->gravity;
    thrust::device_vector<Real4> bodyForceD(numObjectsH->numAllMarkers);
    thrust::fill(bodyForceD.begin(), bodyForceD.end(), mR4(totalFluidBodyForce3));
    thrust::transform(
        fsiGeneralData->derivVelRhoD_old.begin() + fsiGeneralData->referenceArray[0].x,
        fsiGeneralData->derivVelRhoD_old.begin() + fsiGeneralData->referenceArray[0].y, bodyForceD.begin(),
        fsiGeneralData->derivVelRhoD_old.begin() + fsiGeneralData->referenceArray[0].x, thrust::plus<Real4>());
    bodyForceD.clear();
}

}  // namespace fsi
}  // namespace chrono
//================================================================================================================================
