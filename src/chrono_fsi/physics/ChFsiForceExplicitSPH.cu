#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Arman Pazouki, Wei Hu
// =============================================================================

#include <thrust/extrema.h>
#include <thrust/remove.h>
#include <thrust/sort.h>

#include "chrono_fsi/physics/ChFsiForceExplicitSPH.cuh"
#include "chrono_fsi/physics/ChSphGeneral.cuh"
#include "chrono_fsi/math/ExactLinearSolvers.cuh"

namespace chrono {
namespace fsi {

//--------------------------------------------------------------------------------------------------------------------------------
__device__ __inline__ void calc_G_Matrix(Real4* sortedPosRad,
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real* G_i,
                                         const uint* numNeighborsPerPart,
                                         const uint* neighborList,
                                         uint* indexOfIndex) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // uint index = indexOfIndex[id];
    uint index = id;

    if (sortedRhoPreMu[index].w > -0.5f && sortedRhoPreMu[index].w < 0.5f)
        return;

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real h_i = sortedPosRad[index].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    // This is the elements of inverse of G
    Real mGi[9] = {0.0};

    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];
    // examine neighbouring cells
    for (int n = NLStart; n < NLEnd; n++) {
        uint j = neighborList[n];
        if (j == index) {
            continue;
        }
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real dd = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z;
        if (dd > SqRadii || sortedRhoPreMu[j].w < -1.5)
            continue;
        Real3 grad_i_wij = GradWh(rij, h_i);
        Real3 grw_vj = grad_i_wij * paramsD.volume0;
        mGi[0] -= rij.x * grw_vj.x;
        mGi[1] -= rij.x * grw_vj.y;
        mGi[2] -= rij.x * grw_vj.z;
        mGi[3] -= rij.y * grw_vj.x;
        mGi[4] -= rij.y * grw_vj.y;
        mGi[5] -= rij.y * grw_vj.z;
        mGi[6] -= rij.z * grw_vj.x;
        mGi[7] -= rij.z * grw_vj.y;
        mGi[8] -= rij.z * grw_vj.z;
    }

    Real Det = (mGi[0] * mGi[4] * mGi[8] - mGi[0] * mGi[5] * mGi[7] - mGi[1] * mGi[3] * mGi[8] +
                mGi[1] * mGi[5] * mGi[6] + mGi[2] * mGi[3] * mGi[7] - mGi[2] * mGi[4] * mGi[6]);
    if (abs(Det) > 0.01) {
        Real OneOverDet = 1.0 / Det;
        G_i[0] = (mGi[4] * mGi[8] - mGi[5] * mGi[7]) * OneOverDet;
        G_i[1] = -(mGi[1] * mGi[8] - mGi[2] * mGi[7]) * OneOverDet;
        G_i[2] = (mGi[1] * mGi[5] - mGi[2] * mGi[4]) * OneOverDet;
        G_i[3] = -(mGi[3] * mGi[8] - mGi[5] * mGi[6]) * OneOverDet;
        G_i[4] = (mGi[0] * mGi[8] - mGi[2] * mGi[6]) * OneOverDet;
        G_i[5] = -(mGi[0] * mGi[5] - mGi[2] * mGi[3]) * OneOverDet;
        G_i[6] = (mGi[3] * mGi[7] - mGi[4] * mGi[6]) * OneOverDet;
        G_i[7] = -(mGi[0] * mGi[7] - mGi[1] * mGi[6]) * OneOverDet;
        G_i[8] = (mGi[0] * mGi[4] - mGi[1] * mGi[3]) * OneOverDet;
    } else {
        for (int i = 0; i < 9; i++) {
            G_i[i] = 0.0;
        }
        G_i[0] = 1;
        G_i[4] = 1;
        G_i[8] = 1;
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ __inline__ void calc_A_Matrix(Real4* sortedPosRad,
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real* A_i,
                                         Real* G_i,
                                         const uint* numNeighborsPerPart,
                                         const uint* neighborList,
                                         uint* indexOfIndex) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // uint index = indexOfIndex[id];
    uint index = id;

    if (sortedRhoPreMu[index].w > -0.5f && sortedRhoPreMu[index].w < 0.5f)
        return;

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real h_i = sortedPosRad[index].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);

    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];
    // examine neighbouring cells
    for (int n = NLStart; n < NLEnd; n++) {
        uint j = neighborList[n];
        if (j == index) {
            continue;
        }
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real dd = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z;
        if (dd > SqRadii || sortedRhoPreMu[j].w < -1.5)
            continue;
        Real h_j = sortedPosRad[j].w;
        Real h_ij = 0.5 * (h_j + h_i);
        Real3 grad_ij = GradWh(rij, h_ij);
        Real V_j = paramsD.markerMass / paramsD.rho0;
        Real com_part = 0;
        com_part = (G_i[0] * grad_ij.x + G_i[1] * grad_ij.y + G_i[2] * grad_ij.z) * V_j;
        A_i[0] += rij.x * rij.x * com_part;  // 111
        A_i[1] += rij.x * rij.y * com_part;  // 112
        A_i[2] += rij.x * rij.z * com_part;  // 113
        A_i[3] += rij.y * rij.x * com_part;  // 121
        A_i[4] += rij.y * rij.y * com_part;  // 122
        A_i[5] += rij.y * rij.z * com_part;  // 123
        A_i[6] += rij.z * rij.x * com_part;  // 131
        A_i[7] += rij.z * rij.y * com_part;  // 132
        A_i[8] += rij.z * rij.z * com_part;  // 133
        com_part = (G_i[3] * grad_ij.x + G_i[4] * grad_ij.y + G_i[5] * grad_ij.z) * V_j;
        A_i[9] += rij.x * rij.x * com_part;   // 211
        A_i[10] += rij.x * rij.y * com_part;  // 212
        A_i[11] += rij.x * rij.z * com_part;  // 213
        A_i[12] += rij.y * rij.x * com_part;  // 221
        A_i[13] += rij.y * rij.y * com_part;  // 222
        A_i[14] += rij.y * rij.z * com_part;  // 223
        A_i[15] += rij.z * rij.x * com_part;  // 231
        A_i[16] += rij.z * rij.y * com_part;  // 232
        A_i[17] += rij.z * rij.z * com_part;  // 233
        com_part = (G_i[6] * grad_ij.x + G_i[7] * grad_ij.y + G_i[8] * grad_ij.z) * V_j;
        A_i[18] += rij.x * rij.x * com_part;  // 311
        A_i[19] += rij.x * rij.y * com_part;  // 312
        A_i[20] += rij.x * rij.z * com_part;  // 313
        A_i[21] += rij.y * rij.x * com_part;  // 321
        A_i[22] += rij.y * rij.y * com_part;  // 322
        A_i[23] += rij.y * rij.z * com_part;  // 323
        A_i[24] += rij.z * rij.x * com_part;  // 331
        A_i[25] += rij.z * rij.y * com_part;  // 332
        A_i[26] += rij.z * rij.z * com_part;  // 333
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ __inline__ void calc_L_Matrix(Real4* sortedPosRad,
                                         Real3* sortedVelMas,
                                         Real4* sortedRhoPreMu,
                                         Real* A_i,
                                         Real* L_i,
                                         Real* G_i,
                                         const uint* numNeighborsPerPart,
                                         const uint* neighborList,
                                         uint* indexOfIndex) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // uint index = indexOfIndex[id];
    uint index = id;

    if (sortedRhoPreMu[index].w > -0.5f && sortedRhoPreMu[index].w < 0.5f)
        return;

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real h_i = sortedPosRad[index].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;

    Real B[36] = {0.0};
    Real L[6] = {0.0};

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];
    // examine neighbouring cells
    for (int n = NLStart; n < NLEnd; n++) {
        uint j = neighborList[n];
        if (j == index) {
            continue;
        }
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real dd = rij.x * rij.x + rij.y * rij.y + rij.z * rij.z;
        if (dd > SqRadii || sortedRhoPreMu[j].w < -1.5)
            continue;
        Real d = length(rij);
        Real3 eij = rij / d;

        Real h_j = sortedPosRad[j].w;
        // Real m_j = paramsD.markerMass;
        Real h_ij = 0.5 * (h_j + h_i);
        Real3 grad_ij = GradWh(rij, h_ij);
        Real V_j = paramsD.markerMass / paramsD.rho0;
        Real com_part = 0;
        // mn=11

        Real XX = (eij.x * grad_ij.x);
        Real XY = (eij.x * grad_ij.y + eij.y * grad_ij.x);
        Real XZ = (eij.x * grad_ij.z + eij.z * grad_ij.x);
        Real YY = (eij.y * grad_ij.y);
        Real YZ = (eij.y * grad_ij.z + eij.z * grad_ij.y);
        Real ZZ = (eij.z * grad_ij.z);

        com_part = (A_i[0] * eij.x + A_i[9] * eij.y + A_i[18] * eij.z + rij.x * eij.x) * V_j;
        B[6 * 0 + 0] += com_part * XX;  // 11
        B[6 * 0 + 1] += com_part * XY;  // 12
        B[6 * 0 + 2] += com_part * XZ;  // 13
        B[6 * 0 + 3] += com_part * YY;  // 14
        B[6 * 0 + 4] += com_part * YZ;  // 15
        B[6 * 0 + 5] += com_part * ZZ;  // 15
        // mn=12
        com_part = (A_i[1] * eij.x + A_i[10] * eij.y + A_i[19] * eij.z + rij.x * eij.y) * V_j;
        B[6 * 1 + 0] += com_part * XX;  // 21
        B[6 * 1 + 1] += com_part * XY;  // 22
        B[6 * 1 + 2] += com_part * XZ;  // 23
        B[6 * 1 + 3] += com_part * YY;  // 24
        B[6 * 1 + 4] += com_part * YZ;  // 25
        B[6 * 1 + 5] += com_part * ZZ;  // 25

        // mn=13
        com_part = (A_i[2] * eij.x + A_i[11] * eij.y + A_i[20] * eij.z + rij.x * eij.z) * V_j;
        B[6 * 2 + 0] += com_part * XX;  // 31
        B[6 * 2 + 1] += com_part * XY;  // 32
        B[6 * 2 + 2] += com_part * XZ;  // 33
        B[6 * 2 + 3] += com_part * YY;  // 34
        B[6 * 2 + 4] += com_part * YZ;  // 35
        B[6 * 2 + 5] += com_part * ZZ;  // 36

        // Note that we skip mn=21 since it is similar to mn=12
        // mn=22
        com_part = (A_i[4] * eij.x + A_i[13] * eij.y + A_i[22] * eij.z + rij.y * eij.y) * V_j;
        B[6 * 3 + 0] += com_part * XX;  // 41
        B[6 * 3 + 1] += com_part * XY;  // 42
        B[6 * 3 + 2] += com_part * XZ;  // 43
        B[6 * 3 + 3] += com_part * YY;  // 44
        B[6 * 3 + 4] += com_part * YZ;  // 45
        B[6 * 3 + 5] += com_part * ZZ;  // 46

        // mn=23
        com_part = (A_i[5] * eij.x + A_i[14] * eij.y + A_i[23] * eij.z + rij.y * eij.z) * V_j;
        B[6 * 4 + 0] += com_part * XX;  // 51
        B[6 * 4 + 1] += com_part * XY;  // 52
        B[6 * 4 + 2] += com_part * XZ;  // 53
        B[6 * 4 + 3] += com_part * YY;  // 54
        B[6 * 4 + 4] += com_part * YZ;  // 55
        B[6 * 4 + 5] += com_part * ZZ;  // 56
        // mn=33
        com_part = (A_i[8] * eij.x + A_i[17] * eij.y + A_i[26] * eij.z + rij.z * eij.z) * V_j;
        B[6 * 5 + 0] += com_part * XX;  // 61
        B[6 * 5 + 1] += com_part * XY;  // 62
        B[6 * 5 + 2] += com_part * XZ;  // 63
        B[6 * 5 + 3] += com_part * YY;  // 64
        B[6 * 5 + 4] += com_part * YZ;  // 65
        B[6 * 5 + 5] += com_part * ZZ;  // 66
    }

    inv6xdelta_mn(B, L);
    L_i[0] = L[0];
    L_i[1] = L[1];
    L_i[2] = L[2];
    L_i[3] = L[1];
    L_i[4] = L[3];
    L_i[5] = L[4];
    L_i[6] = L[2];
    L_i[7] = L[4];
    L_i[8] = L[5];

    // Real Det = (L_i[0] * L_i[4] * L_i[8] - L_i[0] * L_i[5] * L_i[7] - L_i[1] * L_i[3] * L_i[8] +
    //             L_i[1] * L_i[5] * L_i[6] + L_i[2] * L_i[3] * L_i[7] - L_i[2] * L_i[4] * L_i[6]);
    // if (abs(Det) < 0.01) {
    //     for (int i = 0; i < 9; i++) {
    //         L_i[0 * 9 + i] = 0.0;
    //         L_i[0 * 9 + 0] = 1;
    //         L_i[0 * 9 + 4] = 1;
    //         L_i[0 * 9 + 8] = 1;
    //     }
    // }
    // printf("L Det %f\n", Det);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calIndexOfIndex(uint* indexOfIndex, uint* identityOfIndex, uint* gridMarkerIndex) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    indexOfIndex[id] = id;
    if (gridMarkerIndex[id] >= numObjectsD.numFluidMarkers &&
        gridMarkerIndex[id] < numObjectsD.numFluidMarkers + numObjectsD.numBoundaryMarkers) {
        identityOfIndex[id] = 1;
    } else {
        identityOfIndex[id] = 0;
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Shear_Stress_Rate(uint* indexOfIndex,
                                  Real4* sortedPosRad,
                                  Real4* sortedRhoPreMu,
                                  Real3* sortedVelMas,
                                  Real3* velMas_ModifiedBCE,
                                  Real4* rhoPreMu_ModifiedBCE,
                                  Real3* sortedTauXxYyZz,
                                  Real3* sortedTauXyXzYz,
                                  Real3* sortedDerivTauXxYyZz,
                                  Real3* sortedDerivTauXyXzYz,
                                  uint* gridMarkerIndex,
                                  uint* cellStart,
                                  uint* cellEnd) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    uint index = indexOfIndex[id];

    if (sortedRhoPreMu[index].w > -0.5)
        return;

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real3 velMasA = sortedVelMas[index];
    Real hA = sortedPosRad[index].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;

    Real tauxx = sortedTauXxYyZz[index].x;
    Real tauyy = sortedTauXxYyZz[index].y;
    Real tauzz = sortedTauXxYyZz[index].z;
    Real tauxy = sortedTauXyXzYz[index].x;
    Real tauxz = sortedTauXyXzYz[index].y;
    Real tauyz = sortedTauXyXzYz[index].z;
    Real tauzx = tauxz;
    Real tauzy = tauyz;
    Real tauyx = tauxy;
    Real dTauxx = 0.0;
    Real dTauyy = 0.0;
    Real dTauzz = 0.0;
    Real dTauxy = 0.0;
    Real dTauxz = 0.0;
    Real dTauyz = 0.0;

    Real G_i[9] = {0.0};
    calc_G_Matrix(sortedPosRad, sortedVelMas, sortedRhoPreMu, G_i, cellStart, cellEnd, indexOfIndex);

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
            for (int x = -1; x <= 1; x++) {
                int3 neighbourPos = gridPos + mI3(x, y, z);
                uint gridHash = calcGridHash(neighbourPos);
                uint startIndex = cellStart[gridHash];
                uint endIndex = cellEnd[gridHash];
                for (uint j = startIndex; j < endIndex; j++) {
                    if (j != index) {
                        Real3 posRadB = mR3(sortedPosRad[j]);
                        Real3 dist3 = Distance(posRadA, posRadB);
                        Real dd = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z;
                        if (dd > SqRadii)
                            continue;
                        Real3 velMasB = sortedVelMas[j];
                        Real4 rhoPresMuB = sortedRhoPreMu[j];
                        if (rhoPresMuB.w > -0.5) {
                            int bceIndexB = gridMarkerIndex[j] - numObjectsD.numFluidMarkers;
                            if (bceIndexB < 0 || bceIndexB >= numObjectsD.numBceMarkers) {
                                printf("Error! bceIndex out of bound, Shear_Stress_Rate !\n");
                            }
                            rhoPresMuB = rhoPreMu_ModifiedBCE[bceIndexB];
                            velMasB = velMas_ModifiedBCE[bceIndexB];
                            velMasB = 2.0 * velMasB - velMasA;
                        }
                        Real rhoB = rhoPresMuB.x;
                        Real mB = paramsD.markerMass;
                        Real3 gradW = GradWh(dist3, hA);

                        Real3 gradW_new;
                        gradW_new.x = G_i[0] * gradW.x + G_i[1] * gradW.y + G_i[2] * gradW.z;
                        gradW_new.y = G_i[3] * gradW.x + G_i[4] * gradW.y + G_i[5] * gradW.z;
                        gradW_new.z = G_i[6] * gradW.x + G_i[7] * gradW.y + G_i[8] * gradW.z;
                        gradW = gradW_new;

                        // start to calculate the rate
                        Real Gm = paramsD.G_shear;  // shear modulus of the material
                        Real half_mB_over_rhoB = 0.5 * (mB / rhoB);
                        Real3 vAB = velMasA - velMasB;
                        Real3 vAB_h = (velMasA - velMasB) * half_mB_over_rhoB;
                        // entries of strain rate tensor
                        Real exx = -2.0 * vAB_h.x * gradW.x;
                        Real eyy = -2.0 * vAB_h.y * gradW.y;
                        Real ezz = -2.0 * vAB_h.z * gradW.z;
                        Real exy = -vAB_h.x * gradW.y - vAB_h.y * gradW.x;
                        Real exz = -vAB_h.x * gradW.z - vAB_h.z * gradW.x;
                        Real eyz = -vAB_h.y * gradW.z - vAB_h.z * gradW.y;
                        // entries of rotation rate (spin) tensor
                        // Real wxx = 0.0;
                        // Real wyy = 0.0;
                        // Real wzz = 0.0;
                        Real wxy = -vAB_h.x * gradW.y + vAB_h.y * gradW.x;
                        Real wxz = -vAB_h.x * gradW.z + vAB_h.z * gradW.x;
                        Real wyz = -vAB_h.y * gradW.z + vAB_h.z * gradW.y;
                        Real wyx = -wxy;
                        // Real wzx = -wxz;
                        Real wzy = -wyz;

                        Real edia = 1.0 / 3.0 * (exx + eyy + ezz);
                        Real twoGm = 2.0 * Gm;
                        Real K_edia = paramsD.K_bulk * 1.0 * edia;
                        dTauxx += twoGm * (exx - edia) + 2.0 * (tauxy * wxy + tauxz * wxz) + K_edia;
                        dTauyy += twoGm * (eyy - edia) - 2.0 * (tauyx * wxy - tauyz * wyz) + K_edia;
                        dTauzz += twoGm * (ezz - edia) - 2.0 * (tauzx * wxz + tauzy * wyz) + K_edia;
                        dTauxy += twoGm * exy - (tauxx * wxy + tauxz * wzy) + (wxy * tauyy + wxz * tauzy);
                        dTauxz += twoGm * exz - (tauxx * wxz + tauxy * wyz) + (wxy * tauyz + wxz * tauzz);
                        dTauyz += twoGm * eyz - (tauyx * wxz + tauyy * wyz) + (wyx * tauxz + wyz * tauzz);
                    }
                }
            }
        }
    }
    sortedDerivTauXxYyZz[index] = mR3(dTauxx, dTauyy, dTauzz);
    sortedDerivTauXyXzYz[index] = mR3(dTauxy, dTauxz, dTauyz);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calcRho_kernel(Real4* sortedPosRad,
                               Real4* sortedRhoPreMu,
                               Real4* sortedRhoPreMu_old,
                               const uint* numNeighborsPerPart,
                               const uint* neighborList,
                               int density_reinit,
                               volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numAllMarkers)
        return;

    if (sortedRhoPreMu[index].w > -0.5 && sortedRhoPreMu[index].w < 0.5)
        return;

    sortedRhoPreMu_old[index].y = Eos(sortedRhoPreMu_old[index].x, sortedRhoPreMu_old[index].w);

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real h_i = sortedPosRad[index].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;

    Real sum_mW = 0;
    Real sum_mW_rho = 0.0000001;
    Real sum_W = 0.0;
    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    for (int n = NLStart; n < NLEnd; n++) {
        uint j = neighborList[n];
        if (j == index) {
            continue;
        }
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 dist3 = Distance(posRadA, posRadB);
        Real dd = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z;
        if (dd > SqRadii)
            continue;
        if (sortedRhoPreMu_old[j].w > -1.5 && sortedRhoPreMu_old[j].w < -0.5) {
            Real h_j = sortedPosRad[j].w;
            Real m_j = paramsD.markerMass;
            Real d = length(dist3);
            Real W3 = W3h(d, 0.5 * (h_j + h_i));
            sum_mW += m_j * W3;
            sum_W += W3;
            sum_mW_rho += m_j * W3 / sortedRhoPreMu_old[j].x;
        }
    }

    // sortedRhoPreMu[index].x = sum_mW;
    if ((density_reinit == 0) && (sortedRhoPreMu[index].w > -1.5) && (sortedRhoPreMu[index].w < -0.5))
        sortedRhoPreMu[index].x = sum_mW / sum_mW_rho;

    if ((sortedRhoPreMu[index].x > 3 * paramsD.rho0 || sortedRhoPreMu[index].x < 0.01 * paramsD.rho0) &&
        (sortedRhoPreMu[index].w > -1.5) && (sortedRhoPreMu[index].w < -0.5))
        printf("(calcRho_kernel)density marker %d, sum_mW=%f, sum_W=%f, h_i=%f\n", index, sum_mW, sum_W, h_i);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void calcKernelSupport(const Real4* sortedPosRad,
                                  const Real4* sortedRhoPreMu,
                                  Real3* sortedKernelSupport,
                                  const uint* mapOriginalToSorted,
                                  const uint* numNeighborsPerPart,
                                  const uint* neighborList,
                                  volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numAllMarkers)
        return;

    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];
    Real h_i = sortedPosRad[index].w;
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;
    Real3 posRadA = mR3(sortedPosRad[index]);

    Real W0 = W3h(0, h_i);
    Real sum_W_all = W0;
    Real sum_W_identical = W0;

    // Use the neighbors list
    for (int i = NLStart; i < NLEnd; i++) {
        uint j = neighborList[i];
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 dist3 = Distance(posRadA, posRadB);
        Real dd = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z;

        if (dd > SqRadii)
            continue;
        Real d = sqrt(dd);
        Real W3 = W3h(d, h_i);
        sum_W_all += W3;
        if (abs(sortedRhoPreMu[index].w - sortedRhoPreMu[j].w) < 0.001) {
            sum_W_identical += W3;
        }
    }

    sortedKernelSupport[index].x = sum_W_all;
    sortedKernelSupport[index].y = sum_W_identical;
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ __inline__ void modifyPressure(Real4& rhoPresMuB, const Real3& dist3Alpha) {
    // body force in x direction
    rhoPresMuB.y = (dist3Alpha.x > 0.5 * paramsD.boxDims.x) ? (rhoPresMuB.y - paramsD.deltaPress.x) : rhoPresMuB.y;
    rhoPresMuB.y = (dist3Alpha.x < -0.5 * paramsD.boxDims.x) ? (rhoPresMuB.y + paramsD.deltaPress.x) : rhoPresMuB.y;
    // body force in y direction
    rhoPresMuB.y = (dist3Alpha.y > 0.5 * paramsD.boxDims.y) ? (rhoPresMuB.y - paramsD.deltaPress.y) : rhoPresMuB.y;
    rhoPresMuB.y = (dist3Alpha.y < -0.5 * paramsD.boxDims.y) ? (rhoPresMuB.y + paramsD.deltaPress.y) : rhoPresMuB.y;
    // body force in z direction
    rhoPresMuB.y = (dist3Alpha.z > 0.5 * paramsD.boxDims.z) ? (rhoPresMuB.y - paramsD.deltaPress.z) : rhoPresMuB.y;
    rhoPresMuB.y = (dist3Alpha.z < -0.5 * paramsD.boxDims.z) ? (rhoPresMuB.y + paramsD.deltaPress.z) : rhoPresMuB.y;
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 CubicSolve(Real aa, Real bb, Real cc, Real dd) {
    Real disc, q, r, dum1, dum2, term1, r13;
    bb /= aa;
    cc /= aa;
    dd /= aa;
    if (aa == 0) {
        return mR3(0, 0, 0);
    }
    if (abs(bb) < 1e-9) {
        return mR3(0, 0, 0);
    }
    if (abs(cc) < 1e-9) {
        return mR3(0, 0, 0);
    }
    if (abs(dd) < 1e-9) {
        return mR3(0, 0, 0);
    }
    q = (3.0 * cc - (bb * bb)) / 9.0;
    r = -(27.0 * dd) + bb * (9.0 * cc - 2.0 * (bb * bb));
    r /= 54.0;
    disc = q * q * q + r * r;
    term1 = (bb / 3.0);

    /*     dataForm.x1Im.value = 0; //The first root is always real.
        if (disc > 0) { // one root real, two are complex
            s = r + Math.sqrt(disc);
            s = ((s < 0) ? -Math.pow(-s, (1.0/3.0)) : Math.pow(s, (1.0/3.0)));
            t = r - Math.sqrt(disc);
            t = ((t < 0) ? -Math.pow(-t, (1.0/3.0)) : Math.pow(t, (1.0/3.0)));
            dataForm.x1Re.value = -term1 + s + t;
            term1 += (s + t)/2.0;
            dataForm.x3Re.value = dataForm.x2Re.value = -term1;
            term1 = Math.sqrt(3.0)*(-t + s)/2;
            dataForm.x2Im.value = term1;
            dataForm.x3Im.value = -term1;
            return;
        }
        // End if (disc > 0)
        // The remaining options are all real
        dataForm.x3Im.value = dataForm.x2Im.value = 0;
        if (disc == 0){ // All roots real, at least two are equal.
            r13 = ((r < 0) ? -Math.pow(-r,(1.0/3.0)) : Math.pow(r,(1.0/3.0)));
            dataForm.x1Re.value = -term1 + 2.0*r13;
            dataForm.x3Re.value = dataForm.x2Re.value = -(r13 + term1);
            return;
        } // End if (disc == 0)
    */

    Real xRex, xRey, xRez;
    // have complex root
    if (disc > 0) {
        xRex = 0.0;
        xRey = 0.0;
        xRez = 0.0;
        return mR3(xRex, xRey, xRez);
    }
    // All roots real, at least two are equal.
    if (disc == 0) {
        if (r < 0) {
            r13 = pow(-r, (1.0 / 3.0));
        } else {
            r13 = pow(r, (1.0 / 3.0));
        }
        xRex = -term1 + 2.0 * r13;
        xRey = -(r13 + term1);
        xRez = xRey;
        return mR3(xRex, xRey, xRez);
    }
    // All roots are real and unequal (to get here, q < 0)
    q = -q;
    dum1 = q * q * q;
    dum2 = r / (sqrt(dum1 + 1.0e-9));
    if ((dum2 >= 0) && (dum2 <= 1)) {
        dum1 = acos(dum2);
    } else {
        xRex = 0.0;
        xRey = 0.0;
        xRez = 0.0;
        return mR3(xRex, xRey, xRez);
    }
    r13 = 2.0 * sqrt(q);
    xRex = -term1 + r13 * cos(dum1 / 3.0);
    xRey = -term1 + r13 * cos((dum1 + 2.0 * 3.1415926) / 3.0);
    xRez = -term1 + r13 * cos((dum1 + 4.0 * 3.1415926) / 3.0);

    return mR3(xRex, xRey, xRez);
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 CubicEigen(Real4 c1, Real4 c2, Real4 c3) {
    Real a = c1.x;
    Real b = c1.y;
    Real c = c1.z;
    Real d = c1.w;

    Real l = c2.x;
    Real m = c2.y;
    Real n = c2.z;
    Real k = c2.w;

    Real p = c3.x;
    Real q = c3.y;
    Real r = c3.z;
    Real s = c3.w;

    Real D = (a * m * r + b * p * n + c * l * q) - (a * n * q + b * l * r + c * m * p) + 1.0e-9;
    Real x = ((b * r * k + c * m * s + d * n * q) - (b * n * s + c * q * k + d * m * r)) / D;
    Real y = ((a * n * s + c * p * k + d * l * r) - (a * r * k + c * l * s + d * n * p)) / D;
    Real z = ((a * q * k + b * l * s + d * m * p) - (a * m * s + b * p * k + d * l * q)) / D;

    b = b + 1.0e-9;
    x = 1.0e0;
    z = (-l + a * m / b) / (n - c * m / b);
    y = (-a - c * z) / b;
    Real R = sqrt(x * x + y * y + z * z);
    x = x / R;
    y = y / R;
    z = z / R;

    // if(abs(D) < 1){
    //     return mR3(0,0,0);
    // }

    // if(abs(m) < 0.1){
    //     x=0;
    //     y=1;
    //     z=0;
    //     return mR3(x,y,z);
    // }
    // else{
    //     y=0;
    //     if(abs(c) > 0.1){
    //         x=1;
    //         z=-a/c;
    //         return mR3(x,y,z);
    //     }
    //     if(abs(a) > 0.1){
    //         z=1;
    //         x=-c/a;
    //         return mR3(x,y,z);
    //     }
    // }

    return mR3(x, y, z);
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real4 DifVelocityRho(float G_i[9],
                                       Real3 dist3,
                                       Real d,
                                       Real4 posRadA,
                                       Real4 posRadB,
                                       Real3 velMasA,
                                       Real3 velMasB,
                                       Real4 rhoPresMuA,
                                       Real4 rhoPresMuB,
                                       Real multViscosity) {
    if (rhoPresMuA.w > -0.5 && rhoPresMuB.w > -0.5)
        return mR4(0.0);

    Real3 gradW = GradWh(dist3, (posRadA.w + posRadB.w) * 0.5);

    // Continuty equation
    Real derivRho = paramsD.markerMass * dot(velMasA - velMasB, gradW);

    // Viscosity
    Real rAB_Dot_GradWh = dot(dist3, gradW);
    Real rAB_Dot_GradWh_OverDist = rAB_Dot_GradWh / (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML);
    Real3 derivV = -paramsD.markerMass *
                       (rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)) *
                       gradW +
                   paramsD.markerMass * (8.0f * multViscosity) * paramsD.mu0 * rAB_Dot_GradWh_OverDist *
                       (velMasA - velMasB) / square(rhoPresMuA.x + rhoPresMuB.x);

    // Artificial viscosity
    Real vAB_Dot_rAB = dot(velMasA - velMasB, dist3);
    // change to 1==1 if needs artificial viscosity
    if ((vAB_Dot_rAB < 0.0) && (1 == 1)) {
        Real alpha = paramsD.Ar_vis_alpha;
        Real c_ab = paramsD.Cs;
        Real rho = 0.5f * (rhoPresMuA.x * rhoPresMuB.x);
        Real nu = -alpha * paramsD.HSML * c_ab / rho;
        Real derivM1 =
            -paramsD.markerMass * (nu * vAB_Dot_rAB / (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML));
        derivV.x += derivM1 * gradW.x;
        derivV.y += derivM1 * gradW.y;
        derivV.z += derivM1 * gradW.z;
    }

    return mR4(derivV, derivRho);
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real4 DifVelocityRho_ElasticSPH(Real W_ini_inv,
                                                  Real W_AB,
                                                  Real3 gradW,
                                                  Real3 dist3,
                                                  Real d,
                                                  Real invd,
                                                  Real4 posRadA,
                                                  Real4 posRadB,
                                                  Real3 velMasA_in,
                                                  Real3 velMasB_in,
                                                  Real4 rhoPresMuA,
                                                  Real4 rhoPresMuB,
                                                  Real3 tauXxYyZz_A_in,
                                                  Real3 tauXyXzYz_A_in,
                                                  Real3 tauXxYyZz_B_in,
                                                  Real3 tauXyXzYz_B_in) {
    if (rhoPresMuA.w > -0.5 && rhoPresMuB.w > -0.5)
        return mR4(0.0);

    Real3 velMasA = velMasA_in;
    Real3 velMasB = velMasB_in;
    Real3 tauXxYyZz_A = tauXxYyZz_A_in;
    Real3 tauXxYyZz_B = tauXxYyZz_B_in;
    Real3 tauXyXzYz_A = tauXyXzYz_A_in;
    Real3 tauXyXzYz_B = tauXyXzYz_B_in;

    /*if (rhoPresMuA.w < -0.5 && rhoPresMuB.w > -0.5) {
        tauXxYyZz_B = tauXxYyZz_A;
        tauXyXzYz_B = tauXyXzYz_A;
        // velMasB = 2.0*velMasB - velMasA; // noslip BC
    }
    if (rhoPresMuA.w > -0.5 && rhoPresMuB.w < -0.5) {
        tauXxYyZz_A = tauXxYyZz_B;
        tauXyXzYz_A = tauXyXzYz_B;
        // velMasA = 2.0*velMasA - velMasB; // noslip BC
    }*/

    Real Mass = paramsD.markerMass;
    Real MassOverRho = Mass * paramsD.invrho0 * paramsD.invrho0;
    Real3 MA_gradW = gradW * MassOverRho;

    Real derivVx = (tauXxYyZz_A.x + tauXxYyZz_B.x) * MA_gradW.x + (tauXyXzYz_A.x + tauXyXzYz_B.x) * MA_gradW.y +
                   (tauXyXzYz_A.y + tauXyXzYz_B.y) * MA_gradW.z;
    Real derivVy = (tauXyXzYz_A.x + tauXyXzYz_B.x) * MA_gradW.x + (tauXxYyZz_A.y + tauXxYyZz_B.y) * MA_gradW.y +
                   (tauXyXzYz_A.z + tauXyXzYz_B.z) * MA_gradW.z;
    Real derivVz = (tauXyXzYz_A.y + tauXyXzYz_B.y) * MA_gradW.x + (tauXyXzYz_A.z + tauXyXzYz_B.z) * MA_gradW.y +
                   (tauXxYyZz_A.z + tauXxYyZz_B.z) * MA_gradW.z;

    // TODO: Visco-plastic model
    // Real vel = length(velMasA);
    // if(vel > 0.3){
    //     Real rAB_Dot_GradWh = dot(dist3, gradW);
    //     Real rAB_Dot_GradWh_OverDist = rAB_Dot_GradWh / (d * d + paramsD.epsMinMarkersDis * paramsD.HSML *
    //     paramsD.HSML); Real3 derivV = - paramsD.markerMass *(rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) +
    //     rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)) * gradW
    //                    + paramsD.markerMass * (8.0f * multViscosity) * paramsD.mu_fric_s
    //                    * pow(rhoPresMuA.x + rhoPresMuB.x, Real(-2)) * rAB_Dot_GradWh_OverDist * (velMasA - velMasB);
    //     derivVx = derivV.x;
    //     derivVy = derivV.y;
    //     derivVz = derivV.z;
    // }

    // Artificial viscosity
    Real vAB_rAB = dot(velMasA - velMasB, dist3);
    // if (vAB_rAB < 0.0) {
    Real nu = -paramsD.Ar_vis_alpha * paramsD.HSML * paramsD.Cs * paramsD.invrho0;
    Real derivM1 = -Mass * (nu * vAB_rAB * (invd * invd));  //+ paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML
    derivVx += derivM1 * gradW.x;
    derivVy += derivM1 * gradW.y;
    derivVz += derivM1 * gradW.z;
    // }

    // Artifical pressure to handle tensile instability issue.
    // A complete artifical stress should be implemented in the future.
    /*if (paramsD.Coh_coeff > 1e-5) {
        Real Pa = -1.0 / 3.0 * (tauXxYyZz_A.x + tauXxYyZz_A.y + tauXxYyZz_A.z);
        if (Pa < 0.0) {
            Real Pb = -1.0 / 3.0 * (tauXxYyZz_B.x + tauXxYyZz_B.y + tauXxYyZz_B.z);
            Real epsi = 0.5;
            Real Ra = Pa * epsi * paramsD.invrho0 * paramsD.invrho0;
            Real Rb = Pb * epsi * paramsD.invrho0 * paramsD.invrho0;
            Real fAB = W_AB * W_ini_inv;
            Real small_F = Mass * pow(fAB, 3.0) * (Ra + Rb);
            derivVx += small_F * gradW.x;
            derivVy += small_F * gradW.y;
            derivVz += small_F * gradW.z;
        }
    }*/

    // TOTO: Damping force
    // if (1 == 0) {
    //     Real xi0 = paramsD.Vis_Dam;
    //     Real E0 = paramsD.E_young;
    //     Real h0 = paramsD.HSML;
    //     Real Cd = xi0 * sqrt(E0 / (rhoA * h0 * h0));
    //     derivVx -= Cd * velMasA.x;
    //     derivVy -= Cd * velMasA.y;
    //     derivVz -= Cd * velMasA.z;
    // }

    // Real derivRho = Mass * dot(vel_XSPH_A - vel_XSPH_B, gradW);
    return mR4(derivVx, derivVy, derivVz, 0.0);
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 GradientOperator(float G_i[9],
                                         Real3 dist3,
                                         Real4 posRadA,
                                         Real4 posRadB,
                                         Real fA,
                                         Real fB,
                                         Real4 rhoPresMuA,
                                         Real4 rhoPresMuB) {
    Real3 gradW = GradWh(dist3, (posRadA.w + posRadB.w) * 0.5);
    Real3 gradW_new;
    gradW_new.x = G_i[0] * gradW.x + G_i[1] * gradW.y + G_i[2] * gradW.z;
    gradW_new.y = G_i[3] * gradW.x + G_i[4] * gradW.y + G_i[5] * gradW.z;
    gradW_new.z = G_i[6] * gradW.x + G_i[7] * gradW.y + G_i[8] * gradW.z;

    Real Vol = paramsD.markerMass / rhoPresMuB.x;
    Real fji = fB - fA;
    Real Gra_ij_x = fji * gradW_new.x * Vol;
    Real Gra_ij_y = fji * gradW_new.y * Vol;
    Real Gra_ij_z = fji * gradW_new.z * Vol;

    return mR3(Gra_ij_x, Gra_ij_y, Gra_ij_z);
}

//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real4 LaplacianOperator(float G_i[9],
                                          float L_i[9],
                                          Real3 dist3,
                                          Real4 posRadA,
                                          Real4 posRadB,
                                          Real fA,
                                          Real fB,
                                          Real4 rhoPresMuA,
                                          Real4 rhoPresMuB) {
    Real3 gradW = GradWh(dist3, (posRadA.w + posRadB.w) * 0.5);
    Real d = length(dist3);
    Real3 eij = dist3 / d;

    Real Vol = paramsD.markerMass / rhoPresMuB.x;
    Real fij = fA - fB;

    Real ex_Gwx = eij.x * gradW.x;
    Real ex_Gwy = eij.x * gradW.y;
    Real ex_Gwz = eij.x * gradW.z;
    Real ey_Gwx = eij.y * gradW.x;
    Real ey_Gwy = eij.y * gradW.y;
    Real ey_Gwz = eij.y * gradW.z;
    Real ez_Gwx = eij.z * gradW.x;
    Real ez_Gwy = eij.z * gradW.y;
    Real ez_Gwz = eij.z * gradW.z;

    Real Part1 = L_i[0] * ex_Gwx + L_i[1] * ex_Gwy + L_i[2] * ex_Gwz + L_i[3] * ey_Gwx + L_i[4] * ey_Gwy +
                 L_i[5] * ey_Gwz + L_i[6] * ez_Gwx + L_i[7] * ez_Gwy + L_i[8] * ez_Gwz;
    Real Part2 = fij / d * Vol;
    Real3 Part3 = mR3(-eij.x, -eij.y, -eij.z) * Vol;

    return mR4(2.0 * Part1 * Part2, Part3.x * (2.0 * Part1), Part3.y * (2.0 * Part1), Part3.z * (2.0 * Part1));
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void EOS(Real4* sortedRhoPreMu, volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numAllMarkers)
        return;
    sortedRhoPreMu[index].y = Eos(sortedRhoPreMu[index].x, sortedRhoPreMu[index].w);
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Navier_Stokes(uint* indexOfIndex,
                              Real4* sortedDerivVelRho,
                              Real3* sortedXSPHandShift,
                              Real4* sortedPosRad,
                              Real3* sortedVelMas,
                              Real4* sortedRhoPreMu,
                              uint* gridMarkerIndex,
                              const uint* numNeighborsPerPart,
                              const uint* neighborList,
                              volatile bool* isErrorD) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // uint index = indexOfIndex[id];
    uint index = id;

    // Do nothing for fixed wall BCE particles
    if (sortedRhoPreMu[index].w > -0.5 && sortedRhoPreMu[index].w < 0.5) {
        sortedDerivVelRho[index] = mR4(0.0);
        return;
    }

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real3 velMasA = sortedVelMas[index];
    Real4 rhoPresMuA = sortedRhoPreMu[index];
    Real4 derivVelRho = mR4(0.0);
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;

    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];

    Real G_i[9] = {1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 1.0};
    Real L_i[9] = {1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 1.0};
    if (paramsD.USE_Consistent_G)
        calc_G_Matrix(sortedPosRad, sortedVelMas, sortedRhoPreMu, G_i, numNeighborsPerPart, neighborList, indexOfIndex);

    if (paramsD.USE_Consistent_L) {
        Real A_i[27] = {0.0};
        calc_A_Matrix(sortedPosRad, sortedVelMas, sortedRhoPreMu, A_i, G_i, numNeighborsPerPart, neighborList,
                      indexOfIndex);
        calc_L_Matrix(sortedPosRad, sortedVelMas, sortedRhoPreMu, A_i, L_i, G_i, numNeighborsPerPart, neighborList,
                      indexOfIndex);
    }
    float Gi[9] = {1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 1.0};
    float Li[9] = {1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 1.0};
    Gi[0] = G_i[0];
    Gi[1] = G_i[1];
    Gi[2] = G_i[2];
    Gi[3] = G_i[3];
    Gi[4] = G_i[4];
    Gi[5] = G_i[5];
    Gi[6] = G_i[6];
    Gi[7] = G_i[7];
    Gi[8] = G_i[8];
    Li[0] = L_i[0];
    Li[1] = L_i[1];
    Li[2] = L_i[2];
    Li[3] = L_i[3];
    Li[4] = L_i[4];
    Li[5] = L_i[5];
    Li[6] = L_i[6];
    Li[7] = L_i[7];
    Li[8] = L_i[8];

    Real3 preGra = mR3(0.0);
    Real3 velxGra = mR3(0.0);
    Real3 velyGra = mR3(0.0);
    Real3 velzGra = mR3(0.0);
    Real4 velxLap = mR4(0.0);
    Real4 velyLap = mR4(0.0);
    Real4 velzLap = mR4(0.0);

    Real vA = length(velMasA);
    Real vAdT = vA * paramsD.dT;

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    Real3 inner_sum = mR3(0.0);
    Real sum_w_i = W3h(0.0, sortedPosRad[index].w) * paramsD.volume0;

    for (int n = NLStart; n < NLEnd; n++) {
        uint j = neighborList[n];
        if (j == index) {
            continue;
        }
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 dist3 = Distance(posRadA, posRadB);
        Real dd = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z;
        if (dd > SqRadii)
            continue;
        Real4 rhoPresMuB = sortedRhoPreMu[j];

        // no rigid-rigid force
        if (rhoPresMuA.w > -0.5 && rhoPresMuB.w > -0.5)
            continue;
        Real d = length(dist3);

        // modifyPressure(rhoPresMuB, dist3Alpha);
        // if (!(isfinite(rhoPresMuB.x) && isfinite(rhoPresMuB.y) && isfinite(rhoPresMuB.z))) {
        //     printf("Error! particle rhoPresMuB is NAN: thrown from modifyPressure !\n");
        // }
        Real3 velMasB = sortedVelMas[j];

        Real multViscosit = 1;

        derivVelRho += DifVelocityRho(Gi, dist3, d, sortedPosRad[index], sortedPosRad[j], velMasA, velMasB, rhoPresMuA,
                                      rhoPresMuB, multViscosit);

        preGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], -rhoPresMuA.y, rhoPresMuB.y,
                                   rhoPresMuA, rhoPresMuB);
        velxGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], velMasA.x, velMasB.x, rhoPresMuA,
                                    rhoPresMuB);
        velyGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], velMasA.y, velMasB.y, rhoPresMuA,
                                    rhoPresMuB);
        velzGra += GradientOperator(Gi, dist3, sortedPosRad[index], sortedPosRad[j], velMasA.z, velMasB.z, rhoPresMuA,
                                    rhoPresMuB);
        velxLap += LaplacianOperator(Gi, Li, dist3, sortedPosRad[index], sortedPosRad[j], velMasA.x, velMasB.x,
                                     rhoPresMuA, rhoPresMuB);
        velyLap += LaplacianOperator(Gi, Li, dist3, sortedPosRad[index], sortedPosRad[j], velMasA.y, velMasB.y,
                                     rhoPresMuA, rhoPresMuB);
        velzLap += LaplacianOperator(Gi, Li, dist3, sortedPosRad[index], sortedPosRad[j], velMasA.z, velMasB.z,
                                     rhoPresMuA, rhoPresMuB);

        if (d > paramsD.HSML * 1.0e-9)
            sum_w_i = sum_w_i + W3h(d, sortedPosRad[index].w) * paramsD.volume0;
    }

    Real nu = paramsD.mu0 / paramsD.rho0;
    Real dvxdt = -preGra.x / rhoPresMuA.x +
                 (velxLap.x + velxGra.x * velxLap.y + velxGra.y * velxLap.z + velxGra.z * velxLap.w) * nu;
    Real dvydt = -preGra.y / rhoPresMuA.x +
                 (velyLap.x + velyGra.x * velyLap.y + velyGra.y * velyLap.z + velyGra.z * velyLap.w) * nu;
    Real dvzdt = -preGra.z / rhoPresMuA.x +
                 (velzLap.x + velzGra.x * velzLap.y + velzGra.y * velzLap.z + velzGra.z * velzLap.w) * nu;
    Real drhodt = -paramsD.rho0 * (velxGra.x + velyGra.y + velzGra.z);

    Real Det_G = (Gi[0] * Gi[4] * Gi[8] - Gi[0] * Gi[5] * Gi[7] - Gi[1] * Gi[3] * Gi[8] + Gi[1] * Gi[5] * Gi[6] +
                  Gi[2] * Gi[3] * Gi[7] - Gi[2] * Gi[4] * Gi[6]);
    Real Det_L = (Li[0] * Li[4] * Li[8] - Li[0] * Li[5] * Li[7] - Li[1] * Li[3] * Li[8] + Li[1] * Li[5] * Li[6] +
                  Li[2] * Li[3] * Li[7] - Li[2] * Li[4] * Li[6]);
    if (rhoPresMuA.w > -1.5 && rhoPresMuA.w < -0.5) {
        if (Det_G > 0.9 && Det_G < 1.1 && Det_L > 0.9 && Det_L < 1.1 && sum_w_i > 0.9) {
            derivVelRho = mR4(dvxdt, dvydt, dvzdt, drhodt);
        }
    }

    if (!(isfinite(derivVelRho.x) && isfinite(derivVelRho.y) && isfinite(derivVelRho.z))) {
        printf("Error! particle derivVel is NAN: thrown from ChFsiForceExplicitSPH.cu, collideD !\n");
        *isErrorD = true;
    }

    // add gravity and other body force to fluid markers
    if (rhoPresMuA.w > -1.5 && rhoPresMuA.w < -0.5) {
        Real3 totalFluidBodyForce3 = paramsD.bodyForce3 + paramsD.gravity;
        derivVelRho += mR4(totalFluidBodyForce3);
    }

    sortedDerivVelRho[index] = derivVelRho;

    Real det_r_max = 0.05 * vAdT;
    Real det_r_A = length(inner_sum);
    if (det_r_A < det_r_max) {
        sortedXSPHandShift[index] = inner_sum;
    } else {
        sortedXSPHandShift[index] = inner_sum * det_r_max / (det_r_A + 1e-9);
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void updateBoundaryPres(const uint* activityIdentifierD,
                                   const uint* numNeighborsPerPart,
                                   const uint* neighborList,
                                   const Real4* sortedPosRadD,
                                   Real3* bceAcc,
                                   Real4* sortedRhoPresMuD,
                                   Real3* sortedVelMasD,
                                   Real3* sortedTauXxYyZz,
                                   Real3* sortedTauXyXzYz,
                                   volatile bool* isErrorD) {
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numObjectsD.numAllMarkers)
        return;

    if (activityIdentifierD[index] == 0) {
        return;
    }
    // Ignore all fluid particles
    if (sortedRhoPresMuD[index].w < -0.5f) {
        return;
    }

    Real3 posRadA = mR3(sortedPosRadD[index]);
    Real h_i = sortedPosRadD[index].w;
    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];
    Real sum_pw = 0.0f;
    Real3 sum_rhorw = mR3(0.0);
    Real sum_w = 0.0f;
    Real3 sum_vw = mR3(0.0);
    Real3 sum_tauD = mR3(0.0);
    Real3 sum_tauO = mR3(0.0);

    for (int n = NLStart + 1; n < NLEnd; n++) {
        uint j = neighborList[n];
        // only consider fluid neighbors
        if (sortedRhoPresMuD[j].w > -0.5f) {
            continue;
        }
        Real3 posRadB = mR3(sortedPosRadD[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real d = length(rij);
        Real W3 = W3h(d, h_i);
        sum_w += W3;
        sum_pw += sortedRhoPresMuD[j].y * W3;
        sum_rhorw += sortedRhoPresMuD[j].x * rij * W3;
        sum_vw += sortedVelMasD[j] * W3;
        sum_tauD += sortedTauXxYyZz[j] * W3;
        sum_tauO += sortedTauXyXzYz[j] * W3;
    }
    Real3 prescribedVel;
    if (sum_w > EPSILON) {
        sortedRhoPresMuD[index].y = (sum_pw + dot(paramsD.gravity - bceAcc[index], sum_rhorw)) / sum_w;
        sortedRhoPresMuD[index].x = InvEos(sortedRhoPresMuD[index].y);
        // Applies ADAMI to only Rigid/Flexible markers
        prescribedVel = (sortedRhoPresMuD[index].w > 0.5f) ? (2.0f * sortedVelMasD[index]) : mR3(0.0);
        // prescribedVel = 2.0f * sortedVelMasD[index];
        sortedVelMasD[index] = prescribedVel - sum_vw / sum_w;
        sortedTauXxYyZz[index] = (sum_tauD + dot(paramsD.gravity - bceAcc[index], sum_rhorw)) / sum_w;
        sortedTauXyXzYz[index] = sum_tauO / sum_w;

    } else {
        sortedRhoPresMuD[index].y = 0.0f;
        sortedVelMasD[index] = mR3(0.0);
        sortedTauXxYyZz[index] = mR3(0.0);
        sortedTauXyXzYz[index] = mR3(0.0);
    }
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void NS_SSR(const uint* activityIdentifierD,
                       const Real4* sortedPosRad,
                       const Real3* sortedVelMas,
                       const Real4* sortedRhoPreMu,
                       const Real3* sortedTauXxYyZz,
                       const Real3* sortedTauXyXzYz,
                       const uint* numNeighborsPerPart,
                       const uint* neighborList,
                       Real4* sortedDerivVelRho,
                       Real3* sortedDerivTauXxYyZz,
                       Real3* sortedDerivTauXyXzYz,
                       Real3* sortedXSPHandShift,
                       Real3* sortedKernelSupport,
                       uint* sortedFreeSurfaceIdD,
                       volatile bool* isErrorD) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // uint index = sortedActivityIdD[id];
    uint index = id;
    if (activityIdentifierD[index] == 0) {
        return;
    }

    if (sortedRhoPreMu[index].w > -0.5f && sortedRhoPreMu[index].w < 0.5f)
        return;

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real hA = sortedPosRad[index].w;
    Real3 velMasA = sortedVelMas[index];
    Real4 rhoPresMuA = sortedRhoPreMu[index];
    Real3 TauXxYyZzA = sortedTauXxYyZz[index];
    Real3 TauXyXzYzA = sortedTauXyXzYz[index];
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real4 derivVelRho = mR4(0.0);
    Real3 deltaV = mR3(0.0);

    Real tauxx = sortedTauXxYyZz[index].x;
    Real tauyy = sortedTauXxYyZz[index].y;
    Real tauzz = sortedTauXxYyZz[index].z;
    Real tauxy = sortedTauXyXzYz[index].x;
    Real tauxz = sortedTauXyXzYz[index].y;
    Real tauyz = sortedTauXyXzYz[index].z;
    Real dTauxx = 0.0f;
    Real dTauyy = 0.0f;
    Real dTauzz = 0.0f;
    Real dTauxy = 0.0f;
    Real dTauxz = 0.0f;
    Real dTauyz = 0.0f;
    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];

    // Calculate the correction matrix for gradient operator
    Real G_i[9] = {1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 1.0};
    if (paramsD.USE_Consistent_G) {
        Real mGi[9] = {0.0};
        for (int n = NLStart; n < NLEnd; n++) {
            uint j = neighborList[n];
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real3 grad_i_wij = GradWh(rij, hA);
            Real3 grw_vj = grad_i_wij * paramsD.volume0;
            mGi[0] -= rij.x * grw_vj.x;
            mGi[1] -= rij.x * grw_vj.y;
            mGi[2] -= rij.x * grw_vj.z;
            mGi[3] -= rij.y * grw_vj.x;
            mGi[4] -= rij.y * grw_vj.y;
            mGi[5] -= rij.y * grw_vj.z;
            mGi[6] -= rij.z * grw_vj.x;
            mGi[7] -= rij.z * grw_vj.y;
            mGi[8] -= rij.z * grw_vj.z;
        }
        Real Det = (mGi[0] * mGi[4] * mGi[8] - mGi[0] * mGi[5] * mGi[7] - mGi[1] * mGi[3] * mGi[8] +
                    mGi[1] * mGi[5] * mGi[6] + mGi[2] * mGi[3] * mGi[7] - mGi[2] * mGi[4] * mGi[6]);
        if (abs(Det) > 0.01) {
            Real OneOverDet = 1.0 / Det;
            G_i[0] = (mGi[4] * mGi[8] - mGi[5] * mGi[7]) * OneOverDet;
            G_i[1] = -(mGi[1] * mGi[8] - mGi[2] * mGi[7]) * OneOverDet;
            G_i[2] = (mGi[1] * mGi[5] - mGi[2] * mGi[4]) * OneOverDet;
            G_i[3] = -(mGi[3] * mGi[8] - mGi[5] * mGi[6]) * OneOverDet;
            G_i[4] = (mGi[0] * mGi[8] - mGi[2] * mGi[6]) * OneOverDet;
            G_i[5] = -(mGi[0] * mGi[5] - mGi[2] * mGi[3]) * OneOverDet;
            G_i[6] = (mGi[3] * mGi[7] - mGi[4] * mGi[6]) * OneOverDet;
            G_i[7] = -(mGi[0] * mGi[7] - mGi[1] * mGi[6]) * OneOverDet;
            G_i[8] = (mGi[0] * mGi[4] - mGi[1] * mGi[3]) * OneOverDet;
        }
    }

    Real radii = paramsD.INITSPACE * 1.241f;           // 1.129;//1.241
    Real invRadii = 1.0f / 1.241f * paramsD.INV_INIT;  // 1.0 / radii

    Real vA = length(velMasA);
    Real vAdT = vA * paramsD.dT;
    Real bs_vAdT = paramsD.beta_shifting * vAdT;

    Real3 inner_sum = mR3(0.0);
    Real sum_w_i = W3h(0.0f, hA) * paramsD.volume0;
    Real w_ini_inv = 1.0f / W3h(paramsD.INITSPACE, hA);
    int N_ = 1;
    int N_s = 0;

    // Get the interaction from neighbor particles
    for (int n = NLStart; n < NLEnd; n++) {
        uint j = neighborList[n];
        if (j == index) {
            continue;
        }
        // uint j = neighborListSorted[n];
        Real4 rhoPresMuB = sortedRhoPreMu[j];
        if (rhoPresMuA.w > -0.5f && rhoPresMuB.w > -0.5f)
            continue;  // No BCE-BCE interaction
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 dist3 = Distance(posRadA, posRadB);
        Real d = length(dist3);
        Real invd = 1.0f / d;
        Real3 velMasB = sortedVelMas[j];
        Real3 TauXxYyZzB = sortedTauXxYyZz[j];
        Real3 TauXyXzYzB = sortedTauXyXzYz[j];

        // TODO: Might need to eliminate this double application of ADAMI BC based on what Wei says
        if (rhoPresMuB.w > -0.5) {
            Real chi_A = sortedKernelSupport[index].y / sortedKernelSupport[index].x;
            Real chi_B = sortedKernelSupport[j].y / sortedKernelSupport[j].x;
            Real dA = SuppRadii * (2.0 * chi_A - 1.0);
            Real dB = SuppRadii * (2.0 * chi_B - 1.0);

            int predicateA = (dA < 0.0);
            dA = predicateA ? 0.01 * SuppRadii : dA;

            int predicateB = (dB < 0.0);
            dB = predicateB ? 0.01 * SuppRadii : dB;

            Real dAB = dB / dA;

            // Use predication to avoid branching
            int predicateAB = (dAB > 0.5);
            dAB = predicateAB ? 0.5 : dAB;

            Real3 velMasB_new = dAB * (velMasB - velMasA) + velMasB;

            velMasB = velMasB_new;
        }

        // Correct the kernel function gradient
        Real w_AB = W3h(d, hA);
        Real3 gradW = GradWh(dist3, hA);
        if (paramsD.USE_Consistent_G) {
            Real3 gradW_new;
            gradW_new.x = G_i[0] * gradW.x + G_i[1] * gradW.y + G_i[2] * gradW.z;
            gradW_new.y = G_i[3] * gradW.x + G_i[4] * gradW.y + G_i[5] * gradW.z;
            gradW_new.z = G_i[6] * gradW.x + G_i[7] * gradW.y + G_i[8] * gradW.z;
            gradW = gradW_new;
        }
        // Calculate dv/dt
        derivVelRho += DifVelocityRho_ElasticSPH(w_ini_inv, w_AB, gradW, dist3, d, invd, sortedPosRad[index],
                                                 sortedPosRad[j], velMasA, velMasB, rhoPresMuA, rhoPresMuB, TauXxYyZzA,
                                                 TauXyXzYzA, TauXxYyZzB, TauXyXzYzB);
        // Calculate dsigma/dt
        if (sortedRhoPreMu[index].w < -0.5f) {
            // start to calculate the stress rate
            Real3 vAB = velMasA - velMasB;
            Real3 vAB_h = 0.5f * vAB * paramsD.volume0;
            // entries of strain rate tensor
            Real exx = -2.0f * vAB_h.x * gradW.x;
            Real eyy = -2.0f * vAB_h.y * gradW.y;
            Real ezz = -2.0f * vAB_h.z * gradW.z;
            Real exy = -vAB_h.x * gradW.y - vAB_h.y * gradW.x;
            Real exz = -vAB_h.x * gradW.z - vAB_h.z * gradW.x;
            Real eyz = -vAB_h.y * gradW.z - vAB_h.z * gradW.y;
            // entries of rotation rate (spin) tensor
            Real wxy = -vAB_h.x * gradW.y + vAB_h.y * gradW.x;
            Real wxz = -vAB_h.x * gradW.z + vAB_h.z * gradW.x;
            Real wyz = -vAB_h.y * gradW.z + vAB_h.z * gradW.y;

            Real edia = 0.3333333333333f * (exx + eyy + ezz);
            Real twoG = 2.0f * paramsD.G_shear;
            Real K_edia = paramsD.K_bulk * 1.0 * edia;
            dTauxx += twoG * (exx - edia) + 2.0f * (tauxy * wxy + tauxz * wxz) + K_edia;
            dTauyy += twoG * (eyy - edia) - 2.0f * (tauxy * wxy - tauyz * wyz) + K_edia;
            dTauzz += twoG * (ezz - edia) - 2.0f * (tauxz * wxz + tauyz * wyz) + K_edia;
            dTauxy += twoG * exy - (tauxx * wxy - tauxz * wyz) + (wxy * tauyy + wxz * tauyz);
            dTauxz += twoG * exz - (tauxx * wxz + tauxy * wyz) + (wxy * tauyz + wxz * tauzz);
            dTauyz += twoG * eyz - (tauxy * wxz + tauyy * wyz) - (wxy * tauxz - wyz * tauzz);
        }
        // Do integration for the kernel function, calculate the XSPH term
        if (d > paramsD.HSML * 1.0e-9f) {
            Real Wab = W3h(d, hA);
            // Integration of the kernel function
            sum_w_i += Wab * paramsD.volume0;
            // XSPH
            if (rhoPresMuB.w > -1.5f && rhoPresMuB.w < -0.5f)
                deltaV += paramsD.volume0 * (velMasB - velMasA) * Wab;
            N_ = N_ + 1;
        }
        // Find particles that have contact with this particle
        if (d < 1.25f * radii && rhoPresMuB.w < -0.5f) {
            Real Pen = (radii - d) * invRadii;
            Real3 r_0 = bs_vAdT * invd * dist3;
            Real3 r_s = r_0 * Pen;
            if (d < 1.0f * radii) {
                inner_sum += 3.0f * r_s;
            } else if (d < 1.1f * radii) {
                inner_sum += 1.0f * r_s;
            } else {
                inner_sum += 0.1f * 1.0f * (-r_0);
            }
            N_s = N_s + 1;
        }
    }

    // Check particles who have not enough neighbor particles (only for granular now)
    if (sum_w_i < paramsD.C_Wi) {
        sortedFreeSurfaceIdD[index] = 1;
    } else {
        sortedFreeSurfaceIdD[index] = 0;
    }

    // Calculate the shifting vector
    Real det_r_max = 0.05f * vAdT;
    Real det_r_A = length(inner_sum);
    if (det_r_A < det_r_max) {
        sortedXSPHandShift[index] = inner_sum;
    } else {
        sortedXSPHandShift[index] = inner_sum * det_r_max / (det_r_A + 1e-9f);
    }

    // Add the XSPH term into the shifting vector
    sortedXSPHandShift[index] += paramsD.EPS_XSPH * deltaV * paramsD.dT;

    // Get the shifting velocity
    sortedXSPHandShift[index] = sortedXSPHandShift[index] * paramsD.INV_dT;

    // Add gravity and other body force to fluid markers
    if (rhoPresMuA.w > -1.5f && rhoPresMuA.w < -0.5f) {
        Real3 totalFluidBodyForce3 = paramsD.bodyForce3 + paramsD.gravity;
        derivVelRho += mR4(totalFluidBodyForce3, 0.0f);
    }

    sortedDerivVelRho[index] = derivVelRho;
    sortedDerivTauXxYyZz[index] = mR3(dTauxx, dTauyy, dTauzz);
    sortedDerivTauXyXzYz[index] = mR3(dTauxy, dTauxz, dTauyz);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void CalcVel_XSPH_D(uint* indexOfIndex,
                               Real3* vel_XSPH_Sorted_D,
                               Real4* sortedPosRad,
                               Real3* sortedVelMas,
                               Real4* sortedRhoPreMu,
                               uint* gridMarkerIndex,
                               const uint* numNeighborsPerPart,
                               const uint* neighborList,
                               volatile bool* isErrorD) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // uint index = indexOfIndex[id];
    uint index = id;

    // Do nothing for wall
    if (sortedRhoPreMu[index].w > -0.5 && sortedRhoPreMu[index].w < 0.5) {
        return;
    }

    Real4 rhoPreMuA = sortedRhoPreMu[index];
    Real3 velMasA = sortedVelMas[index];
    Real SuppRadii = RESOLUTION_LENGTH_MULT * paramsD.HSML;
    Real SqRadii = SuppRadii * SuppRadii;
    uint NLStart = numNeighborsPerPart[index];
    uint NLEnd = numNeighborsPerPart[index + 1];

    Real3 posRadA = mR3(sortedPosRad[index]);
    Real3 deltaV = mR3(0);

    // get address in grid
    int3 gridPos = calcGridPos(posRadA);
    Real3 inner_sum = mR3(0.0);
    // Real mi_bar = 0.0, r0 = 0.0;
    Real3 dV = mR3(0.0f);
    // examine neighbouring cells
    for (int n = NLStart; n < NLEnd; n++) {
        uint j = neighborList[n];
        if (j == index) {
            continue;
        }
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 dist3 = Distance(posRadA, posRadB);
        Real dd = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z;
        if (dd > SqRadii)
            continue;
        Real4 rhoPresMuB = sortedRhoPreMu[j];
        if (rhoPresMuB.w > -0.5 || rhoPresMuB.w < -1.5)
            continue;
        Real3 velMasB = sortedVelMas[j];
        Real rho_bar = 0.5 * (rhoPreMuA.x + rhoPresMuB.x);
        Real d = length(dist3);
        deltaV += paramsD.markerMass * (velMasB - velMasA) * W3h(d, paramsD.HSML) / rho_bar;
    }

    vel_XSPH_Sorted_D[index] = paramsD.EPS_XSPH * deltaV + vel_XSPH_Sorted_D[index] * paramsD.INV_dT;

    if (!(isfinite(vel_XSPH_Sorted_D[index].x) && isfinite(vel_XSPH_Sorted_D[index].y) &&
          isfinite(vel_XSPH_Sorted_D[index].z))) {
        printf("Error! particle vXSPH is NAN: thrown from ChFsiForceExplicitSPH.cu, CalcVel_XSPH_D !\n");
        *isErrorD = true;
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
// TODO (Huzaifa): Why have so many seperate SortedToOriginal function (one below this and one in ChFluidDynamics.cu) -
// Can these be combined?
__global__ void CopySortedToOriginal_D(const Real4* sortedDerivVelRho,
                                       const Real3* sortedDerivTauXxYyZz,
                                       const Real3* sortedDerivTauXyXzYz,
                                       Real4* originalDerivVelRho,
                                       Real3* originalDerivTauXxYyZz,
                                       Real3* originalDerivTauXyXzYz,
                                       const uint* gridMarkerIndex,
                                       const uint* activityIdentifierD,
                                       const uint* mapOriginalToSorted,
                                       uint* originalFreeSurfaceId,
                                       const uint* sortedFreeSurfaceId) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // Check the activity of this particle
    uint activity = activityIdentifierD[id];
    if (activity == 0)
        return;

    uint index = mapOriginalToSorted[id];

    originalDerivVelRho[id] = sortedDerivVelRho[index];
    if (paramsD.elastic_SPH) {
        originalDerivTauXxYyZz[id] = sortedDerivTauXxYyZz[index];
        originalDerivTauXyXzYz[id] = sortedDerivTauXyXzYz[index];
        originalFreeSurfaceId[id] = sortedFreeSurfaceId[index];
    }
    return;
}

//--------------------------------------------------------------------------------------------------------------------------------
__global__ void CopySortedToOriginal_XSPH_D(const Real3* sortedXSPH,
                                            Real3* originalXSPH,
                                            const uint* gridMarkerIndex,
                                            const uint* activityIdentifierD,
                                            const uint* mapOriginalToSorted) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numObjectsD.numAllMarkers)
        return;

    // Check the activity of this particle
    uint activity = activityIdentifierD[id];
    if (activity == 0)
        return;

    uint index = mapOriginalToSorted[id];

    originalXSPH[id] = sortedXSPH[index];
}

// ===============================================================================================================================

ChFsiForceExplicitSPH::ChFsiForceExplicitSPH(std::shared_ptr<ChBce> otherBceWorker,
                                             std::shared_ptr<SphMarkerDataD> otherSortedSphMarkersD,
                                             std::shared_ptr<ProximityDataD> otherMarkersProximityD,
                                             std::shared_ptr<FsiData> otherFsiData,
                                             std::shared_ptr<SimParams> params,
                                             std::shared_ptr<ChCounters> numObjects,
                                             bool verb)
    : ChFsiForce(otherBceWorker,
                 otherSortedSphMarkersD,
                 otherMarkersProximityD,
                 otherFsiData,
                 params,
                 numObjects,
                 verb) {
    CopyParams_NumberOfObjects(paramsH, numObjectsH);
    density_initialization = 0;
}

ChFsiForceExplicitSPH::~ChFsiForceExplicitSPH() {}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::Initialize() {
    ChFsiForce::Initialize();
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), paramsH.get(), sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjectsH.get(), sizeof(ChCounters));
    hipMemcpyFromSymbol(paramsH.get(), paramsD, sizeof(SimParams));
    hipDeviceSynchronize();
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::ForceSPH(std::shared_ptr<SphMarkerDataD> otherSortedSphMarkersD,
                                     std::shared_ptr<FsiBodyStateD> fsiBodyStateD,
                                     std::shared_ptr<FsiMeshStateD> fsiMesh1DStateD,
                                     std::shared_ptr<FsiMeshStateD> fsiMesh2DStateD,
                                     Real time,
                                     bool firstHalfStep) {
    sortedSphMarkers_D = otherSortedSphMarkersD;
    bceWorker->updateBCEAcc(fsiBodyStateD, fsiMesh1DStateD, fsiMesh2DStateD);
    CollideWrapper(time, firstHalfStep);
    CalculateXSPH_velocity();
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::neighborSearch() {
    bool *isErrorH, *isErrorD;
    isErrorH = (bool*)malloc(sizeof(bool));
    hipMalloc((void**)&isErrorD, sizeof(bool));
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    // thread per particle
    uint numBlocksShort, numThreadsShort;
    computeGridSize(numObjectsH->numAllMarkers, 256, numBlocksShort, numThreadsShort);
    // Execute the kernel
    *isErrorH = false;
    hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
    thrust::fill(fsiData->numNeighborsPerPart.begin(), fsiData->numNeighborsPerPart.end(), 0);

    // start neighbor search
    // first pass
    neighborSearchNum<<<numBlocksShort, numThreadsShort>>>(
        mR4CAST(sortedSphMarkers_D->posRadD), mR4CAST(sortedSphMarkers_D->rhoPresMuD),
        U1CAST(markersProximity_D->cellStartD), U1CAST(markersProximity_D->cellEndD),
        U1CAST(fsiData->activityIdentifierD), U1CAST(fsiData->numNeighborsPerPart), isErrorD);
    ChUtilsDevice::Sync_CheckError(isErrorH, isErrorD, "neighborSearchNum");

    // in-place exclusive scan for num of neighbors
    thrust::exclusive_scan(fsiData->numNeighborsPerPart.begin(), fsiData->numNeighborsPerPart.end(),
                           fsiData->numNeighborsPerPart.begin());
    // std::cout << "numNeighbors: " << fsiData->numNeighborsPerPart.back() << std::endl;
    fsiData->neighborList.resize(fsiData->numNeighborsPerPart.back());
    thrust::fill(fsiData->neighborList.begin(), fsiData->neighborList.end(), 0);

    // second pass
    neighborSearchID<<<numBlocksShort, numThreadsShort>>>(
        mR4CAST(sortedSphMarkers_D->posRadD), mR4CAST(sortedSphMarkers_D->rhoPresMuD),
        U1CAST(markersProximity_D->cellStartD), U1CAST(markersProximity_D->cellEndD),
        U1CAST(fsiData->activityIdentifierD), U1CAST(fsiData->numNeighborsPerPart), U1CAST(fsiData->neighborList),
        isErrorD);
    ChUtilsDevice::Sync_CheckError(isErrorH, isErrorD, "neighborSearchID");
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::CollideWrapper(Real time, bool firstHalfStep) {
    bool* isErrorD;
    hipMalloc((void**)&isErrorD, sizeof(bool));

    // thread per particle
    uint numBlocks, numThreads;
    computeGridSize((int)numObjectsH->numAllMarkers, 256, numBlocks, numThreads);

    // Re-Initialize the density after several time steps if needed
    if (density_initialization >= paramsH->densityReinit) {
        thrust::device_vector<Real4> rhoPresMuD_old = sortedSphMarkers_D->rhoPresMuD;
        printf("Re-initializing density after %d steps.\n", paramsH->densityReinit);
        cudaResetErrorFlag(isErrorD);
        calcRho_kernel<<<numBlocks, numThreads>>>(
            mR4CAST(sortedSphMarkers_D->posRadD), mR4CAST(sortedSphMarkers_D->rhoPresMuD), mR4CAST(rhoPresMuD_old),
            U1CAST(fsiData->numNeighborsPerPart), U1CAST(fsiData->neighborList), density_initialization, isErrorD);
        cudaCheckErrorFlag(isErrorD, "calcRho_kernel");
        density_initialization = 0;
    }
    density_initialization++;

    // Perform Proxmity search at specified frequency
    if (firstHalfStep && (time < 1e-6 || int(round(time / paramsH->dT)) % paramsH->num_proximity_search_steps == 0))
        neighborSearch();

    thrust::device_vector<Real3> sortedKernelSupport(numObjectsH->numAllMarkers);
    // Calculate the kernel support of each particle
    cudaResetErrorFlag(isErrorD);
    calcKernelSupport<<<numBlocks, numThreads>>>(
        mR4CAST(sortedSphMarkers_D->posRadD), mR4CAST(sortedSphMarkers_D->rhoPresMuD), mR3CAST(sortedKernelSupport),
        U1CAST(markersProximity_D->mapOriginalToSorted), U1CAST(fsiData->numNeighborsPerPart),
        U1CAST(fsiData->neighborList), isErrorD);
    cudaCheckErrorFlag(isErrorD, "calcKernelSupport");

    cudaResetErrorFlag(isErrorD);
    updateBoundaryPres<<<numBlocks, numThreads>>>(
        U1CAST(fsiData->activityIdentifierD), U1CAST(fsiData->numNeighborsPerPart), U1CAST(fsiData->neighborList),
        mR4CAST(sortedSphMarkers_D->posRadD), mR3CAST(fsiData->bceAcc), mR4CAST(sortedSphMarkers_D->rhoPresMuD),
        mR3CAST(sortedSphMarkers_D->velMasD), mR3CAST(sortedSphMarkers_D->tauXxYyZzD),
        mR3CAST(sortedSphMarkers_D->tauXyXzYzD), isErrorD);
    cudaCheckErrorFlag(isErrorD, "updateBoundaryPres");

    // Execute the kernel
    if (paramsH->elastic_SPH) {  // For granular material
        // execute the kernel Navier_Stokes and Shear_Stress_Rate in one kernel
        cudaResetErrorFlag(isErrorD);
        NS_SSR<<<numBlocks, numThreads>>>(
            U1CAST(fsiData->activityIdentifierD), mR4CAST(sortedSphMarkers_D->posRadD),
            mR3CAST(sortedSphMarkers_D->velMasD), mR4CAST(sortedSphMarkers_D->rhoPresMuD),
            mR3CAST(sortedSphMarkers_D->tauXxYyZzD), mR3CAST(sortedSphMarkers_D->tauXyXzYzD),
            U1CAST(fsiData->numNeighborsPerPart), U1CAST(fsiData->neighborList), mR4CAST(fsiData->derivVelRhoD),
            mR3CAST(fsiData->derivTauXxYyZzD), mR3CAST(fsiData->derivTauXyXzYzD), mR3CAST(fsiData->vel_XSPH_D),
            mR3CAST(sortedKernelSupport), U1CAST(fsiData->freeSurfaceIdD), isErrorD);
        cudaCheckErrorFlag(isErrorD, "NS_SSR");
    } else {  // For fluid

        // Find the index which is related to the wall boundary particle
        thrust::device_vector<uint> indexOfIndex(numObjectsH->numAllMarkers);
        thrust::device_vector<uint> identityOfIndex(numObjectsH->numAllMarkers);
        calIndexOfIndex<<<numBlocks, numThreads>>>(U1CAST(indexOfIndex), U1CAST(identityOfIndex),
                                                   U1CAST(markersProximity_D->gridMarkerIndexD));
        thrust::remove_if(indexOfIndex.begin(), indexOfIndex.end(), identityOfIndex.begin(), thrust::identity<int>());

        // execute the kernel
        cudaResetErrorFlag(isErrorD);
        // TOUnderstand: Why is the blocks NumBlocks1 and threads NumThreads1?
        Navier_Stokes<<<numBlocks, numThreads>>>(
            U1CAST(indexOfIndex), mR4CAST(fsiData->derivVelRhoD), mR3CAST(fsiData->vel_XSPH_D),
            mR4CAST(sortedSphMarkers_D->posRadD), mR3CAST(sortedSphMarkers_D->velMasD),
            mR4CAST(sortedSphMarkers_D->rhoPresMuD), U1CAST(markersProximity_D->gridMarkerIndexD),
            U1CAST(fsiData->numNeighborsPerPart), U1CAST(fsiData->neighborList), isErrorD);
        cudaCheckErrorFlag(isErrorD, "Navier_Stokes");
    }

    sortedKernelSupport.clear();
    hipFree(isErrorD);
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceExplicitSPH::CalculateXSPH_velocity() {
    // Calculate vel_XSPH
    if (fsiData->vel_XSPH_D.size() != numObjectsH->numAllMarkers) {
        printf("fsiData->vel_XSPH_D.size() %zd numObjectsH->numAllMarkers %zd \n", fsiData->vel_XSPH_D.size(),
               numObjectsH->numAllMarkers);
        throw std::runtime_error(
            "Error! size error fsiData->vel_XSPH_D Thrown from "
            "CalculateXSPH_velocity!\n");
    }

    bool* isErrorD;
    hipMalloc((void**)&isErrorD, sizeof(bool));

    //------------------------------------------------------------------------
    if (!paramsH->elastic_SPH) {
        // thread per particle
        uint numBlocks, numThreads;
        computeGridSize((int)numObjectsH->numAllMarkers, 256, numBlocks, numThreads);

        thrust::fill(fsiData->vel_XSPH_D.begin(), fsiData->vel_XSPH_D.end(), mR3(0.0));

        // Find the index which is related to the wall boundary particle
        thrust::device_vector<uint> indexOfIndex(numObjectsH->numAllMarkers);
        thrust::device_vector<uint> identityOfIndex(numObjectsH->numAllMarkers);
        calIndexOfIndex<<<numBlocks, numThreads>>>(U1CAST(indexOfIndex), U1CAST(identityOfIndex),
                                                   U1CAST(markersProximity_D->gridMarkerIndexD));
        thrust::remove_if(indexOfIndex.begin(), indexOfIndex.end(), identityOfIndex.begin(), thrust::identity<int>());

        // Execute the kernel
        cudaResetErrorFlag(isErrorD);
        CalcVel_XSPH_D<<<numBlocks, numThreads>>>(
            U1CAST(indexOfIndex), mR3CAST(fsiData->vel_XSPH_D), mR4CAST(sortedSphMarkers_D->posRadD),
            mR3CAST(sortedSphMarkers_D->velMasD), mR4CAST(sortedSphMarkers_D->rhoPresMuD),
            U1CAST(markersProximity_D->gridMarkerIndexD), U1CAST(fsiData->numNeighborsPerPart),
            U1CAST(fsiData->neighborList), isErrorD);
        cudaCheckErrorFlag(isErrorD, "CalcVel_XSPH_D");
    }

    hipFree(isErrorD);
}

}  // namespace fsi
}  // namespace chrono
//================================================================================================================================
