#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================

#include <cstdio>
#include <algorithm>
#include <cstring>
#include <fstream>
#include <sstream>

#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/sort.h>

#include "hipblas.h"

#include "chrono_fsi/sph/physics/ChFsiForceI2SPH.cuh"
#include "chrono_fsi/sph/physics/ChSphGeneral.cuh"
#include "chrono_fsi/sph/math/ChFsiLinearSolverBiCGStab.h"
#include "chrono_fsi/sph/math/ChFsiLinearSolverGMRES.h"

using std::cout;
using std::cerr;
using std::endl;

namespace chrono {
namespace fsi {
namespace sph {

__device__ void BCE_Vel_Acc(int i_idx,
                            Real3& myAcc,         // output: BCE marker acceleration
                            Real3& V_prescribed,  // output: BCE marker velocity
                            Real4* sortedPosRad,
                            int4 updatePortion,
                            uint* gridMarkerIndexD,
                            Real4* qD,
                            Real3* rigid_BCEcoords_D,
                            Real3* posRigid_fsiBodies_D,
                            Real3* velRigid_fsiBodies_D,
                            Real3* omegaVelLRF_fsiBodies_D,
                            Real3* accRigid_fsiBodies_D,
                            Real3* omegaAccLRF_fsiBodies_D,
                            uint* rigid_BCEsolids_D,

                            Real3* flex1D_vel_fsi_fea_D,  // vel of fea 1d element
                            Real3* flex1D_acc_fsi_fea_D,  // acc of fea 1d element
                            Real3* flex2D_vel_fsi_fea_D,  // vel of fea 2d element
                            Real3* flex2D_acc_fsi_fea_D,  // acc of fea 2d element

                            uint2* flex1D_Nodes_D,      // segment node indices
                            uint3* flex1D_BCEsolids_D,  // association of flex BCEs with a mesh and segment
                            Real3* flex1D_BCEcoords_D,  // local coordinates of BCE markers on FEA 1-D segments
                            uint3* flex2D_Nodes_D,      // triangle node indices
                            uint3* flex2D_BCEsolids_D,  // association of flex BCEs with a mesh and face
                            Real3* flex2D_BCEcoords_D   // local coordinates of BCE markers on FEA 2-D faces
) {
    int Original_idx = gridMarkerIndexD[i_idx];

    // See if this belongs to a fixed boundary
    if (Original_idx >= updatePortion.x && Original_idx < updatePortion.y) {
        myAcc = mR3(0.0);
        V_prescribed = mR3(0.0);
        if (paramsD.Apply_BC_U)
            V_prescribed = user_BC_U(mR3(sortedPosRad[i_idx]));
    } else if (Original_idx >= updatePortion.y && Original_idx < updatePortion.z) {
        int rigidIndex = rigid_BCEsolids_D[Original_idx - updatePortion.y];

        Real4 q4 = qD[rigidIndex];
        Real3 a1, a2, a3;
        RotationMatirixFromQuaternion(a1, a2, a3, q4);
        Real3 rigidSPH_MeshPos_LRF__ = rigid_BCEcoords_D[Original_idx - updatePortion.y];

        // Real3 p_com = mR3(posRigid_fsiBodies_D[rigidIndex]);
        Real3 v_com = velRigid_fsiBodies_D[rigidIndex];
        Real3 a_com = accRigid_fsiBodies_D[rigidIndex];
        Real3 angular_v_com = omegaVelLRF_fsiBodies_D[rigidIndex];
        Real3 angular_a_com = omegaAccLRF_fsiBodies_D[rigidIndex];
        // Real3 p_rel = mR3(sortedPosRad[i_idx]) - p_com;
        Real3 omegaCrossS = cross(angular_v_com, rigidSPH_MeshPos_LRF__);
        V_prescribed = v_com + mR3(dot(a1, omegaCrossS), dot(a2, omegaCrossS), dot(a3, omegaCrossS));
        // V_prescribed = v_com + cross(angular_v_com, rigidSPH_MeshPos_LRF);

        Real3 alphaCrossS = cross(angular_a_com, rigidSPH_MeshPos_LRF__);
        Real3 alphaCrossScrossS = cross(angular_v_com, cross(angular_v_com, rigidSPH_MeshPos_LRF__));
        // myAcc = a_com + cross(angular_a_com, p_rel) + cross(angular_v_com, cross(angular_v_com,
        // rigidSPH_MeshPos_LRF__));

        myAcc = a_com + mR3(dot(a1, alphaCrossS), dot(a2, alphaCrossS), dot(a3, alphaCrossS)) +
                mR3(dot(a1, alphaCrossScrossS), dot(a2, alphaCrossScrossS), dot(a3, alphaCrossScrossS));

        // Or not, Flexible bodies for sure
    } else if (Original_idx >= updatePortion.z && Original_idx < updatePortion.w) {
        int FlexIndex = Original_idx - updatePortion.z;  // offset index for bce markers on flex bodies

        // FlexIndex iterates through both 1D and 2D ones
        if (FlexIndex < countersD.numFlexMarkers1D) {
            // 1D element case
            uint3 flex_solid = flex1D_BCEsolids_D[FlexIndex];  // associated flex mesh and segment
            // Luning TODO: do we need flex_mesh and flex_mesh_seg?
            ////uint flex_mesh = flex_solid.x;                 // index of associated mesh
            ////uint flex_mesh_seg = flex_solid.y;             // index of segment in associated mesh
            uint flex_seg = flex_solid.z;  // index of segment in global list

            uint2 seg_nodes = flex1D_Nodes_D[flex_seg];    // indices of the 2 nodes on associated segment
            Real3 A0 = flex1D_acc_fsi_fea_D[seg_nodes.x];  // (absolute) acceleration of node 0
            Real3 A1 = flex1D_acc_fsi_fea_D[seg_nodes.y];  // (absolute) acceleration of node 1

            Real3 V0 = flex1D_vel_fsi_fea_D[seg_nodes.x];  // (absolute) acceleration of node 0
            Real3 V1 = flex1D_vel_fsi_fea_D[seg_nodes.y];  // (absolute) acceleration of node 1

            Real lambda0 = flex1D_BCEcoords_D[FlexIndex].x;  // segment coordinate
            Real lambda1 = 1 - lambda0;                      // segment coordinate

            V_prescribed = V0 * lambda0 + V1 * lambda1;
            myAcc = A0 * lambda0 + A1 * lambda1;
        }
        if (FlexIndex >= countersD.numFlexMarkers1D) {
            int flex2d_index = FlexIndex - countersD.numFlexMarkers1D;

            uint3 flex_solid = flex2D_BCEsolids_D[flex2d_index];  // associated flex mesh and face
            ////uint flex_mesh = flex_solid.x;                 // index of associated mesh
            ////uint flex_mesh_tri = flex_solid.y;             // index of triangle in associated mesh
            uint flex_tri = flex_solid.z;  // index of triangle in global list

            auto tri_nodes = flex2D_Nodes_D[flex_tri];     // indices of the 3 nodes on associated face
            Real3 A0 = flex2D_acc_fsi_fea_D[tri_nodes.x];  // (absolute) acceleration of node 0
            Real3 A1 = flex2D_acc_fsi_fea_D[tri_nodes.y];  // (absolute) acceleration of node 1
            Real3 A2 = flex2D_acc_fsi_fea_D[tri_nodes.z];  // (absolute) acceleration of node 2

            Real3 V0 = flex2D_vel_fsi_fea_D[tri_nodes.x];  // (absolute) acceleration of node 0
            Real3 V1 = flex2D_vel_fsi_fea_D[tri_nodes.y];  // (absolute) acceleration of node 1
            Real3 V2 = flex2D_vel_fsi_fea_D[tri_nodes.z];  // (absolute) acceleration of node 2

            Real lambda0 = flex2D_BCEcoords_D[flex2d_index].x;  // barycentric coordinate
            Real lambda1 = flex2D_BCEcoords_D[flex2d_index].y;  // barycentric coordinate
            Real lambda2 = 1 - lambda0 - lambda1;               // barycentric coordinate

            V_prescribed = V0 * lambda0 + V1 * lambda1 + V2 * lambda2;
            myAcc = A0 * lambda0 + A1 * lambda1 + A2 * lambda2;
        }

    } else {
        printf("i_idx=%d, Original_idx:%d was not found \n\n", i_idx, Original_idx);
    }
}

//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Viscosity_correction(Real4* sortedPosRad,  // input: sorted positions
                                     Real3* sortedVelMas,
                                     Real4* sortedRhoPreMu,
                                     Real4* sortedRhoPreMu_old,
                                     Real3* sortedTauXxYyZz,
                                     Real3* sortedTauXyXzYz,
                                     Real4* sr_tau_I_mu_i,

                                     const Real* A_L,
                                     const Real3* A_G,
                                     const Real* A_f,
                                     const Real* sumWij_inv,
                                     const uint* csrColInd,
                                     const uint* numContacts,
                                     int4 updatePortion,
                                     uint* gridMarkerIndexD,

                                     size_t numAllMarkers,
                                     Real delta_t,
                                     Real gamma_y,

                                     volatile bool* error_flag) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];

    //    bool Fluid_Marker = sortedRhoPreMu_old[i_idx].w == -1.0;
    //    bool Boundary_Marker = sortedRhoPreMu_old[i_idx].w > -1.0;
    Real mu_ave = 0.0;
    Real p_ave = 0.0;
    Real3 posRadA = mR3(sortedPosRad[i_idx]);
    Real3 grad_ux = mR3(0.0), grad_uy = mR3(0.0), grad_uz = mR3(0.0);
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real d = length(rij);
        Real3 eij = rij / d;
        Real h_j = sortedPosRad[j].w;
        Real m_j = cube(h_j / paramsD.d0_multiplier) * paramsD.rho0;
        Real W3 = W3h(paramsD.kernel_type, d, paramsD.ooh);
        Real3 grad_i_wij = GradW3h(paramsD.kernel_type, rij, paramsD.ooh);

        if (sortedRhoPreMu_old[i_idx].w != -1)
            continue;
        bool fluid = sortedRhoPreMu_old[j].w == -1;
        Real3 coeff = -m_j / sortedRhoPreMu_old[j].x * grad_i_wij;
        grad_ux += coeff * (sortedVelMas[i_idx].x - sortedVelMas[j].x * fluid);
        grad_uy += coeff * (sortedVelMas[i_idx].y - sortedVelMas[j].y * fluid);
        grad_uz += coeff * (sortedVelMas[i_idx].z - sortedVelMas[j].z * fluid);

        //        grad_ux += A_G[count] * sortedVelMas[j].x;
        //        grad_uy += A_G[count] * sortedVelMas[j].y;
        //        grad_uz += A_G[count] * sortedVelMas[j].z;
        mu_ave += A_f[count] * sortedRhoPreMu_old[j].z;
        p_ave += A_f[count] * sortedRhoPreMu_old[j].y;
    }

    Real sr = Strain_Rate(grad_ux, grad_uy, grad_uz);

    Real mu_0 = Herschel_Bulkley_mu_eff(paramsD.HB_sr0, paramsD.HB_k, paramsD.HB_n, paramsD.HB_tau0);
    mu_0 = paramsD.mu_max;
    Real tau_yeild = paramsD.HB_tau0;

    if (paramsD.non_newtonian) {
        if (sr < tau_yeild / paramsD.mu_max)
            sortedRhoPreMu[i_idx].z = paramsD.mu_max;
        //        if (sr < rmaxr(gamma_y, paramsD.HB_sr0))
        //    if (sr < paramsD.HB_sr0)
        //        sortedRhoPreMu[i_idx].z = paramsD.HB_tau0 * pow(sr / paramsD.HB_sr0, 1000.0) / paramsD.HB_sr0;
        //    else if (sortedRhoPreMu_old[i_idx].x < paramsD.rho0)
        //        sortedRhoPreMu[i_idx].z = mu_ave;
        else
            sortedRhoPreMu[i_idx].z = Herschel_Bulkley_mu_eff(sr, paramsD.HB_k, paramsD.HB_n, tau_yeild);
    }
    sr_tau_I_mu_i[i_idx].x = sr;
    sr_tau_I_mu_i[i_idx].y = Sym_Tensor_Norm(sortedTauXxYyZz[i_idx], sortedTauXyXzYz[i_idx]);
}

//--------------------------------------------------------------------------------------------------------------------------------

__global__ void V_star_Predictor(Real4* sortedPosRad,  // input: sorted positions
                                 Real3* sortedVelMas,
                                 Real4* sortedRhoPreMu,
                                 Real3* sortedTauXxYyZz,
                                 Real3* sortedTauXyXzYz,
                                 Real* A_Matrix,
                                 Real3* Bi,
                                 Real3* v_old,

                                 const Real* A_L,
                                 const Real3* A_G,
                                 const Real* A_f,

                                 const Real* sumWij_inv,
                                 Real3* Normals,

                                 const uint* csrColInd,
                                 const uint* numContacts,

                                 Real4* qD,
                                 Real3* rigid_BCEcoords_D,
                                 Real3* posRigid_fsiBodies_D,
                                 Real3* velRigid_fsiBodies_D,
                                 Real3* omegaVelLRF_fsiBodies_D,
                                 Real3* accRigid_fsiBodies_D,
                                 Real3* omegaAccLRF_fsiBodies_D,
                                 uint* rigid_BCEsolids_D,

                                 Real3* flex1D_vel_fsi_fea_D,  // vel of fea 1d element
                                 Real3* flex1D_acc_fsi_fea_D,  // acc of fea 1d element
                                 Real3* flex2D_vel_fsi_fea_D,  // vel of fea 2d element
                                 Real3* flex2D_acc_fsi_fea_D,  // acc of fea 2d element

                                 size_t numFlex1D,

                                 uint2* flex1D_Nodes_D,
                                 uint3* flex1D_BCEsolids_D,
                                 Real3* flex1D_BCEcoords_D,
                                 uint3* flex2D_Nodes_D,
                                 uint3* flex2D_BCEsolids_D,
                                 Real3* flex2D_BCEcoords_D,

                                 int4 updatePortion,
                                 uint* gridMarkerIndexD,

                                 size_t numAllMarkers,
                                 Real delta_t,

                                 volatile bool* error_flag) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    // Crank-Nicolson
    Real CN = 0.5;
    Real CN2 = 0.5;

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];  //- uint(paramsD.Pressure_Constraint);

    //    if (paramsD.Pressure_Constraint) {
    //        A_Matrix[csrEndIdx + 1] = 0;
    //        A_Matrix[numContacts[numAllMarkers] + i_idx] = 0;
    //    }

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        A_Matrix[csrStartIdx] = 1;
        Bi[i_idx] = mR3(0.0);
        return;
    }

    //    Real rho0 = paramsD.rho0;
    Real rhoi = sortedRhoPreMu[i_idx].x;
    Real mu_i = sortedRhoPreMu[i_idx].z;
    Real3 grad_rho_i = mR3(0.0), grad_mu_i = mR3(0.0);
    Real3 grad_ux = mR3(0.0), grad_uy = mR3(0.0), grad_uz = mR3(0.0);
    Real3 gradP = mR3(0.0), Laplacian_u = mR3(0.0);
    Real3 posRadA = mR3(sortedPosRad[i_idx]);

    //    bool full_support = (csrEndIdx - csrStartIdx) > 0.9 * paramsD.num_neighbors;
    //    bool full_support = (rhoi >= paramsD.rho0);
    //    printf("full support nn=%d, rho_i=%f\n", csrEndIdx - csrStartIdx, rhoi);
    //    bool ON_FREE_SURFACE = (rhoi < paramsD.rho0);
    //    bool ON_FREE_SURFACE = (rhoi < 0.90 * paramsD.rho0 || csrEndIdx - csrStartIdx < 20);
    //    bool ON_FREE_SURFACE = false;

    //    bool ON_FREE_SURFACE = (csrEndIdx - csrStartIdx < 20);
    int num_fluid = 0;
    Real3 granular_source = mR3(0.0);
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        grad_rho_i += A_G[count] * sortedRhoPreMu[j].x;
        grad_mu_i += A_G[count] * sortedRhoPreMu[j].z;
        Real3 posRadB = mR3(sortedPosRad[j]);
        Real3 rij = Distance(posRadA, posRadB);
        Real d = length(rij);
        Real3 eij = rij / d;
        Real h_j = sortedPosRad[j].w;
        Real m_j = cube(h_j / paramsD.d0_multiplier) * paramsD.rho0;
        Real W3 = W3h(paramsD.kernel_type, d, paramsD.ooh);
        Real3 grad_i_wij = GradW3h(paramsD.kernel_type, rij, paramsD.ooh);

        Real3 coeff = -m_j / sortedRhoPreMu[j].x * grad_i_wij;
        grad_ux += coeff * (sortedVelMas[i_idx].x - sortedVelMas[j].x);
        grad_uy += coeff * (sortedVelMas[i_idx].y - sortedVelMas[j].y);
        grad_uz += coeff * (sortedVelMas[i_idx].z - sortedVelMas[j].z);
        //        grad_ux += A_G[count] * sortedVelMas[j].x;
        //        grad_uy += A_G[count] * sortedVelMas[j].y;
        //        grad_uz += A_G[count] * sortedVelMas[j].z;
        Laplacian_u += A_L[count] * sortedVelMas[j];
        gradP += A_G[count] * sortedRhoPreMu[j].y;
        if (sortedRhoPreMu[j].w == -1)
            num_fluid++;
    }
    bool full_support = true;  //(rhoi >= 0.2 * paramsD.rho0) && num_fluid > uint(0.1 * paramsD.num_neighbors);

    Real sr = Strain_Rate(grad_ux, grad_uy, grad_uz);
    Real I = Inertia_num(sr, sortedRhoPreMu[i_idx].x, sortedRhoPreMu[i_idx].y, paramsD.ave_diam);
    Real mu_i_p = mu_I(sr, I) * rmaxr(sortedRhoPreMu[i_idx].y, 0);
    Real3 sgn_grad_ux = mu_i_p * sgn(grad_ux), sgn_grad_uy = mu_i_p * sgn(grad_uy), sgn_grad_uz = mu_i_p * sgn(grad_uz);
    Real3 graduxT = mR3(grad_ux.x, grad_uy.x, grad_uz.x);
    Real3 graduyT = mR3(grad_ux.y, grad_uy.y, grad_uz.y);
    Real3 graduzT = mR3(grad_ux.z, grad_uy.z, grad_uz.z);

    Real3 grad_mu_dot_gradu_u = mR3(dot(grad_mu_i, grad_ux), dot(grad_mu_i, grad_uy), dot(grad_mu_i, grad_uz));
    Real3 grad_mu_dot_gradu_uT = mR3(dot(grad_mu_i, graduxT), dot(grad_mu_i, graduyT), dot(grad_mu_i, graduzT));

    if (Fluid_Marker) {  // ----- Interior

        // Navier-Stokes
        if (full_support) {
            for (int count = csrStartIdx; count < csrEndIdx; count++) {
                //                int j = csrColInd[count];
                A_Matrix[count] = -CN * mu_i * A_L[count] * full_support +  //
                                  -CN2 * paramsD.non_newtonian * dot(grad_mu_i, A_G[count]) * full_support;
            }
            A_Matrix[csrStartIdx] += rhoi / delta_t;
            Bi[i_idx] += rhoi * sortedVelMas[i_idx] / delta_t +                     // forward euler term from lhs
                         +(1 - CN) * mu_i * Laplacian_u                             // viscous term;
                         + (1 - CN2) * paramsD.non_newtonian * grad_mu_dot_gradu_u  // Non-Newtonian term
                         + paramsD.non_newtonian * grad_mu_dot_gradu_uT             // Non - Newtonian term
                         + rhoi * (paramsD.gravity + paramsD.bodyForce3);           // body force

        } else {
            A_Matrix[csrStartIdx] = 1.0;
            Bi[i_idx] = sortedVelMas[i_idx] + (paramsD.gravity + paramsD.bodyForce3) * delta_t;
        }
    }

    else if (Boundary_Marker) {  // ----- Boundary

        Real3 posRadA = mR3(sortedPosRad[i_idx]);
        Real den = 0.0;

        for (uint count = csrStartIdx + 1; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            if (sortedRhoPreMu[j].w != -1)
                continue;
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real W3 = W3h(paramsD.kernel_type, length(rij), paramsD.ooh);
            A_Matrix[count] = W3;
            // A_Matrix[count] = A_f[count];
            den += W3;
        }

        Real3 myAcc = mR3(0);
        Real3 V_prescribed = mR3(0);

        BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, qD, rigid_BCEcoords_D,
                    posRigid_fsiBodies_D, velRigid_fsiBodies_D, omegaVelLRF_fsiBodies_D, accRigid_fsiBodies_D,
                    omegaAccLRF_fsiBodies_D, rigid_BCEsolids_D, flex1D_vel_fsi_fea_D, flex1D_acc_fsi_fea_D,
                    flex2D_vel_fsi_fea_D, flex2D_acc_fsi_fea_D, flex1D_Nodes_D, flex1D_BCEsolids_D, flex1D_BCEcoords_D,
                    flex2D_Nodes_D, flex2D_BCEsolids_D, flex2D_BCEcoords_D);

        if (den < EPSILON) {
            A_Matrix[csrStartIdx] = 1.0;
            Bi[i_idx] = V_prescribed;
        } else {
            A_Matrix[csrStartIdx] = den;
            Bi[i_idx] = 2 * V_prescribed * den;
        }
    }

    //    v_old[i_idx] = sortedVelMas[i_idx];

    if (abs(A_Matrix[csrStartIdx]) < EPSILON)
        printf("V_star_Predictor %d A_Matrix[csrStartIdx]= %f, type=%f \n", i_idx, A_Matrix[csrStartIdx],
               sortedRhoPreMu[i_idx].w);
}

//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Pressure_Equation(Real4* sortedPosRad,  // input: sorted positions
                                  Real3* sortedVelMas,
                                  Real4* sortedRhoPreMu,
                                  Real* A_Matrix,
                                  Real* Bi,
                                  Real3* Vstar,
                                  Real* q_new,

                                  const Real* A_f,
                                  const Real* A_L,
                                  const Real3* A_G,
                                  const Real* sumWij_inv,
                                  Real3* Normals,
                                  uint* csrColInd,
                                  const uint* numContacts,

                                  Real4* qD,
                                  Real3* rigid_BCEcoords_D,
                                  Real3* posRigid_fsiBodies_D,
                                  Real3* velRigid_fsiBodies_D,
                                  Real3* omegaVelLRF_fsiBodies_D,
                                  Real3* accRigid_fsiBodies_D,
                                  Real3* omegaAccLRF_fsiBodies_D,
                                  uint* rigid_BCEsolids_D,

                                  Real3* flex1D_vel_fsi_fea_D,
                                  Real3* flex1D_acc_fsi_fea_D,
                                  Real3* flex2D_vel_fsi_fea_D,
                                  Real3* flex2D_acc_fsi_fea_D,

                                  size_t numFlex1D,
                                  uint2* flex1D_Nodes_D,
                                  uint3* flex1D_BCEsolids_D,
                                  Real3* flex1D_BCEcoords_D,
                                  uint3* flex2D_Nodes_D,
                                  uint3* flex2D_BCEsolids_D,
                                  Real3* flex2D_BCEcoords_D,

                                  int4 updatePortion,
                                  uint* gridMarkerIndexD,
                                  size_t numAllMarkers,
                                  size_t numFluidMarkers,
                                  Real delta_t,
                                  volatile bool* error_flag) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];  //- uint(paramsD.Pressure_Constraint);

    bool Fluid_Marker = sortedRhoPreMu[i_idx].w == -1.0;
    bool Boundary_Marker = sortedRhoPreMu[i_idx].w > -1.0;

    if (sortedRhoPreMu[i_idx].w <= -2) {
        A_Matrix[csrStartIdx] = 1.0;
        Bi[i_idx] = 0.0;
        return;
    }

    Real rhoi = sortedRhoPreMu[i_idx].x;
    Real TIME_SCALE = paramsD.DensityBaseProjection ? (delta_t * delta_t) : delta_t;
    //    Real TIME_SCALE = 1.0;

    //    bool ON_FREE_SURFACE = (rhoi < paramsD.rho0 || csrEndIdx - csrStartIdx < paramsD.num_neighbors * 0.5);
    bool full_support = (rhoi >= 0.8 * paramsD.rho0) && (csrEndIdx - csrStartIdx) > uint(0.2 * paramsD.num_neighbors);

    //    bool ON_FREE_SURFACE = (rhoi < paramsD.rho0);
    //    Real rho0 = paramsD.rho0;
    Real3 body_force = paramsD.gravity + paramsD.bodyForce3;
    Real3 grad_rho_i = mR3(0.0);
    Real div_vi_star = 0;
    Real div_vi = 0;

    // Calculating the div.v* and grad(rho)
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        int j = csrColInd[count];
        div_vi_star += dot(A_G[count], Vstar[j]);
        div_vi += dot(A_G[count], sortedVelMas[j]);
        grad_rho_i += A_G[count] * sortedRhoPreMu[j].x;
    }

    Real rhoi_star = rhoi - paramsD.rho0 * div_vi_star * delta_t;
    //    Real rhoi_star = rhoi + dot(grad_rho_i, Vstar[i_idx] * delta_t);

    if (Fluid_Marker) {  // ----- Interior

        if (full_support || !paramsD.Conservative_Form) {
            for (int count = csrStartIdx; count < csrEndIdx; count++) {
                // Note that including the second term creates problems with density based projection
                A_Matrix[count] = 1 / rhoi * A_L[count] - 1.0 / (rhoi * rhoi) * dot(grad_rho_i, A_G[count]);
            }

            Real alpha = paramsD.Alpha;  // square(rhoi / paramsD.rho0);
            //            alpha = (alpha > 1) ? 1.0 : alpha;
            if (paramsD.DensityBaseProjection)
                Bi[i_idx] = alpha * (paramsD.rho0 - rhoi_star) / paramsD.rho0 * (TIME_SCALE / (delta_t * delta_t)) +
                            +0 * (1 - alpha) * div_vi_star * (TIME_SCALE / delta_t);
            else
                Bi[i_idx] = div_vi_star * (TIME_SCALE / delta_t);
        } else {
            //            clearRow(i_idx, csrStartIdx, csrEndIdx, A_Matrix, Bi);
            //            for (int count = csrStartIdx; count < csrEndIdx; count++) {
            //                A_Matrix[count] = A_f[count];
            //            }
            Bi[i_idx] = paramsD.base_pressure;
            A_Matrix[csrStartIdx] = 1.0;
        }

    } else if (Boundary_Marker) {  // ----- Boundary Adami

        Real3 posRadA = mR3(sortedPosRad[i_idx]);
        Real3 myAcc = mR3(0);
        Real3 V_prescribed = mR3(0);
        BCE_Vel_Acc(i_idx, myAcc, V_prescribed, sortedPosRad, updatePortion, gridMarkerIndexD, qD, rigid_BCEcoords_D,
                    posRigid_fsiBodies_D, velRigid_fsiBodies_D, omegaVelLRF_fsiBodies_D, accRigid_fsiBodies_D,
                    omegaAccLRF_fsiBodies_D, rigid_BCEsolids_D, flex1D_vel_fsi_fea_D, flex1D_acc_fsi_fea_D,
                    flex2D_vel_fsi_fea_D, flex2D_acc_fsi_fea_D, flex1D_Nodes_D, flex1D_BCEsolids_D, flex1D_BCEcoords_D,
                    flex2D_Nodes_D, flex2D_BCEsolids_D, flex2D_BCEcoords_D);

        Real pRHS = 0.0;
        Real den = 0.0;

        for (int count = csrStartIdx; count < csrEndIdx; count++) {
            uint j = csrColInd[count];
            if (sortedRhoPreMu[j].w != -1.0)
                continue;
            Real3 posRadB = mR3(sortedPosRad[j]);
            Real3 rij = Distance(posRadA, posRadB);
            Real W3 = W3h(paramsD.kernel_type, length(rij), paramsD.ooh);
            // fluid pressures are actually p*TIME_SCALE, so divide by TIME_SCALE to get the actual formula
            A_Matrix[count] = -W3;
            // pressure of the boundary marker should be calculated as p*TIME_SCALE
            // so divide its multiplier by TIME_SCALE
            Real rho_bar = 0.5 * (sortedRhoPreMu[i_idx].x + sortedRhoPreMu[j].x);

            den += W3;
            pRHS += dot(body_force - myAcc, rij) * rho_bar * W3;
        }

        if (abs(den) > EPSILON) {
            A_Matrix[csrStartIdx] = den;
            Bi[i_idx] = pRHS;
            // Scale to make the diagonal element 1

        } else {
            A_Matrix[csrStartIdx] = 1.0;
            Bi[i_idx] = paramsD.base_pressure;
        }

        q_new[i_idx] = sortedRhoPreMu[i_idx].y * TIME_SCALE;
        Bi[i_idx] *= TIME_SCALE;
    }

    //    if (paramsD.Pressure_Constraint) {
    //        A_Matrix[csrEndIdx] = (double)Fluid_Marker / (double)numFluidMarkers;
    //        csrColInd[csrEndIdx] = numAllMarkers;
    //        uint last_row_start = numContacts[numAllMarkers];
    //        A_Matrix[last_row_start + i_idx] = (double)Fluid_Marker / (double)numFluidMarkers;
    //        csrColInd[last_row_start + i_idx] = i_idx;
    //    }

    //    // Diagonal preconditioner
    //    if (abs(A_Matrix[csrStartIdx]) > EPSILON) {
    //        for (int count = csrStartIdx + 1; count < csrEndIdx; count++)
    //            A_Matrix[count] /= A_Matrix[csrStartIdx];
    //        Bi[i_idx] /= A_Matrix[csrStartIdx];
    //        A_Matrix[csrStartIdx] = 1.0;
    //    }

    //    if (sortedRhoPreMu[i_idx].w > -1)
    //    A_Matrix[csrStartIdx] = 1.0 + paramsD.epsMinMarkersDis;

    //    if (abs(A_Matrix[csrStartIdx]) < EPSILON)
    //        printf("Pressure_Equation %d A_Matrix[csrStartIdx]= %f, type=%f \n", i_idx, A_Matrix[csrStartIdx],
    //               sortedRhoPreMu[i_idx].w);
}

//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Velocity_Correction_and_update(Real4* sortedPosRad,
                                               Real4* sortedPosRad_old,

                                               Real4* sortedRhoPreMu,
                                               Real4* sortedRhoPreMu_old,

                                               Real3* sortedVelMas,
                                               Real3* sortedVelMas_old,

                                               Real3* sortedTauXxYyZz,
                                               Real3* sortedTauXyXzYz,

                                               Real4* sr_tau_I_mu_i,

                                               Real3* sortedVisVel,
                                               Real4* derivVelRho,

                                               Real3* Vstar,
                                               Real* q_i,  // q=p^(n+1)-p^n

                                               const Real* A_f,
                                               const Real3* A_G,
                                               const Real* A_L,
                                               const uint* csrColInd,
                                               const uint* numContacts,
                                               size_t numAllMarkers,
                                               const Real MaxVel,
                                               Real delta_t,
                                               volatile bool* error_flag) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }
    // Note that every variable that is used inside the for loops should not be overwritten later otherwise there would
    // be a race condition. For such variables one must use the old values.
    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];  //- uint(paramsD.Pressure_Constraint);
    //    Real m_i = cube(sortedPosRad_old[i_idx].w / paramsD.d0_multiplier) * paramsD.rho0;
    Real m_i = paramsD.markerMass;
    Real TIME_SCALE = paramsD.DensityBaseProjection ? (delta_t * delta_t) : delta_t;
    //    Real TIME_SCALE = 1.0;

    Real3 grad_p_nPlus1 = mR3(0.0);

    Real divV_star = 0;
    Real rho_i = sortedRhoPreMu_old[i_idx].x;
    Real3 posA = mR3(sortedPosRad_old[i_idx]);
    Real3 grad_q_i_conservative = mR3(0.0), grad_q_i_consistent = mR3(0.0), laplacian_V = mR3(0.0);
    Real3 grad_ux = mR3(0.0), grad_uy = mR3(0.0), grad_uz = mR3(0.0);

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        //        Real m_j = cube(sortedPosRad_old[j].w / paramsD.d0_multiplier) * paramsD.rho0;
        Real m_j = paramsD.markerMass;
        Real rho_j = sortedRhoPreMu_old[j].x;
        Real3 rij = Distance(posA, mR3(sortedPosRad_old[j]));
        Real3 gradW = GradW3h(paramsD.kernel_type, rij, paramsD.ooh);
        bool fluid_j = sortedRhoPreMu_old[j].w == -1;
        bool fluid_i = sortedRhoPreMu_old[i_idx].w == -1;

        if (fluid_i || fluid_j) {
            Real3 coeff = -m_j / sortedRhoPreMu_old[j].x * gradW;
            grad_ux += coeff * (Vstar[i_idx].x * fluid_i - Vstar[j].x * fluid_j);
            grad_uy += coeff * (Vstar[i_idx].y * fluid_i - Vstar[j].y * fluid_j);
            grad_uz += coeff * (Vstar[i_idx].z * fluid_i - Vstar[j].z * fluid_j);
            //            grad_ux += A_G[count] * Vstar[j].x;
            //            grad_uy += A_G[count] * Vstar[j].y;
            //            grad_uz += A_G[count] * Vstar[j].z;
        }

        // No forces for BCE to BCE markers
        if (sortedRhoPreMu_old[j].w > -1 && sortedRhoPreMu_old[i_idx].w > -1)
            continue;

        grad_q_i_conservative += m_j * rho_i * (q_i[i_idx] / (rho_i * rho_i) + q_i[j] / (rho_j * rho_j)) * gradW;
        grad_q_i_consistent += A_G[count] * q_i[j];
        divV_star += dot(A_G[count], Vstar[j]);
        grad_p_nPlus1 += A_G[count] * (sortedRhoPreMu_old[j].y + q_i[j] / TIME_SCALE);
        laplacian_V += A_L[count] * sortedVelMas_old[j];
    }

    Real3 grad_q_i = (paramsD.Conservative_Form ? grad_q_i_conservative : grad_q_i_consistent) / TIME_SCALE;
    Real3 Pressure_correction_term = -grad_q_i * (delta_t) / paramsD.rho0;

    //    if (rho_i < paramsD.rho0)
    //        Pressure_correction_term = mR3(0);

    //    if (!(isfinite(q_i[i_idx]))) {
    //        printf("Error! particle %d q_i is NAN: thrown from ChFsiForceI2SPH.cu  %f\n", i_idx, q_i[i_idx]);
    //    }
    Real3 V_new = Vstar[i_idx] + Pressure_correction_term;
    //    if (sortedRhoPreMu[i_idx].w == -1)
    //        printf("Vstar=%f, Pressure_correction_term=%f\n", Vstar[i_idx], Pressure_correction_term);

    //    Real4 x_new = sortedPosRad[i_idx] + mR4(delta_t / 2 * (V_new + sortedVelMas_old[i_idx]), 0.0);
    Real4 x_new = sortedPosRad_old[i_idx] + mR4(delta_t * (V_new), 0.0);

    sortedVelMas[i_idx] = V_new;
    //    sortedRhoPreMu[i_idx].x = sortedRhoPreMu_old[i_idx].x - delta_t * sortedRhoPreMu_old[i_idx].x * divV_star;
    Real mu_i = sortedRhoPreMu_old[i_idx].z;

    Real3 FS_force = (-grad_q_i_conservative / TIME_SCALE + laplacian_V * mu_i + paramsD.bodyForce3 + paramsD.gravity) *
                     m_i / sortedRhoPreMu_old[i_idx].x;
    derivVelRho[i_idx] = mR4(FS_force, 0.0);

    //    if (sortedRhoPreMu[i_idx].w > 0 && length(derivVelRho[i_idx]) > 0)
    //        printf("%f\t", length(derivVelRho[i_idx]));

    //    Real3 m_dv_dt = m_i * (V_new - sortedVelMas_old[i_idx]) / delta_t;
    //    derivVelRho[i_idx] = mR4(-m_dv_dt, 0);

    sortedRhoPreMu[i_idx].y = (q_i[i_idx]) / TIME_SCALE;

    Real3 updatedTauXxYyZz = sortedTauXxYyZz[i_idx];
    Real3 updatedTauXyXzYz = sortedTauXyXzYz[i_idx];

    Real tau_norm = Sym_Tensor_Norm(updatedTauXxYyZz, updatedTauXyXzYz);
    ////Real yeild_tau = sr_tau_I_mu_i[i_idx].w * rmaxr(sortedRhoPreMu_old[i_idx].y, 0);
    sr_tau_I_mu_i[i_idx].y = Sym_Tensor_Norm(sortedTauXxYyZz[i_idx], sortedTauXyXzYz[i_idx]);

    if (sortedRhoPreMu_old[i_idx].w == -1.0) {
        sortedPosRad[i_idx] = x_new;
    }

    //    if (!IsFinite(sortedPosRad[i_idx])) {
    //        printf("Error! particle %d position is NAN: thrown from ChFsiForceI2SPH.cu  %f,%f,%f,%f\n", i_idx,
    //               sortedPosRad[i_idx].x, sortedPosRad[i_idx].y, sortedPosRad[i_idx].z, sortedPosRad[i_idx].w);
    //    }
    //    if (!IsFinite(sortedRhoPreMu[i_idx])) {
    //        printf("Error! particle %d rhoPreMu is NAN: thrown from ChFsiForceI2SPH.cu %f,%f,%f,%f\n", i_idx,
    //               sortedRhoPreMu[i_idx].x, sortedRhoPreMu[i_idx].y, sortedRhoPreMu[i_idx].z,
    //               sortedRhoPreMu[i_idx].w);
    //    }
    //
    //    if (!IsFinite(sortedVelMas[i_idx])) {
    //        printf("Error! particle %d velocity is NAN: thrown from ChFsiForceI2SPH.cu %f,%f,%f\n", i_idx,
    //               sortedVelMas[i_idx].x, sortedVelMas[i_idx].y, sortedVelMas[i_idx].z);
    //    }
}

//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Shifting(Real4* sortedPosRad,
                         Real4* sortedPosRad_old,
                         Real4* sortedRhoPreMu,
                         Real4* sortedRhoPreMu_old,
                         Real3* sortedVelMas,
                         Real3* sortedVelMas_old,
                         Real3* sortedVisVel,
                         const Real* A_f,
                         const Real3* A_G,
                         const uint* csrColInd,
                         const uint* numContacts,
                         size_t numAllMarkers,
                         const Real MaxVel,
                         Real delta_t,
                         volatile bool* error_flag) {
    uint i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx >= numAllMarkers) {
        return;
    }

    uint csrStartIdx = numContacts[i_idx];
    uint csrEndIdx = numContacts[i_idx + 1];  //- uint(paramsD.Pressure_Constraint);
    Real3 inner_sum = mR3(0.0), shift_r = mR3(0.0);
    Real mi_bar = 0.0, r0 = 0.0;  // v_bar = 0.0;
    Real3 xSPH_Sum = mR3(0.0);
    if (!IsFinite(sortedPosRad_old[i_idx])) {
        printf("Error! particle %d position is NAN: thrown from 1 Shifting ChFsiForceI2SPH.cu  %f,%f,%f,%f\n", i_idx,
               sortedPosRad_old[i_idx].x, sortedPosRad_old[i_idx].y, sortedPosRad_old[i_idx].z,
               sortedPosRad_old[i_idx].w);
    }
    if (!IsFinite(sortedRhoPreMu_old[i_idx])) {
        printf("Error! particle %d rhoPreMu is NAN: thrown from 1 Shifting ChFsiForceI2SPH.cu %f,%f,%f,%f\n", i_idx,
               sortedRhoPreMu_old[i_idx].x, sortedRhoPreMu_old[i_idx].y, sortedRhoPreMu_old[i_idx].z,
               sortedRhoPreMu_old[i_idx].w);
    }

    if (!IsFinite(sortedVelMas_old[i_idx])) {
        printf("Error! particle %d velocity is NAN: thrown from 1 Shifting ChFsiForceI2SPH.cu %f,%f,%f\n", i_idx,
               sortedVelMas_old[i_idx].x, sortedVelMas_old[i_idx].y, sortedVelMas_old[i_idx].z);
    }
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        Real3 rij = Distance(mR3(sortedPosRad_old[i_idx]), mR3(sortedPosRad_old[j]));
        Real d = length(rij);
        Real m_j = paramsD.markerMass;

        if (sortedRhoPreMu_old[j].w == -1.0) {
            Real Wd = W3h(paramsD.kernel_type, d, paramsD.ooh);
            Real rho_bar = 0.5 * (sortedRhoPreMu_old[i_idx].x + sortedRhoPreMu_old[j].x);
            xSPH_Sum += (sortedVelMas_old[j] - sortedVelMas_old[i_idx]) * Wd * m_j / rho_bar;
        }
        //        v_bar += length(A_f[count] * (sortedVelMas_old[j]));
        //        Real m_j = cube(sortedPosRad_old[j].w / paramsD.d0_multiplier) * paramsD.rho0;

        mi_bar += m_j;
        r0 += d;
        if (d > 0)
            //        inner_sum += rij / (d * d * d);
            inner_sum += m_j * rij / (d * d * d);
    }

    if (sortedRhoPreMu_old[i_idx].w == -1.0) {
        r0 /= (csrEndIdx - csrStartIdx + 1);
        mi_bar /= (csrEndIdx - csrStartIdx + 1);
    }

    if (abs(mi_bar) > EPSILON)
        shift_r = paramsD.beta_shifting * r0 * r0 * length(MaxVel) * delta_t * inner_sum / mi_bar;
    //    shift_r = paramsD.beta_shifting * r0 * r0 * length(MaxVel) * delta_t * inner_sum;

    Real3 grad_p = mR3(0.0);
    Real3 grad_rho = mR3(0.0);
    Real3 grad_ux = mR3(0.0);
    Real3 grad_uy = mR3(0.0);
    Real3 grad_uz = mR3(0.0);
    Real p_smooth = 0, rho_smooth = 0;

    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        grad_p += A_G[count] * sortedRhoPreMu_old[j].y;
        grad_rho += A_G[count] * sortedRhoPreMu_old[j].x;
        grad_ux += A_G[count] * sortedVelMas_old[j].x;
        grad_uy += A_G[count] * sortedVelMas_old[j].y;
        grad_uz += A_G[count] * sortedVelMas_old[j].z;
        p_smooth += A_f[count] * (sortedRhoPreMu_old[j].y);
        rho_smooth += A_f[count] * (sortedRhoPreMu_old[j].x);
    }

    if (sortedRhoPreMu_old[i_idx].w == -1.0) {
        sortedPosRad[i_idx] += mR4(shift_r, 0.0);
        sortedRhoPreMu[i_idx].y += dot(shift_r, grad_p);
        sortedRhoPreMu[i_idx].x += dot(shift_r, grad_rho);
        sortedVelMas[i_idx].x += dot(shift_r, grad_ux);
        sortedVelMas[i_idx].y += dot(shift_r, grad_uy);
        sortedVelMas[i_idx].z += dot(shift_r, grad_uz);
        sortedVelMas[i_idx] += paramsD.EPS_XSPH * xSPH_Sum;
        //        sortedPosRad[i_idx] += mR4(paramsD.EPS_XSPH * xSPH_Sum * delta_t, 0.0);
    }

    Real3 vis_vel = mR3(0.0);
    for (int count = csrStartIdx; count < csrEndIdx; count++) {
        uint j = csrColInd[count];
        if (sortedRhoPreMu_old[j].w == -1.0)
            vis_vel += A_f[count] * (sortedVelMas_old[j]);
    }

    sortedVisVel[i_idx] = vis_vel;

    if (!IsFinite(sortedPosRad[i_idx])) {
        printf("Error! particle %d position is NAN: thrown from Shifting ChFsiForceI2SPH.cu  %f,%f,%f,%f\n", i_idx,
               sortedPosRad[i_idx].x, sortedPosRad[i_idx].y, sortedPosRad[i_idx].z, sortedPosRad[i_idx].w);
    }
    if (!IsFinite(sortedRhoPreMu[i_idx])) {
        printf("Error! particle %d rhoPreMu is NAN: thrown from Shifting ChFsiForceI2SPH.cu %f,%f,%f,%f\n", i_idx,
               sortedRhoPreMu[i_idx].x, sortedRhoPreMu[i_idx].y, sortedRhoPreMu[i_idx].z, sortedRhoPreMu[i_idx].w);
    }

    if (!IsFinite(sortedVelMas[i_idx])) {
        printf("Error! particle %d velocity is NAN: thrown from Shifting ChFsiForceI2SPH.cu %f,%f,%f\n", i_idx,
               sortedVelMas[i_idx].x, sortedVelMas[i_idx].y, sortedVelMas[i_idx].z);
    }
}

//--------------------------------------------------------------------------------------------------------------------------------

ChFsiForceI2SPH::ChFsiForceI2SPH(FsiDataManager& data_mgr, BceManager& bce_mgr, bool verbose)
    : ChFsiForce(data_mgr, bce_mgr, verbose) {
    CopyParametersToDevice(m_data_mgr.paramsH, m_data_mgr.countersH);
}

ChFsiForceI2SPH::~ChFsiForceI2SPH() {
    cudaFreeErrorFlag(error_flagD);
}

//--------------------------------------------------------------------------------------------------------------------------------

void ChFsiForceI2SPH::Initialize() {
    ChFsiForce::Initialize();

    // Create linear solver object
    switch (m_data_mgr.paramsH->LinearSolver) {
        case SolverType::BICGSTAB:
            myLinearSolver = chrono_types::make_shared<ChFsiLinearSolverBiCGStab>();
            break;
        case SolverType::GMRES:
            myLinearSolver = chrono_types::make_shared<ChFsiLinearSolverGMRES>();
            break;
        case SolverType::JACOBI:
            break;
        default:
            std::cout << "The ChFsiLinearSolver you chose has not been implemented, reverting to JACOBI";
            break;
    }

    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), m_data_mgr.paramsH.get(), sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(countersD), m_data_mgr.countersH.get(), sizeof(Counters));

    int numAllMarkers = (int)m_data_mgr.countersH->numAllMarkers;
    _sumWij_inv.resize(numAllMarkers);
    Normals.resize(numAllMarkers);
    G_i.resize(numAllMarkers * 9);
    A_i.resize(numAllMarkers * 27);
    L_i.resize(numAllMarkers * 6);
    // Contact_i.resize(numAllMarkers);
    V_star_new.resize(numAllMarkers);
    V_star_old.resize(numAllMarkers);
    q_new.resize(numAllMarkers);
    q_old.resize(numAllMarkers);
    b1Vector.resize(numAllMarkers);
    b3Vector.resize(numAllMarkers);
    Residuals.resize(numAllMarkers);

    cudaMallocErrorFlag(error_flagD);
}

//--------------------------------------------------------------------------------------------------------------------------------

struct my_Functor {
    Real ave;
    my_Functor(Real s) { ave = s; }
    __host__ __device__ void operator()(Real& i) { i -= ave; }
};

struct my_Functor_real4y {
    Real ave;
    my_Functor_real4y(Real s) { ave = s; }
    __host__ __device__ void operator()(Real4& i) { i.y -= ave; }
};

void ChFsiForceI2SPH::ForceSPH(std::shared_ptr<SphMarkerDataD> sortedSphMarkers_D, Real time, bool firstHalfStep) {
    // Readability replacements
    auto& pH = m_data_mgr.paramsH;
    auto& cH = m_data_mgr.countersH;

    cudaResetErrorFlag(error_flagD);

    m_sortedSphMarkers_D = sortedSphMarkers_D;

    thrust::device_vector<Real3>::iterator iter = thrust::max_element(
        m_sortedSphMarkers_D->velMasD.begin(), m_sortedSphMarkers_D->velMasD.end(), compare_Real3_mag());
    Real MaxVel = length(*iter);

    thrust::device_vector<Real4>::iterator iter_mu = thrust::max_element(
        m_sortedSphMarkers_D->rhoPresMuD.begin(), m_sortedSphMarkers_D->rhoPresMuD.end(), compare_Real4_z());
    Real Maxmu = length(*iter_mu);

    size_t end_fluid = cH->numGhostMarkers + cH->numHelperMarkers + cH->numFluidMarkers;
    size_t end_bndry = end_fluid + cH->numBoundaryMarkers;
    size_t end_rigid = end_bndry + cH->numRigidMarkers;
    size_t end_flex = end_rigid + cH->numFlexMarkers1D + cH->numFlexMarkers2D;
    int4 updatePortion = mI4((int)end_fluid, (int)end_bndry, (int)end_rigid, (int)end_flex);

    if (m_verbose)
        cout << "update portion: " << updatePortion.x << " " << updatePortion.y << " " << updatePortion.z << " "
             << updatePortion.w << endl;

    uint numThreads, numBlocks;
    computeGridSize((int)numAllMarkers + 1, 256, numBlocks, numThreads);

    // TODO: shall i include the proximity search freq?
    neighborSearch();

    // ----- calcRho_kernel=== calc_A_tensor==calc_L_tensor==Function_Gradient_Laplacian_Operator
    PreProcessor(true);

    // ------

    thrust::device_vector<Real4> rhoPresMuD_old = m_sortedSphMarkers_D->rhoPresMuD;
    thrust::device_vector<Real4> posRadD_old = m_sortedSphMarkers_D->posRadD;
    thrust::device_vector<Real3> velMasD_old = m_sortedSphMarkers_D->velMasD;

    thrust::fill(V_star_old.begin(), V_star_old.end(), mR3(0.0));
    thrust::fill(V_star_new.begin(), V_star_new.end(), mR3(0.0));
    thrust::fill(b3Vector.begin(), b3Vector.end(), mR3(0.0));
    thrust::fill(Residuals.begin(), Residuals.end(), 0.0);
    Real yeild_strain = MaxVel / pH->h * 0.05;

    if (pH->non_newtonian) {
        Viscosity_correction<<<numBlocks, numThreads>>>(
            mR4CAST(m_sortedSphMarkers_D->posRadD), mR3CAST(m_sortedSphMarkers_D->velMasD),
            mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), mR4CAST(rhoPresMuD_old),
            mR3CAST(m_sortedSphMarkers_D->tauXxYyZzD), mR3CAST(m_sortedSphMarkers_D->tauXyXzYzD),
            mR4CAST(m_data_mgr.sr_tau_I_mu_i), R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(csrValFunction),
            R1CAST(_sumWij_inv), U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart), updatePortion,
            U1CAST(m_data_mgr.markersProximity_D->gridMarkerIndexD), numAllMarkers, pH->dT, yeild_strain, error_flagD);
        cudaCheckErrorFlag(error_flagD, "Viscosity_correction");
    }

    // ----- V_star_Predictor

    double LinearSystemClock_V = clock();
    V_star_Predictor<<<numBlocks, numThreads>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR3CAST(m_sortedSphMarkers_D->velMasD),
        mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), mR3CAST(m_sortedSphMarkers_D->tauXxYyZzD),
        mR3CAST(m_sortedSphMarkers_D->tauXyXzYzD), R1CAST(AMatrix), mR3CAST(b3Vector), mR3CAST(V_star_old),
        R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(csrValFunction), R1CAST(_sumWij_inv), mR3CAST(Normals),
        U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart),

        mR4CAST(m_data_mgr.fsiBodyState_D->rot), mR3CAST(m_data_mgr.rigid_BCEcoords_D),
        mR3CAST(m_data_mgr.fsiBodyState_D->pos), mR3CAST(m_data_mgr.fsiBodyState_D->lin_vel),
        mR3CAST(m_data_mgr.fsiBodyState_D->ang_vel), mR3CAST(m_data_mgr.fsiBodyState_D->lin_acc),
        mR3CAST(m_data_mgr.fsiBodyState_D->ang_acc), U1CAST(m_data_mgr.rigid_BCEsolids_D),

        mR3CAST(m_data_mgr.fsiMesh1DState_D->vel_fsi_fea_D), mR3CAST(m_data_mgr.fsiMesh1DState_D->acc_fsi_fea_D),
        mR3CAST(m_data_mgr.fsiMesh2DState_D->vel_fsi_fea_D), mR3CAST(m_data_mgr.fsiMesh2DState_D->acc_fsi_fea_D),

        cH->numFsiElements1D,

        U2CAST(m_data_mgr.flex1D_Nodes_D), U3CAST(m_data_mgr.flex1D_BCEsolids_D),
        mR3CAST(m_data_mgr.flex1D_BCEcoords_D), U3CAST(m_data_mgr.flex2D_Nodes_D),
        U3CAST(m_data_mgr.flex2D_BCEsolids_D), mR3CAST(m_data_mgr.flex2D_BCEcoords_D),

        updatePortion, U1CAST(m_data_mgr.markersProximity_D->gridMarkerIndexD), numAllMarkers, pH->dT, error_flagD);
    cudaCheckErrorFlag(error_flagD, "V_star_Predictor");

    int Iteration = 0;
    Real MaxRes = 100;
    while ((MaxRes > 1e-10 || Iteration < 3) && Iteration < pH->LinearSolver_Max_Iter) {
        Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(
            mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), R1CAST(AMatrix), mR3CAST(V_star_old), mR3CAST(V_star_new),
            mR3CAST(b3Vector), R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(m_data_mgr.neighborList),
            U1CAST(m_data_mgr.numNeighborsPerPart), true, error_flagD);
        cudaCheckErrorFlag(error_flagD, "Jacobi_SOR_Iter");

        Update_AND_Calc_Res<<<numBlocks, numThreads>>>(mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), mR3CAST(V_star_old),
                                                       mR3CAST(V_star_new), R1CAST(q_old), R1CAST(q_new),
                                                       R1CAST(Residuals), true, error_flagD);
        cudaCheckErrorFlag(error_flagD, "Update_AND_Calc_Res");

        Iteration++;
        thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
        auto position = iter - Residuals.begin();
        MaxRes = *iter;
        if (pH->Verbose_monitoring)
            printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
    }
    //
    //    thrust::device_vector<Real3>::iterator iter =
    //        thrust::max_element(V_star_new.begin(), V_star_new.end(), compare_Real3_mag());
    //    unsigned int position = iter - V_star_new.begin();
    //    Real MaxVel = length(*iter);
    //
    //    uint FixedMarker = 0;
    double V_star_Predictor = (clock() - LinearSystemClock_V) / (double)CLOCKS_PER_SEC;
    printf("| V_star_Predictor Equation: %f (sec) - Final Residual=%.3e - #Iter=%d\n", V_star_Predictor, MaxRes,
           Iteration);

    // ----- Pressure_Equation
    Iteration = 0;
    MaxRes = 100;
    double LinearSystemClock_p = clock();

    //    if (pH->Pressure_Constraint) {
    //        AMatrix.resize(NNZ + numAllMarkers + 1);
    //        csrColInd.resize(NNZ + numAllMarkers + 1);
    //        b1Vector.resize(numAllMarkers + 1);
    //        Contact_i.push_back(Contact_i[numAllMarkers] + numAllMarkers + 1);
    //        q_old.resize(numAllMarkers + 1);
    //        q_new.resize(numAllMarkers + 1);
    //        int Contact_i_last = Contact_i[numAllMarkers + 1];
    //        AMatrix[Contact_i[numAllMarkers + 1] - 1] = 0;
    //        csrColInd[Contact_i[numAllMarkers + 1] - 1] = numAllMarkers;
    //        //        printf(" NNZ=%d, NNZ_new=%d, Contact_i.size()=%d, csrColInd.size()=%d, Contact_i_last=%d\n",
    //        NNZ,
    //        //               AMatrix.size(), Contact_i.size(), csrColInd.size(), Contact_i_last);
    //    }

    Real TIME_SCALE = pH->DensityBaseProjection ? (pH->dT * pH->dT) : pH->dT;

    thrust::fill(AMatrix.begin(), AMatrix.end(), 0.0);
    thrust::fill(b1Vector.begin(), b1Vector.end(), 0.0);
    thrust::fill(q_old.begin(), q_old.end(), pH->Pressure_Constraint * pH->base_pressure);
    thrust::fill(q_new.begin(), q_new.end(), pH->Pressure_Constraint * pH->base_pressure);

    Pressure_Equation<<<numBlocks, numThreads>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR3CAST(m_sortedSphMarkers_D->velMasD),
        mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), R1CAST(AMatrix), R1CAST(b1Vector), mR3CAST(V_star_new),
        R1CAST(q_new), R1CAST(csrValFunction), R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(_sumWij_inv),
        mR3CAST(Normals), U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart),

        mR4CAST(m_data_mgr.fsiBodyState_D->rot), mR3CAST(m_data_mgr.rigid_BCEcoords_D),
        mR3CAST(m_data_mgr.fsiBodyState_D->pos), mR3CAST(m_data_mgr.fsiBodyState_D->lin_vel),
        mR3CAST(m_data_mgr.fsiBodyState_D->ang_vel), mR3CAST(m_data_mgr.fsiBodyState_D->lin_acc),
        mR3CAST(m_data_mgr.fsiBodyState_D->ang_acc), U1CAST(m_data_mgr.rigid_BCEsolids_D),

        mR3CAST(m_data_mgr.fsiMesh1DState_D->vel_fsi_fea_D), mR3CAST(m_data_mgr.fsiMesh1DState_D->acc_fsi_fea_D),
        mR3CAST(m_data_mgr.fsiMesh2DState_D->vel_fsi_fea_D), mR3CAST(m_data_mgr.fsiMesh2DState_D->acc_fsi_fea_D),

        cH->numFsiElements1D, U2CAST(m_data_mgr.flex1D_Nodes_D), U3CAST(m_data_mgr.flex1D_BCEsolids_D),
        mR3CAST(m_data_mgr.flex1D_BCEcoords_D), U3CAST(m_data_mgr.flex2D_Nodes_D),
        U3CAST(m_data_mgr.flex2D_BCEsolids_D), mR3CAST(m_data_mgr.flex2D_BCEcoords_D),

        updatePortion, U1CAST(m_data_mgr.markersProximity_D->gridMarkerIndexD), numAllMarkers, cH->numFluidMarkers,
        pH->dT, error_flagD);
    cudaCheckErrorFlag(error_flagD, "Pressure_Equation");

    Real Ave_RHS = thrust::reduce(b1Vector.begin(), b1Vector.end(), 0.0) / numAllMarkers;

    my_Functor mf(Ave_RHS);
    if (pH->Pressure_Constraint) {
        thrust::for_each(b1Vector.begin(), b1Vector.end(), mf);
        Real Ave_after = thrust::reduce(b1Vector.begin(), b1Vector.end(), 0.0) / numAllMarkers;
        printf("Ave RHS =%f, Ave after removing null space=%f\n", Ave_RHS, Ave_after);
    }

    if (pH->LinearSolver != SolverType::JACOBI) {
        myLinearSolver->SetVerbose(pH->Verbose_monitoring);
        myLinearSolver->SetAbsRes(pH->LinearSolver_Abs_Tol);
        myLinearSolver->SetRelRes(pH->LinearSolver_Rel_Tol);
        myLinearSolver->SetIterationLimit(pH->LinearSolver_Max_Iter);

        myLinearSolver->Solve((int)numAllMarkers, NNZ, R1CAST(AMatrix), U1CAST(m_data_mgr.numNeighborsPerPart),
                              U1CAST(m_data_mgr.neighborList), R1CAST(q_new), R1CAST(b1Vector));

        cudaCheckError();
        MaxRes = myLinearSolver->GetResidual();
        Iteration = myLinearSolver->GetNumIterations();

        if (myLinearSolver->GetSolverStatus()) {
            if (m_verbose) {
                cout << " Linear solver converged to " << myLinearSolver->GetResidual() << " tolerance";
                cout << " after " << myLinearSolver->GetNumIterations() << " iterations" << endl;
            }
        } else {
            cerr << " Failed to converge after " << myLinearSolver->GetNumIterations() << " iterations" << endl;
            cerr << " Falling back on Jacobi" << endl;
        }
    }

    if (pH->LinearSolver == SolverType::JACOBI || !myLinearSolver->GetSolverStatus()) {
        thrust::fill(Residuals.begin(), Residuals.end(), 0.0);
        while ((MaxRes > pH->LinearSolver_Abs_Tol || Iteration < 3) && Iteration < pH->LinearSolver_Max_Iter) {
            Jacobi_SOR_Iter<<<numBlocks, numThreads>>>(
                mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), R1CAST(AMatrix), mR3CAST(V_star_old), mR3CAST(V_star_new),
                mR3CAST(b3Vector), R1CAST(q_old), R1CAST(q_new), R1CAST(b1Vector), U1CAST(m_data_mgr.neighborList),
                U1CAST(m_data_mgr.numNeighborsPerPart), false, error_flagD);
            cudaCheckErrorFlag(error_flagD, "Jacobi_SOR_Iter");

            //            if (pH->Pressure_Constraint) {
            //                Real sum_last = 0;
            //                hipblasHandle_t cublasHandle = 0;
            //                uint Start_last = Contact_i[numAllMarkers];
            //                hipblasDdot(cublasHandle, numAllMarkers, R1CAST(b1Vector), 1,
            //                           (double*)thrust::raw_pointer_cast(&AMatrix[Start_last]), 1, &sum_last);
            //                hipDeviceSynchronize();
            //                b1Vector[numAllMarkers] += b1Vector[0];
            //                q_new[numAllMarkers] = b1Vector[numAllMarkers] - sum_last -
            //                                       q_new[numAllMarkers] * AMatrix[Contact_i[numAllMarkers + 1] - 1];
            //            }mu_s_

            Update_AND_Calc_Res<<<numBlocks, numThreads>>>(mR4CAST(m_sortedSphMarkers_D->rhoPresMuD),
                                                           mR3CAST(V_star_old), mR3CAST(V_star_new), R1CAST(q_old),
                                                           R1CAST(q_new), R1CAST(Residuals), false, error_flagD);
            cudaCheckErrorFlag(error_flagD, "Update_AND_Calc_Res");

            Iteration++;
            thrust::device_vector<Real>::iterator iter = thrust::max_element(Residuals.begin(), Residuals.end());
            auto position = iter - Residuals.begin();
            MaxRes = *iter;

            if (pH->Verbose_monitoring)
                printf("Iter= %.4d, Res= %.4e\n", Iteration, MaxRes);
        }
    }
    //    Real4_y unary_op_p;
    //    thrust::plus<Real> binary_op;
    //    Real Ave_pressure = thrust::transform_reduce(m_sortedSphMarkers_D->rhoPresMuD.begin(),
    //                                                 m_sortedSphMarkers_D->rhoPresMuD.end(), unary_op_p, 0.0,
    //                                                 binary_op)
    //                                                 /
    //                        (cH->numFluidMarkers);

    Real Ave_pressure = thrust::reduce(q_new.begin(), q_new.end(), 0.0) / (cH->numAllMarkers) / TIME_SCALE;

    thrust::for_each(b1Vector.begin(), b1Vector.end(), mf);
    Real Ave_after = thrust::reduce(b1Vector.begin(), b1Vector.end(), 0.0) / numAllMarkers;
    if (m_verbose) {
        double Pressure_Computation = (clock() - LinearSystemClock_p) / (double)CLOCKS_PER_SEC;
        printf("Ave RHS =%.3e, Ave after removing null space=%.3e\n", Ave_RHS, Ave_after);
        printf("| Pressure Poisson Equation: %f (sec) - Final Residual=%.3e - #Iter=%d, Ave_p=%.3e\n",
               Pressure_Computation, MaxRes, Iteration, Ave_pressure);
    }

    // ----- Velocity_Correction_and_update
    double updateClock = clock();
    rhoPresMuD_old = m_sortedSphMarkers_D->rhoPresMuD;
    posRadD_old = m_sortedSphMarkers_D->posRadD;
    velMasD_old = m_sortedSphMarkers_D->velMasD;

    thrust::fill(m_data_mgr.vis_vel_SPH_D.begin(), m_data_mgr.vis_vel_SPH_D.end(), mR3(0.0));

    Velocity_Correction_and_update<<<numBlocks, numThreads>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR4CAST(posRadD_old), mR4CAST(m_sortedSphMarkers_D->rhoPresMuD),
        mR4CAST(rhoPresMuD_old), mR3CAST(m_sortedSphMarkers_D->velMasD), mR3CAST(velMasD_old),
        mR3CAST(m_sortedSphMarkers_D->tauXxYyZzD), mR3CAST(m_sortedSphMarkers_D->tauXyXzYzD),
        mR4CAST(m_data_mgr.sr_tau_I_mu_i), mR3CAST(m_data_mgr.vis_vel_SPH_D), mR4CAST(m_data_mgr.derivVelRhoD),
        mR3CAST(V_star_new), R1CAST(q_new), R1CAST(csrValFunction), mR3CAST(csrValGradient), R1CAST(csrValLaplacian),
        U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart), numAllMarkers, MaxVel, pH->dT,
        error_flagD);
    cudaCheckErrorFlag(error_flagD, "Velocity_Correction_and_update");

    // fsiCollisionSystem->ArrangeData(sphMarkersD);
    PreProcessor(false);

    rhoPresMuD_old = m_sortedSphMarkers_D->rhoPresMuD;
    posRadD_old = m_sortedSphMarkers_D->posRadD;
    velMasD_old = m_sortedSphMarkers_D->velMasD;

    Shifting<<<numBlocks, numThreads>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR4CAST(posRadD_old), mR4CAST(m_sortedSphMarkers_D->rhoPresMuD),
        mR4CAST(rhoPresMuD_old), mR3CAST(m_sortedSphMarkers_D->velMasD), mR3CAST(velMasD_old),
        mR3CAST(m_data_mgr.vis_vel_SPH_D), R1CAST(csrValFunction), mR3CAST(csrValGradient),
        U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart), numAllMarkers, MaxVel, pH->dT,
        error_flagD);
    cudaCheckErrorFlag(error_flagD, "Shifting");

    Real4_x unary_op(pH->rho0);
    thrust::plus<Real> binary_op;
    Real Ave_density_Err = thrust::transform_reduce(m_sortedSphMarkers_D->rhoPresMuD.begin(),
                                                    m_sortedSphMarkers_D->rhoPresMuD.end(), unary_op, 0.0, binary_op) /
                           (cH->numFluidMarkers * pH->rho0);

    double updateComputation = (clock() - updateClock) / (double)CLOCKS_PER_SEC;
    Real Re = pH->L_Characteristic * pH->rho0 * MaxVel / pH->mu0;

    printf("| Velocity_Correction_and_update: %f (sec), Ave_density_Err=%.3e, Re=%.1f\n", updateComputation,
           Ave_density_Err, Re);

    // post-processing for conservative formulation
    if (pH->Conservative_Form && pH->ClampPressure) {
        Real minP =
            thrust::transform_reduce(m_sortedSphMarkers_D->rhoPresMuD.begin(), m_sortedSphMarkers_D->rhoPresMuD.end(),
                                     Real4_y_min(), 1e9, thrust::minimum<Real>());
        my_Functor_real4y negate(minP);
        thrust::for_each(m_sortedSphMarkers_D->rhoPresMuD.begin(), m_sortedSphMarkers_D->rhoPresMuD.end(), negate);
        printf("Shifting min pressure of %.3e to 0\n", minP);
    }

    csrValGradient.clear();
    csrValLaplacian.clear();
    csrValFunction.clear();
    AMatrix.clear();
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChFsiForceI2SPH::neighborSearch() {
    bool* error_flagD;
    cudaMallocErrorFlag(error_flagD);
    cudaResetErrorFlag(error_flagD);

    // thread per particle
    uint numBlocksShort, numThreadsShort;
    computeGridSize(m_data_mgr.countersH->numAllMarkers, 256, numBlocksShort, numThreadsShort);

    // Execute the kernel
    thrust::fill(m_data_mgr.numNeighborsPerPart.begin(), m_data_mgr.numNeighborsPerPart.end(), 0);

    neighborSearchNum<<<numBlocksShort, numThreadsShort>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR4CAST(m_sortedSphMarkers_D->rhoPresMuD),
        U1CAST(m_data_mgr.markersProximity_D->cellStartD), U1CAST(m_data_mgr.markersProximity_D->cellEndD),
        U1CAST(m_data_mgr.activityIdentifierD), U1CAST(m_data_mgr.numNeighborsPerPart), error_flagD);
    cudaCheckErrorFlag(error_flagD, "neighborSearchNum");

    // in-place exclusive scan for num of neighbors
    thrust::exclusive_scan(m_data_mgr.numNeighborsPerPart.begin(), m_data_mgr.numNeighborsPerPart.end(),
                           m_data_mgr.numNeighborsPerPart.begin());
    m_data_mgr.neighborList.resize(m_data_mgr.numNeighborsPerPart.back());
    thrust::fill(m_data_mgr.neighborList.begin(), m_data_mgr.neighborList.end(), 0);

    // second pass
    neighborSearchID<<<numBlocksShort, numThreadsShort>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR4CAST(m_sortedSphMarkers_D->rhoPresMuD),
        U1CAST(m_data_mgr.markersProximity_D->cellStartD), U1CAST(m_data_mgr.markersProximity_D->cellEndD),
        U1CAST(m_data_mgr.activityIdentifierD), U1CAST(m_data_mgr.numNeighborsPerPart), U1CAST(m_data_mgr.neighborList),
        error_flagD);
    cudaCheckErrorFlag(error_flagD, "neighborSearchID");

    cudaFreeErrorFlag(error_flagD);
}

//--------------------------------------------------------------------------------------------------------------------------------

//--------------------------------------------------------------------------------------------------------------------------------

void ChFsiForceI2SPH::PreProcessor(bool calcLaplacianOperator) {
    cudaResetErrorFlag(error_flagD);

    numAllMarkers = m_data_mgr.countersH->numAllMarkers;
    uint numThreads, numBlocks;
    computeGridSize((int)numAllMarkers, 128, numBlocks, numThreads);
    thrust::fill(_sumWij_inv.begin(), _sumWij_inv.end(), 1e-3);
    thrust::fill(A_i.begin(), A_i.end(), 0);
    thrust::fill(L_i.begin(), L_i.end(), 0);
    thrust::fill(G_i.begin(), G_i.end(), 0);

    calcRho_kernel<<<numBlocks, numThreads>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), R1CAST(_sumWij_inv),
        U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart), error_flagD);
    cudaCheckErrorFlag(error_flagD, "calcRho_kernel");

    NNZ = m_data_mgr.neighborList.size();
    csrValGradient.resize(NNZ);
    csrValLaplacian.resize(NNZ);
    csrValFunction.resize(NNZ);
    AMatrix.resize(NNZ);
    thrust::fill(csrValGradient.begin(), csrValGradient.end(), mR3(0.0));
    thrust::fill(csrValLaplacian.begin(), csrValLaplacian.end(), 0.0);
    thrust::fill(csrValFunction.begin(), csrValFunction.end(), 0.0);

    calcNormalizedRho_Gi_fillInMatrixIndices<<<numBlocks, numThreads>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR3CAST(m_sortedSphMarkers_D->velMasD),
        mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), mR3CAST(Normals),
        U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart), error_flagD);

    cudaCheckErrorFlag(error_flagD, "calcNormalizedRho_Gi_fillInMatrixIndices");

    double A_L_Tensor_GradLaplacian = clock();

    if (calcLaplacianOperator && !m_data_mgr.paramsH->Conservative_Form) {
        printf("| calc_A_tensor+");

        calc_A_tensor<<<numBlocks, numThreads>>>(
            R1CAST(A_i), R1CAST(G_i), mR4CAST(m_sortedSphMarkers_D->posRadD), mR4CAST(m_sortedSphMarkers_D->rhoPresMuD),
            R1CAST(_sumWij_inv), U1CAST(m_data_mgr.neighborList), U1CAST(m_data_mgr.numNeighborsPerPart), error_flagD);
        cudaCheckErrorFlag(error_flagD, "calc_A_tensor");

        calc_L_tensor<<<numBlocks, numThreads>>>(
            R1CAST(A_i), R1CAST(L_i), R1CAST(G_i), mR4CAST(m_sortedSphMarkers_D->posRadD),
            mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), R1CAST(_sumWij_inv), U1CAST(m_data_mgr.neighborList),
            U1CAST(m_data_mgr.numNeighborsPerPart), error_flagD);
        cudaCheckErrorFlag(error_flagD, "calc_L_tensor");
    }

    Function_Gradient_Laplacian_Operator<<<numBlocks, numThreads>>>(
        mR4CAST(m_sortedSphMarkers_D->posRadD), mR3CAST(m_sortedSphMarkers_D->velMasD),
        mR4CAST(m_sortedSphMarkers_D->rhoPresMuD), R1CAST(_sumWij_inv), R1CAST(G_i), R1CAST(L_i),
        R1CAST(csrValLaplacian), mR3CAST(csrValGradient), R1CAST(csrValFunction), U1CAST(m_data_mgr.neighborList),
        U1CAST(m_data_mgr.numNeighborsPerPart), error_flagD);
    cudaCheckErrorFlag(error_flagD, "Gradient_Laplacian_Operator");

    double Gradient_Laplacian_Operator = (clock() - A_L_Tensor_GradLaplacian) / (double)CLOCKS_PER_SEC;
}

}  // namespace sph
}  // namespace fsi
}  // namespace chrono
