// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha
// =============================================================================
//
// Base class for processing sph force in fsi system.
// =============================================================================

#include <thrust/extrema.h>
#include <thrust/sort.h>

#include "chrono_fsi/sph/physics/FsiForce.cuh"
#include "chrono_fsi/sph/utils/UtilsDevice.cuh"
#include "chrono_fsi/sph/physics/SphGeneral.cuh"

namespace chrono {
namespace fsi {
namespace sph {

FsiForce::FsiForce(FsiDataManager& data_mgr,
                       BceManager& bce_mgr,
                       bool verbose)
    : m_data_mgr(data_mgr),
      m_bce_mgr(bce_mgr),
      m_verbose(verbose),
      m_sortedSphMarkers_D(nullptr) {
    fsiCollisionSystem = chrono_types::make_shared<CollisionSystem>(data_mgr);
}

void FsiForce::Initialize() {
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), m_data_mgr.paramsH.get(), sizeof(ChFsiParamsSPH));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(countersD), m_data_mgr.countersH.get(), sizeof(Counters));

    fsiCollisionSystem->Initialize();
}

FsiForce::~FsiForce() {}

// Use invasive to avoid one extra copy.
// However, keep in mind that sorted is changed.
void FsiForce::CopySortedToOriginal_Invasive_R3(thrust::device_vector<Real3>& original,
                                                  thrust::device_vector<Real3>& sorted,
                                                  const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
    thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(), sorted.begin());
    dummyMarkerIndex.clear();
    thrust::copy(sorted.begin(), sorted.end(), original.begin());
}

void FsiForce::CopySortedToOriginal_NonInvasive_R3(thrust::device_vector<Real3>& original,
                                                     const thrust::device_vector<Real3>& sorted,
                                                     const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<Real3> dummySorted = sorted;
    CopySortedToOriginal_Invasive_R3(original, dummySorted, gridMarkerIndex);
}

// Use invasive to avoid one extra copy.
// However, keep in mind that sorted is changed.
void FsiForce::CopySortedToOriginal_Invasive_R4(thrust::device_vector<Real4>& original,
                                                  thrust::device_vector<Real4>& sorted,
                                                  const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
    thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(), sorted.begin());
    dummyMarkerIndex.clear();
    thrust::copy(sorted.begin(), sorted.end(), original.begin());
}

void FsiForce::CopySortedToOriginal_NonInvasive_R4(thrust::device_vector<Real4>& original,
                                                     thrust::device_vector<Real4>& sorted,
                                                     const thrust::device_vector<uint>& gridMarkerIndex) {
    thrust::device_vector<Real4> dummySorted = sorted;
    CopySortedToOriginal_Invasive_R4(original, dummySorted, gridMarkerIndex);
}

}  // namespace sph
}  // namespace fsi
}  // namespace chrono
