#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Arman Pazouki, Milad Rakhsha, Wei Hu
// =============================================================================
//
// Base class for processing proximity in fsi system.
// =============================================================================

#include <thrust/sort.h>

#include "chrono_fsi/sph/physics/ChCollisionSystemFsi.cuh"
#include "chrono_fsi/sph/physics/ChSphGeneral.cuh"
#include "chrono_fsi/sph/utils/ChUtilsDevice.cuh"

namespace chrono {
namespace fsi {
namespace sph {

// calcHashD :
// 1. Get particle index determined by the block and thread we are in.
// 2. From x, y, z position, determine which bin it is in.
// 3. Calculate hash from bin index.
// 4. Store hash and particle index associated with it.
__global__ void calcHashD(uint* gridMarkerHashD,   // gridMarkerHash Store particle hash here
                          uint* gridMarkerIndexD,  // gridMarkerIndex Store particle index here
                          const Real4* posRad,     // positions of all particles (SPH and BCE)
                          volatile bool* error_flag) {
    // Calculate the index of where the particle is stored in posRad.
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= countersD.numAllMarkers)
        return;

    Real3 p = mR3(posRad[index]);

    if (!IsFinite(p)) {
        printf("[calcHashD] index %d position is NaN\n", index);
        *error_flag = true;
        return;
    }

    // Check particle is inside the domain.
    Real3 boxCorner = paramsD.worldOrigin - mR3(40 * paramsD.h);
    if (p.x < boxCorner.x || p.y < boxCorner.y || p.z < boxCorner.z) {
        printf("[calcHashD] index %u (%f %f %f) out of min boundary (%f %f %f)\n",  //
               index, p.x, p.y, p.z, boxCorner.x, boxCorner.y, boxCorner.z);
        *error_flag = true;
        return;
    }
    boxCorner = paramsD.worldOrigin + paramsD.boxDims + mR3(40 * paramsD.h);
    if (p.x > boxCorner.x || p.y > boxCorner.y || p.z > boxCorner.z) {
        printf("[calcHashD] index %u (%f %f %f) out of max boundary (%f %f %f)\n",  //
               index, p.x, p.y, p.z, boxCorner.x, boxCorner.y, boxCorner.z);
        *error_flag = true;
        return;
    }

    // Get x,y,z bin index in grid
    int3 gridPos = calcGridPos(p);
    // Calculate a hash from the bin index
    uint hash = calcGridHash(gridPos);
    // Store grid hash
    // grid hash is a scalar cell ID
    gridMarkerHashD[index] = hash;
    // Store particle index associated to the hash we stored in gridMarkerHashD
    gridMarkerIndexD[index] = index;
}
// ------------------------------------------------------------------------------
__global__ void findCellStartEndD(uint* cellStartD,       // output: cell start index
                                  uint* cellEndD,         // output: cell end index
                                  uint* gridMarkerHashD,  // input: sorted grid hashes
                                  uint* gridMarkerIndexD  // input: sorted particle indices
) {
    extern __shared__ uint sharedHash[];  // blockSize + 1 elements
    // Get the particle index the current thread is supposed to be looking at.
    uint index = blockIdx.x * blockDim.x + threadIdx.x;
    uint hash;
    // handle case when no. of particles not multiple of block size
    if (index < countersD.numAllMarkers) {
        hash = gridMarkerHashD[index];
        // Load hash data into shared memory so that we can look at neighboring
        // particle's hash value without loading two hash values per thread
        sharedHash[threadIdx.x + 1] = hash;

        // first thread in block must load neighbor particle hash
        if (index > 0 && threadIdx.x == 0)
            sharedHash[0] = gridMarkerHashD[index - 1];
    }

    __syncthreads();

    if (index < countersD.numAllMarkers) {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell. As it
        // isn't the first particle, it must also be the cell end of
        // the previous particle's cell.
        if (index == 0 || hash != sharedHash[threadIdx.x]) {
            cellStartD[hash] = index;
            if (index > 0)
                cellEndD[sharedHash[threadIdx.x]] = index;
        }

        if (index == countersD.numAllMarkers - 1)
            cellEndD[hash] = index + 1;
    }
}
// ------------------------------------------------------------------------------
__global__ void reorderDataD(uint* gridMarkerIndexD,     // input: sorted particle indices
                             uint* extendedActivityIdD,  // input: particles in an extended active sub-domain
                             uint* mapOriginalToSorted,  // input: original index to sorted index
                             Real4* sortedPosRadD,       // output: sorted positions
                             Real3* sortedVelMasD,       // output: sorted velocities
                             Real4* sortedRhoPreMuD,     // output: sorted density pressure
                             Real3* sortedTauXxYyZzD,    // output: sorted total stress xxyyzz
                             Real3* sortedTauXyXzYzD,    // output: sorted total stress xyzxyz
                             Real4* posRadD,             // input: original position array
                             Real3* velMasD,             // input: original velocity array
                             Real4* rhoPresMuD,          // input: original density pressure
                             Real3* tauXxYyZzD,          // input: original total stress xxyyzz
                             Real3* tauXyXzYzD           // input: original total stress xyzxyz
) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= countersD.numAllMarkers)
        return;

    // Now use the sorted index to reorder the pos and vel data
    uint originalIndex = id;

    // no need to do anything if it is not an active particle
    // uint activity = extendedActivityIdD[originalIndex];
    // if (activity == 0)
    //     return;

    // map original to sorted
    uint index = mapOriginalToSorted[originalIndex];

    Real3 posRad = mR3(posRadD[originalIndex]);
    Real3 velMas = velMasD[originalIndex];
    Real4 rhoPreMu = rhoPresMuD[originalIndex];

    if (!IsFinite(posRad)) {
        printf(
            "Error! particle position is NAN: thrown from "
            "ChCollisionSystemFsi.cu, reorderDataD !\n");
    }
    if (!IsFinite(velMas)) {
        printf(
            "Error! particle velocity is NAN: thrown from "
            "ChCollisionSystemFsi.cu, reorderDataD !\n");
    }
    if (!IsFinite(rhoPreMu)) {
        printf(
            "Error! particle rhoPreMu is NAN: thrown from "
            "ChCollisionSystemFsi.cu, reorderDataD !\n");
    }

    sortedPosRadD[index] = mR4(posRad, posRadD[originalIndex].w);
    sortedVelMasD[index] = velMas;
    sortedRhoPreMuD[index] = rhoPreMu;

    // For granular material
    if (paramsD.elastic_SPH) {
        Real3 tauXxYyZz = tauXxYyZzD[originalIndex];
        Real3 tauXyXzYz = tauXyXzYzD[originalIndex];
        if (!IsFinite(tauXxYyZz)) {
            printf(
                "Error! particle tauXxYyZz is NAN: thrown from "
                "ChCollisionSystemFsi.cu, reorderDataD !\n");
        }
        if (!IsFinite(tauXyXzYz)) {
            printf(
                "Error! particle tauXyXzYz is NAN: thrown from "
                "ChCollisionSystemFsi.cu, reorderDataD !\n");
        }
        sortedTauXxYyZzD[index] = tauXxYyZz;
        sortedTauXyXzYzD[index] = tauXyXzYz;
    }
}
// ------------------------------------------------------------------------------
__global__ void OriginalToSortedD(uint* mapOriginalToSorted, uint* gridMarkerIndex) {
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= countersD.numAllMarkers)
        return;

    uint index = gridMarkerIndex[id];

    mapOriginalToSorted[index] = id;
}
// ------------------------------------------------------------------------------
ChCollisionSystemFsi::ChCollisionSystemFsi(FsiDataManager& data_mgr) : m_data_mgr(data_mgr), m_sphMarkersD(nullptr) {}

ChCollisionSystemFsi::~ChCollisionSystemFsi() {}
// ------------------------------------------------------------------------------
void ChCollisionSystemFsi::Initialize() {
    hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), m_data_mgr.paramsH.get(), sizeof(SimParams));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(countersD), m_data_mgr.countersH.get(), sizeof(Counters));
}

// ------------------------------------------------------------------------------

void ChCollisionSystemFsi::ArrangeData(std::shared_ptr<SphMarkerDataD> sphMarkersD) {
    bool* error_flagD;
    cudaMallocErrorFlag(error_flagD);
    cudaResetErrorFlag(error_flagD);

    m_sphMarkersD = sphMarkersD;
    int3 cellsDim = m_data_mgr.paramsH->gridSize;
    int numCells = cellsDim.x * cellsDim.y * cellsDim.z;

    uint numThreads, numBlocks;
    computeGridSize((uint)m_data_mgr.countersH->numAllMarkers, 256, numBlocks, numThreads);

    // Reset cell size
    m_data_mgr.markersProximity_D->cellStartD.resize(numCells);
    m_data_mgr.markersProximity_D->cellEndD.resize(numCells);

    // =========================================================================================================
    // Calculate Hash
    // =========================================================================================================
    if (!(m_data_mgr.markersProximity_D->gridMarkerHashD.size() == m_data_mgr.countersH->numAllMarkers &&
          m_data_mgr.markersProximity_D->gridMarkerIndexD.size() == m_data_mgr.countersH->numAllMarkers)) {
        printf("[calcHashD] marker hash size: %u | marker index size: %u\n            num markers: %u\n",  //
               (uint)m_data_mgr.markersProximity_D->gridMarkerHashD.size(),                                //
               (uint)m_data_mgr.markersProximity_D->gridMarkerIndexD.size(),                               //
               (uint)m_data_mgr.countersH->numAllMarkers);                                                 //
        throw std::runtime_error("Error! size error, calcHash!");
    }

    // Execute Kernel
    calcHashD<<<numBlocks, numThreads>>>(U1CAST(m_data_mgr.markersProximity_D->gridMarkerHashD),
                                         U1CAST(m_data_mgr.markersProximity_D->gridMarkerIndexD),
                                         mR4CAST(m_sphMarkersD->posRadD), error_flagD);
    cudaCheckErrorFlag(error_flagD, "calcHashD");

    // =========================================================================================================
    // Sort Particles based on Hash
    // =========================================================================================================
    thrust::sort_by_key(m_data_mgr.markersProximity_D->gridMarkerHashD.begin(),
                        m_data_mgr.markersProximity_D->gridMarkerHashD.end(),
                        m_data_mgr.markersProximity_D->gridMarkerIndexD.begin());

    // =========================================================================================================
    // Find the start index and the end index of the sorted array in each cell
    // =========================================================================================================
    // Reset proximity cell data
    if (!(m_data_mgr.markersProximity_D->cellStartD.size() == numCells &&
          m_data_mgr.markersProximity_D->cellEndD.size() == numCells)) {
        throw std::runtime_error("Error! size error, ArrangeData!\n");
    }

    thrust::fill(m_data_mgr.markersProximity_D->cellStartD.begin(), m_data_mgr.markersProximity_D->cellStartD.end(), 0);
    thrust::fill(m_data_mgr.markersProximity_D->cellEndD.begin(), m_data_mgr.markersProximity_D->cellEndD.end(), 0);

    uint smemSize = sizeof(uint) * (numThreads + 1);
    findCellStartEndD<<<numBlocks, numThreads, smemSize>>>(U1CAST(m_data_mgr.markersProximity_D->cellStartD),
                                                           U1CAST(m_data_mgr.markersProximity_D->cellEndD),
                                                           U1CAST(m_data_mgr.markersProximity_D->gridMarkerHashD),
                                                           U1CAST(m_data_mgr.markersProximity_D->gridMarkerIndexD));

    // =========================================================================================================
    // Launch a kernel to find the location of original particles in the sorted arrays.
    // This is faster than using thrust::sort_by_key()
    // =========================================================================================================
    OriginalToSortedD<<<numBlocks, numThreads>>>(U1CAST(m_data_mgr.markersProximity_D->mapOriginalToSorted),
                                                 U1CAST(m_data_mgr.markersProximity_D->gridMarkerIndexD));

    // =========================================================================================================
    // Reorder the arrays according to the sorted index of all particles
    // =========================================================================================================
    reorderDataD<<<numBlocks, numThreads>>>(
        U1CAST(m_data_mgr.markersProximity_D->gridMarkerIndexD), U1CAST(m_data_mgr.extendedActivityIdD),
        U1CAST(m_data_mgr.markersProximity_D->mapOriginalToSorted), mR4CAST(m_data_mgr.sortedSphMarkers2_D->posRadD),
        mR3CAST(m_data_mgr.sortedSphMarkers2_D->velMasD), mR4CAST(m_data_mgr.sortedSphMarkers2_D->rhoPresMuD),
        mR3CAST(m_data_mgr.sortedSphMarkers2_D->tauXxYyZzD), mR3CAST(m_data_mgr.sortedSphMarkers2_D->tauXyXzYzD),
        mR4CAST(m_sphMarkersD->posRadD), mR3CAST(m_sphMarkersD->velMasD), mR4CAST(m_sphMarkersD->rhoPresMuD),
        mR3CAST(m_sphMarkersD->tauXxYyZzD), mR3CAST(m_sphMarkersD->tauXyXzYzD));

    hipDeviceSynchronize();
    cudaCheckError();

    cudaFreeErrorFlag(error_flagD);
}

}  // namespace sph
}  // end namespace fsi
}  // end namespace chrono
