// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha, Arman Pazouki
// =============================================================================
//
// Base class for changing device arrays in non-cuda files
// =============================================================================
/**
 * @brief See collideSphereSphere.cuh for documentation.
 */

#include "chrono_fsi/utils/ChUtilsDevice.cuh"

namespace chrono {
namespace fsi {

void ChUtilsDevice::ResizeMyThrust3(thrust::device_vector<Real3>& mThrustVec, int mSize) {
    mThrustVec.resize(mSize);
}
void ChUtilsDevice::ResizeMyThrust4(thrust::device_vector<Real4>& mThrustVec, int mSize) {
    mThrustVec.resize(mSize);
}
void ChUtilsDevice::FillMyThrust3(thrust::device_vector<Real3>& mThrustVec, Real3 v) {  //
    thrust::fill(mThrustVec.begin(), mThrustVec.end(), v);
}
void ChUtilsDevice::FillMyThrust4(thrust::device_vector<Real4>& mThrustVec, Real4 v) {
    thrust::fill(mThrustVec.begin(), mThrustVec.end(), v);
}
void ChUtilsDevice::ClearMyThrustR3(thrust::device_vector<Real3>& mThrustVec) {
    mThrustVec.clear();
}
void ChUtilsDevice::ClearMyThrustR4(thrust::device_vector<Real4>& mThrustVec) {
    mThrustVec.clear();
}
void ChUtilsDevice::ClearMyThrustU1(thrust::device_vector<uint>& mThrustVec) {
    mThrustVec.clear();
}
void ChUtilsDevice::PushBackR3(thrust::device_vector<Real3>& mThrustVec, Real3 a3) {
    mThrustVec.push_back(a3);
}
void ChUtilsDevice::PushBackR4(thrust::device_vector<Real4>& mThrustVec, Real4 a4) {
    mThrustVec.push_back(a4);
}
void ChUtilsDevice::ResizeR3(thrust::device_vector<Real3>& mThrustVec, int size) {
    mThrustVec.resize(size);
}
void ChUtilsDevice::ResizeR4(thrust::device_vector<Real4>& mThrustVec, int size) {
    mThrustVec.resize(size);
}
void ChUtilsDevice::ResizeU1(thrust::device_vector<uint>& mThrustVec, int size) {
    mThrustVec.resize(size);
}

Real3 ChUtilsDevice::FetchElement(const thrust::device_vector<Real3>& DevVec, size_t i) {
    return DevVec[i];
}

void ChUtilsDevice::CopyD2H(thrust::device_vector<Real4>& DevVec, thrust::host_vector<Real4>& HostVec) {
    thrust::copy(DevVec.begin(), DevVec.end(), HostVec.begin());
}
void ChUtilsDevice::CopyD2H(thrust::device_vector<Real3>& DevVec, thrust::host_vector<Real3>& HostVec) {
    thrust::copy(DevVec.begin(), DevVec.end(), HostVec.begin());
}
void ChUtilsDevice::CopyD2H(thrust::device_vector<Real>& DevVec, thrust::host_vector<Real>& HostVec) {
    thrust::copy(DevVec.begin(), DevVec.end(), HostVec.begin());
}

void ChUtilsDevice::Sync_CheckError(bool* isErrorH, bool* isErrorD, std::string carshReport) {
    hipDeviceSynchronize();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after " + carshReport + " !\n");
    }
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        throw std::runtime_error("Error! program crashed after " + carshReport + " !\n");
    }
    cudaCheckError();
}

}  // end namespace fsi
}  // end namespace chrono
