// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha, Arman Pazouki, RFadu Serban
// =============================================================================
//
// Utilities for changing device arrays in non-cuda files
// =============================================================================

#include "chrono_fsi/utils/ChUtilsDevice.cuh"

namespace chrono {
namespace fsi {

GpuTimer::GpuTimer(hipStream_t stream) : m_stream(stream) {
    hipEventCreate(&m_start);
    hipEventCreate(&m_stop);
}

GpuTimer::~GpuTimer() {
    hipEventDestroy(m_start);
    hipEventDestroy(m_stop);
}

void GpuTimer::Start() {
    hipEventRecord(m_start, m_stream);
}

void GpuTimer::Stop() {
    hipEventRecord(m_stop, m_stream);
}

float GpuTimer::Elapsed() {
    float elapsed;
    hipEventSynchronize(m_stop);
    hipEventElapsedTime(&elapsed, m_start, m_stop);
    return elapsed;
}

void ChUtilsDevice::FillVector(thrust::device_vector<Real3>& vector, const Real3& value) {
    thrust::fill(vector.begin(), vector.end(), value);
}

void ChUtilsDevice::FillVector(thrust::device_vector<Real4>& vector, const Real4& value) {
    thrust::fill(vector.begin(), vector.end(), value);
}

void ChUtilsDevice::FillVector(thrust::device_vector<uint>& vector, const uint& value) {
    thrust::fill(vector.begin(), vector.end(), value);
}

void ChUtilsDevice::FillVector(thrust::device_vector<Real>& vector, const Real& value) {
    thrust::fill(vector.begin(), vector.end(), value);
}

void ChUtilsDevice::Sync_CheckError(bool* isErrorH, bool* isErrorD, std::string crashReport) {
    hipDeviceSynchronize();
    hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
    if (*isErrorH == true) {
        throw std::runtime_error("Error! program crashed after " + crashReport + " !\n");
    }
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        throw std::runtime_error("Error! program crashed after " + crashReport + " !\n");
    }
    cudaCheckError();
}

}  // end namespace fsi
}  // end namespace chrono
