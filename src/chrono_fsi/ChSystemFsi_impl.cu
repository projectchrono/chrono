#include "hip/hip_runtime.h"
// =============================================================================
// PROJECT CHRONO - http://projectchrono.org
//
// Copyright (c) 2014 projectchrono.org
// All rights reserved.
//
// Use of this source code is governed by a BSD-style license that can be found
// in the LICENSE file at the top level of the distribution and at
// http://projectchrono.org/license-chrono.txt.
//
// =============================================================================
// Author: Milad Rakhsha, Arman Pazouki
// =============================================================================
//
// Implementation of FSI system that includes all subclasses for proximity and
// force calculation, and time integration.
//
// =============================================================================

#include "chrono_fsi/ChSystemFsi_impl.cuh"

namespace chrono {
namespace fsi {

struct sphTypeCompEqual {
    __host__ __device__ bool operator()(const Real4& o1, const Real4& o2) { return o1.w == o2.w; }
};
//---------------------------------------------------------------------------------------
zipIterSphD SphMarkerDataD::iterator() {
    return thrust::make_zip_iterator(thrust::make_tuple(posRadD.begin(), velMasD.begin(), rhoPresMuD.begin(),
                                                        tauXxYyZzD.begin(), tauXyXzYzD.begin()));
}

void SphMarkerDataD::resize(size_t s) {
    posRadD.resize(s);
    velMasD.resize(s);
    rhoPresMuD.resize(s);
    tauXxYyZzD.resize(s);
    tauXyXzYzD.resize(s);
}

//---------------------------------------------------------------------------------------
zipIterSphH SphMarkerDataH::iterator() {
    return thrust::make_zip_iterator(thrust::make_tuple(posRadH.begin(), velMasH.begin(), rhoPresMuH.begin(),
                                                        tauXxYyZzH.begin(), tauXyXzYzH.begin()));
}

// resize
void SphMarkerDataH::resize(size_t s) {
    posRadH.resize(s);
    velMasH.resize(s);
    rhoPresMuH.resize(s);
    tauXxYyZzH.resize(s);
    tauXyXzYzH.resize(s);
}

//---------------------------------------------------------------------------------------
zipIterRigidD FsiBodiesDataD::iterator() {
    return thrust::make_zip_iterator(thrust::make_tuple(posRigid_fsiBodies_D.begin(), 
                                                        velMassRigid_fsiBodies_D.begin(), 
                                                        accRigid_fsiBodies_D.begin(),
                                                        q_fsiBodies_D.begin(), 
                                                        omegaVelLRF_fsiBodies_D.begin(), 
                                                        omegaAccLRF_fsiBodies_D.begin()));
}

void FsiBodiesDataD::resize(size_t s) {
    posRigid_fsiBodies_D.resize(s);
    velMassRigid_fsiBodies_D.resize(s);
    accRigid_fsiBodies_D.resize(s);
    q_fsiBodies_D.resize(s);
    omegaVelLRF_fsiBodies_D.resize(s);
    omegaAccLRF_fsiBodies_D.resize(s);
}

void FsiShellsDataH::resize(size_t s) {
    posFlex_fsiBodies_nA_H.resize(s);
    posFlex_fsiBodies_nB_H.resize(s);
    posFlex_fsiBodies_nC_H.resize(s);
    posFlex_fsiBodies_nD_H.resize(s);

    velFlex_fsiBodies_nA_H.resize(s);
    velFlex_fsiBodies_nB_H.resize(s);
    velFlex_fsiBodies_nC_H.resize(s);
    velFlex_fsiBodies_nD_H.resize(s);

    accFlex_fsiBodies_nA_H.resize(s);
    accFlex_fsiBodies_nB_H.resize(s);
    accFlex_fsiBodies_nC_H.resize(s);
    accFlex_fsiBodies_nD_H.resize(s);
}

void FsiShellsDataD::resize(size_t s) {
    posFlex_fsiBodies_nA_D.resize(s);
    posFlex_fsiBodies_nB_D.resize(s);
    posFlex_fsiBodies_nC_D.resize(s);
    posFlex_fsiBodies_nD_D.resize(s);

    velFlex_fsiBodies_nA_D.resize(s);
    velFlex_fsiBodies_nB_D.resize(s);
    velFlex_fsiBodies_nC_D.resize(s);
    velFlex_fsiBodies_nD_D.resize(s);

    accFlex_fsiBodies_nA_D.resize(s);
    accFlex_fsiBodies_nB_D.resize(s);
    accFlex_fsiBodies_nC_D.resize(s);
    accFlex_fsiBodies_nD_D.resize(s);
}
void FsiMeshDataH::resize(size_t s) {
    pos_fsi_fea_H.resize(s);
    vel_fsi_fea_H.resize(s);
    acc_fsi_fea_H.resize(s);
}
void FsiMeshDataD::resize(size_t s) {
    pos_fsi_fea_D.resize(s);
    vel_fsi_fea_D.resize(s);
    acc_fsi_fea_D.resize(s);
}

void FsiBodiesDataD::CopyFromH(const FsiBodiesDataH& other) {
    thrust::copy(other.posRigid_fsiBodies_H.begin(), other.posRigid_fsiBodies_H.end(), 
                 posRigid_fsiBodies_D.begin());
    thrust::copy(other.velMassRigid_fsiBodies_H.begin(), other.velMassRigid_fsiBodies_H.end(),
                 velMassRigid_fsiBodies_D.begin());
    thrust::copy(other.accRigid_fsiBodies_H.begin(), other.accRigid_fsiBodies_H.end(), 
                 accRigid_fsiBodies_D.begin());
    thrust::copy(other.q_fsiBodies_H.begin(), other.q_fsiBodies_H.end(), 
                 q_fsiBodies_D.begin());
    thrust::copy(other.omegaVelLRF_fsiBodies_H.begin(), other.omegaVelLRF_fsiBodies_H.end(),
                 omegaVelLRF_fsiBodies_D.begin());
    thrust::copy(other.omegaAccLRF_fsiBodies_H.begin(), other.omegaAccLRF_fsiBodies_H.end(),
                 omegaAccLRF_fsiBodies_D.begin());
}

void FsiShellsDataD::CopyFromH(const FsiShellsDataH& other) {
    thrust::copy(other.posFlex_fsiBodies_nA_H.begin(), other.posFlex_fsiBodies_nA_H.end(),
                 posFlex_fsiBodies_nA_D.begin());
    thrust::copy(other.posFlex_fsiBodies_nB_H.begin(), other.posFlex_fsiBodies_nB_H.end(),
                 posFlex_fsiBodies_nB_D.begin());
    thrust::copy(other.posFlex_fsiBodies_nC_H.begin(), other.posFlex_fsiBodies_nC_H.end(),
                 posFlex_fsiBodies_nC_D.begin());
    thrust::copy(other.posFlex_fsiBodies_nD_H.begin(), other.posFlex_fsiBodies_nD_H.end(),
                 posFlex_fsiBodies_nD_D.begin());

    thrust::copy(other.velFlex_fsiBodies_nA_H.begin(), other.velFlex_fsiBodies_nA_H.end(),
                 velFlex_fsiBodies_nA_D.begin());
    thrust::copy(other.velFlex_fsiBodies_nB_H.begin(), other.velFlex_fsiBodies_nB_H.end(),
                 velFlex_fsiBodies_nB_D.begin());
    thrust::copy(other.velFlex_fsiBodies_nC_H.begin(), other.velFlex_fsiBodies_nC_H.end(),
                 velFlex_fsiBodies_nC_D.begin());
    thrust::copy(other.velFlex_fsiBodies_nD_H.begin(), other.velFlex_fsiBodies_nD_H.end(),
                 velFlex_fsiBodies_nD_D.begin());

    thrust::copy(other.accFlex_fsiBodies_nA_H.begin(), other.accFlex_fsiBodies_nA_H.end(),
                 accFlex_fsiBodies_nA_D.begin());
    thrust::copy(other.accFlex_fsiBodies_nB_H.begin(), other.accFlex_fsiBodies_nB_H.end(),
                 accFlex_fsiBodies_nB_D.begin());
    thrust::copy(other.accFlex_fsiBodies_nC_H.begin(), other.accFlex_fsiBodies_nC_H.end(),
                 accFlex_fsiBodies_nC_D.begin());
    thrust::copy(other.accFlex_fsiBodies_nD_H.begin(), other.accFlex_fsiBodies_nD_H.end(),
                 accFlex_fsiBodies_nD_D.begin());
}

void FsiMeshDataD::CopyFromH(const FsiMeshDataH& other) {
    thrust::copy(other.pos_fsi_fea_H.begin(), other.pos_fsi_fea_H.end(), pos_fsi_fea_D.begin());
    thrust::copy(other.vel_fsi_fea_H.begin(), other.vel_fsi_fea_H.end(), vel_fsi_fea_D.begin());
    thrust::copy(other.acc_fsi_fea_H.begin(), other.acc_fsi_fea_H.end(), acc_fsi_fea_D.begin());
}

FsiBodiesDataD& FsiBodiesDataD::operator=(const FsiBodiesDataD& other) {
    if (this == &other) {
        return *this;
    }
    thrust::copy(other.posRigid_fsiBodies_D.begin(), other.posRigid_fsiBodies_D.end(), 
                 posRigid_fsiBodies_D.begin());
    thrust::copy(other.velMassRigid_fsiBodies_D.begin(), other.velMassRigid_fsiBodies_D.end(),
                 velMassRigid_fsiBodies_D.begin());
    thrust::copy(other.accRigid_fsiBodies_D.begin(), other.accRigid_fsiBodies_D.end(), 
                 accRigid_fsiBodies_D.begin());
    thrust::copy(other.q_fsiBodies_D.begin(), other.q_fsiBodies_D.end(), 
                 q_fsiBodies_D.begin());
    thrust::copy(other.omegaVelLRF_fsiBodies_D.begin(), other.omegaVelLRF_fsiBodies_D.end(),
                 omegaVelLRF_fsiBodies_D.begin());
    thrust::copy(other.omegaAccLRF_fsiBodies_D.begin(), other.omegaAccLRF_fsiBodies_D.end(),
                 omegaAccLRF_fsiBodies_D.begin());
    return *this;
}

FsiShellsDataD& FsiShellsDataD::operator=(const FsiShellsDataD& other) {
    if (this == &other) {
        return *this;
    }
    thrust::copy(other.posFlex_fsiBodies_nA_D.begin(), other.posFlex_fsiBodies_nA_D.end(),
                 posFlex_fsiBodies_nA_D.begin());

    thrust::copy(other.posFlex_fsiBodies_nB_D.begin(), other.posFlex_fsiBodies_nB_D.end(),
                 posFlex_fsiBodies_nB_D.begin());
    thrust::copy(other.posFlex_fsiBodies_nC_D.begin(), other.posFlex_fsiBodies_nC_D.end(),
                 posFlex_fsiBodies_nC_D.begin());
    thrust::copy(other.posFlex_fsiBodies_nD_D.begin(), other.posFlex_fsiBodies_nD_D.end(),
                 posFlex_fsiBodies_nD_D.begin());

    thrust::copy(other.velFlex_fsiBodies_nA_D.begin(), other.velFlex_fsiBodies_nA_D.end(),
                 velFlex_fsiBodies_nA_D.begin());
    thrust::copy(other.velFlex_fsiBodies_nB_D.begin(), other.velFlex_fsiBodies_nB_D.end(),
                 velFlex_fsiBodies_nB_D.begin());
    thrust::copy(other.velFlex_fsiBodies_nC_D.begin(), other.velFlex_fsiBodies_nC_D.end(),
                 velFlex_fsiBodies_nC_D.begin());
    thrust::copy(other.velFlex_fsiBodies_nD_D.begin(), other.velFlex_fsiBodies_nD_D.end(),
                 velFlex_fsiBodies_nD_D.begin());

    thrust::copy(other.accFlex_fsiBodies_nA_D.begin(), other.accFlex_fsiBodies_nA_D.end(),
                 posFlex_fsiBodies_nA_D.begin());
    thrust::copy(other.accFlex_fsiBodies_nB_D.begin(), other.accFlex_fsiBodies_nB_D.end(),
                 accFlex_fsiBodies_nB_D.begin());
    thrust::copy(other.accFlex_fsiBodies_nC_D.begin(), other.accFlex_fsiBodies_nC_D.end(),
                 accFlex_fsiBodies_nC_D.begin());
    thrust::copy(other.accFlex_fsiBodies_nD_D.begin(), other.accFlex_fsiBodies_nD_D.end(),
                 accFlex_fsiBodies_nD_D.begin());
    return *this;
}

FsiMeshDataD& FsiMeshDataD::operator=(const FsiMeshDataD& other) {
    if (this == &other) {
        return *this;
    }
    thrust::copy(other.pos_fsi_fea_D.begin(), other.pos_fsi_fea_D.end(), pos_fsi_fea_D.begin());
    thrust::copy(other.vel_fsi_fea_D.begin(), other.vel_fsi_fea_D.end(), vel_fsi_fea_D.begin());
    thrust::copy(other.acc_fsi_fea_D.begin(), other.acc_fsi_fea_D.end(), acc_fsi_fea_D.begin());
    return *this;
}

//---------------------------------------------------------------------------------------
zipIterRigidH FsiBodiesDataH::iterator() {
    return thrust::make_zip_iterator(
        thrust::make_tuple(posRigid_fsiBodies_H.begin(), velMassRigid_fsiBodies_H.begin(), accRigid_fsiBodies_H.begin(),
                           q_fsiBodies_H.begin(), omegaVelLRF_fsiBodies_H.begin(), omegaAccLRF_fsiBodies_H.begin()));
}

void FsiBodiesDataH::resize(size_t s) {
    posRigid_fsiBodies_H.resize(s);
    velMassRigid_fsiBodies_H.resize(s);
    accRigid_fsiBodies_H.resize(s);
    q_fsiBodies_H.resize(s);
    omegaVelLRF_fsiBodies_H.resize(s);
    omegaAccLRF_fsiBodies_H.resize(s);
}

//---------------------------------------------------------------------------------------
void ProximityDataD::resize(size_t s) {
    gridMarkerHashD.resize(s);
    gridMarkerIndexD.resize(s);
    mapOriginalToSorted.resize(s);
}

//---------------------------------------------------------------------------------------
ChronoBodiesDataH::ChronoBodiesDataH(size_t s) {
    resize(s);
}

ChronoShellsDataH::ChronoShellsDataH(size_t s) {
    resize(s);
}

ChronoMeshDataH::ChronoMeshDataH(size_t s) {
    resize(s);
}
zipIterChronoBodiesH ChronoBodiesDataH::iterator() {
    return thrust::make_zip_iterator(thrust::make_tuple(
        pos_ChSystemH.begin(), vel_ChSystemH.begin(),
        acc_ChSystemH.begin(), quat_ChSystemH.begin(),
        omegaVelGRF_ChSystemH.begin(), omegaAccGRF_ChSystemH.begin()));
}

void ChronoBodiesDataH::resize(size_t s) {
    pos_ChSystemH.resize(s);
    vel_ChSystemH.resize(s);
    acc_ChSystemH.resize(s);
    quat_ChSystemH.resize(s);
    omegaVelGRF_ChSystemH.resize(s);
    omegaAccGRF_ChSystemH.resize(s);
}

void ChronoShellsDataH::resize(size_t s) {
    posFlex_ChSystemH_nA_H.resize(s);
    posFlex_ChSystemH_nB_H.resize(s);
    posFlex_ChSystemH_nC_H.resize(s);
    posFlex_ChSystemH_nD_H.resize(s);

    velFlex_ChSystemH_nA_H.resize(s);
    velFlex_ChSystemH_nB_H.resize(s);
    velFlex_ChSystemH_nC_H.resize(s);
    velFlex_ChSystemH_nD_H.resize(s);

    accFlex_ChSystemH_nA_H.resize(s);
    accFlex_ChSystemH_nB_H.resize(s);
    accFlex_ChSystemH_nC_H.resize(s);
    accFlex_ChSystemH_nD_H.resize(s);
}

void ChronoMeshDataH::resize(size_t s) {
    posFlex_ChSystemH_H.resize(s);
    velFlex_ChSystemH_H.resize(s);
    accFlex_ChSystemH_H.resize(s);
}

//---------------------------------------------------------------------------------------

ChSystemFsi_impl::ChSystemFsi_impl() {
    numObjects = chrono_types::make_shared<NumberOfObjects>();
    InitNumObjects();
    sphMarkersD1 = chrono_types::make_shared<SphMarkerDataD>();
    sphMarkersD2 = chrono_types::make_shared<SphMarkerDataD>();
    sortedSphMarkersD = chrono_types::make_shared<SphMarkerDataD>();
    sphMarkersH = chrono_types::make_shared<SphMarkerDataH>();
    fsiBodiesD1 = chrono_types::make_shared<FsiBodiesDataD>();
    fsiBodiesD2 = chrono_types::make_shared<FsiBodiesDataD>();
    fsiBodiesH = chrono_types::make_shared<FsiBodiesDataH>();
    fsiMeshD = chrono_types::make_shared<FsiMeshDataD>();
    fsiMeshH = chrono_types::make_shared<FsiMeshDataH>();
    fsiGeneralData = chrono_types::make_shared<FsiGeneralData>();
    markersProximityD = chrono_types::make_shared<ProximityDataD>();
}

ChSystemFsi_impl::~ChSystemFsi_impl() {}

void ChSystemFsi_impl::AddSphMarker(Real4 pos, Real4 rhoPresMu, Real3 vel, Real3 tauXxYyZz, Real3 tauXyXzYz) {
    sphMarkersH->posRadH.push_back(pos);
    sphMarkersH->velMasH.push_back(vel);
    sphMarkersH->rhoPresMuH.push_back(rhoPresMu);
    sphMarkersH->tauXyXzYzH.push_back(tauXyXzYz);
    sphMarkersH->tauXxYyZzH.push_back(tauXxYyZz);
}
void ChSystemFsi_impl::ArrangeDataManager() {
    thrust::host_vector<Real4> dummyRhoPresMuH = sphMarkersH->rhoPresMuH;
    dummyRhoPresMuH.clear();
}

void ChSystemFsi_impl::InitNumObjects() {
    numObjects->numRigidBodies = 0;      /* Number of rigid bodies */
    numObjects->numFlexBodies1D = 0;     /* Number of 1D Flexible bodies */
    numObjects->numFlexBodies2D = 0;     /* Number of 2D Flexible bodies */
    numObjects->numFlexNodes = 0;        /* Number of FE nodes */
    numObjects->numGhostMarkers = 0;     /* Number of ghost particles */
    numObjects->numHelperMarkers = 0;    /* Number of helper particles */
    numObjects->numFluidMarkers = 0;     /* Number of fluid SPH particles */
    numObjects->numBoundaryMarkers = 0;  /* Number of boundary SPH particles */
    numObjects->startRigidMarkers = 0;   /* Start index of the rigid SPH particles */
    numObjects->startFlexMarkers = 0;    /* Start index of the flexible SPH particles */
    numObjects->numRigid_SphMarkers = 0; /* Number of rigid SPH particles */
    numObjects->numFlex_SphMarkers = 0;  /* Number of flexible SPH particles */
    numObjects->numAllMarkers = 0;       /* Total number of SPH particles */
}

void ChSystemFsi_impl::CalcNumObjects() {
    InitNumObjects();
    size_t rSize = fsiGeneralData->referenceArray.size();
    bool flagRigid = false;
    bool flagFlex = false;
    std::cout << "ChSystemFsi_impl::CalcNumObjects" << std::endl;

    for (size_t i = 0; i < rSize; i++) {
        int4 rComp4 = fsiGeneralData->referenceArray[i];
        int numMerkers = rComp4.y - rComp4.x;

        switch (rComp4.z) {
            case -3:
                numObjects->numHelperMarkers += numMerkers;
                std::cout << "Added " << numMerkers << " helper particles\n";
                break;
            case -2:
                numObjects->numGhostMarkers += numMerkers;
                std::cout << "Added " << numMerkers << " ghost particles\n";
                break;
            case -1:
                numObjects->numFluidMarkers += numMerkers;
                std::cout << "Added " << numMerkers << " fluid particles\n";
                break;
            case 0:
                numObjects->numBoundaryMarkers += numMerkers;
                std::cout << "Added " << numMerkers << " boundary particles\n";
                break;
            case 1:
                numObjects->numRigid_SphMarkers += numMerkers;
                std::cout << "Added " << numMerkers << " rigid particles\n";
                numObjects->numRigidBodies++;
                flagRigid = true;
                break;
            case 2:
                numObjects->numFlex_SphMarkers += numMerkers;
                std::cout << "Added " << numMerkers << " 1D flexible particles\n";
                numObjects->numFlexBodies1D++;
                flagFlex = true;
                break;
            case 3:
                numObjects->numFlex_SphMarkers += numMerkers;
                std::cout << "Added " << numMerkers << " 2D flexible particles\n";
                numObjects->numFlexBodies2D++;
                flagFlex = true;
                break;
            default:
                std::cout << "Error! particle type not defined! Thrown from CalcNumObjects\n";
                break;
        }
    }

    numObjects->numFluidMarkers += numObjects->numGhostMarkers + numObjects->numHelperMarkers;
    numObjects->numAllMarkers = numObjects->numFluidMarkers 
                              + numObjects->numBoundaryMarkers 
                              + numObjects->numRigid_SphMarkers 
                              + numObjects->numFlex_SphMarkers;

    numObjects->startRigidMarkers = 
        (flagRigid) ? (numObjects->numFluidMarkers + numObjects->numBoundaryMarkers) 
                    : numObjects->numAllMarkers;
    numObjects->startFlexMarkers =
        (flagFlex) ? (numObjects->numFluidMarkers + numObjects->numBoundaryMarkers + numObjects->numRigid_SphMarkers)
                   : numObjects->numAllMarkers;

    printf("Number of Helper particles = %zd\n",numObjects->numHelperMarkers);
    printf("Number of Ghost particles = %zd\n",numObjects->numGhostMarkers);
    printf("Number of Fluid particles = %zd\n",numObjects->numFluidMarkers);
    printf("Number of Boundary particles = %zd\n",numObjects->numBoundaryMarkers);
    printf("Number of Rigid particles = %zd\n",numObjects->numRigid_SphMarkers);
    printf("Number of Flexible particles = %zd\n",numObjects->numFlex_SphMarkers);
    printf("Total number particles = %zd\n",numObjects->numAllMarkers);
    printf("Rigid particles start at = %zd\n",numObjects->startRigidMarkers);
    printf("Flexible particles start at = %zd\n",numObjects->startFlexMarkers);
}

void ChSystemFsi_impl::ConstructReferenceArray() {

    CalcNumObjects();

    // determine the number of each component
    if (numObjects->numAllMarkers != sphMarkersH->rhoPresMuH.size()) {
        printf(
            "\nChSystemFsi_impl::ConstructReferenceArray()    numObjects->numAllMarkers=%zd, "
            "sphMarkersH->rhoPresMuH.size()=%zd\n",
            numObjects->numAllMarkers, sphMarkersH->rhoPresMuH.size());
        throw std::runtime_error("Error! numObjects wrong! thrown from ConstructReferenceArray !\n");
    }
    thrust::host_vector<int> numComponentMarkers(numObjects->numAllMarkers);
    thrust::fill(numComponentMarkers.begin(), numComponentMarkers.end(), 1);
    thrust::host_vector<Real4> dummyRhoPresMuH = sphMarkersH->rhoPresMuH;
    thrust::copy(sphMarkersH->rhoPresMuH.begin(), sphMarkersH->rhoPresMuH.end(), dummyRhoPresMuH.begin());
    size_t numberOfComponents =
        (thrust::reduce_by_key(dummyRhoPresMuH.begin(), dummyRhoPresMuH.end(), numComponentMarkers.begin(),
                               dummyRhoPresMuH.begin(), numComponentMarkers.begin(), sphTypeCompEqual()))
            .first - dummyRhoPresMuH.begin();
    printf("Number of particle types = %zd\n", numberOfComponents);

    fsiGeneralData->referenceArray.resize(numberOfComponents);
    dummyRhoPresMuH.resize(numberOfComponents);
    numComponentMarkers.resize(numberOfComponents);
    int savedNumber = 0;
    for (size_t i = 0; i < numberOfComponents; i++) {
        int compType = (int)std::floor(dummyRhoPresMuH[i].w + .1);
        int phaseType = -1;
        if (compType == -3) {
            phaseType = -1; // For helper
        } else if (compType == -2) {
            phaseType = -1; // For ghost
        } else if (compType == -1) {
            phaseType = -1; // For fluid/granular
        } else if (compType == 0) {
            phaseType = 0;  // For boundary
        } else if (compType == 1) {
            phaseType = 1;  // For rigid
        } else if (compType == 2) {
            phaseType = 1;  // For 1D cable elements
        } else if (compType == 3) {
            phaseType = 1;  // For 2D shell elements
        } else {
            phaseType = 1;
        }
        fsiGeneralData->referenceArray[i] = mI4(savedNumber, savedNumber + numComponentMarkers[i], compType, phaseType);
        savedNumber += numComponentMarkers[i];
    }
    dummyRhoPresMuH.clear();
    numComponentMarkers.clear();

    printf("Reference array \n");
    for (size_t i = 0; i < fsiGeneralData->referenceArray.size(); i++) {
        int4 num = fsiGeneralData->referenceArray[i];
        printf("%d %d %d %d \n", num.x, num.y, num.z, num.w);
    }
}

//--------------------------------------------------------------------------------------------------------------------------------
void ChSystemFsi_impl::ResizeDataManager(int numNodes) {
    ConstructReferenceArray();
    if (numObjects->numAllMarkers != sphMarkersH->rhoPresMuH.size()) {
        throw std::runtime_error("Error! numObjects wrong! thrown from FinalizeDataManager !\n");
    }

    numObjects->numFlexNodes = numNodes;

    printf("fsiData->ResizeDataManager (fsiGeneralData)...\n");
    sphMarkersD1->resize(numObjects->numAllMarkers);
    sphMarkersD2->resize(numObjects->numAllMarkers);
    sortedSphMarkersD->resize(numObjects->numAllMarkers);
    sphMarkersH->resize(numObjects->numAllMarkers);
    markersProximityD->resize(numObjects->numAllMarkers);

    fsiGeneralData->derivVelRhoD.resize(numObjects->numAllMarkers);
    fsiGeneralData->derivVelRhoD_old.resize(numObjects->numAllMarkers);

    fsiGeneralData->derivTauXxYyZzD.resize(numObjects->numAllMarkers);
    fsiGeneralData->derivTauXyXzYzD.resize(numObjects->numAllMarkers);

    fsiGeneralData->vel_XSPH_D.resize(numObjects->numAllMarkers);
    fsiGeneralData->vis_vel_SPH_D.resize(numObjects->numAllMarkers, mR3(1e-20));
    fsiGeneralData->sr_tau_I_mu_i.resize(numObjects->numAllMarkers, mR4(1e-20));

    fsiGeneralData->activityIdentifierD.resize(numObjects->numAllMarkers, 1);
    fsiGeneralData->extendedActivityIdD.resize(numObjects->numAllMarkers, 1);

    printf("fsiData->ResizeDataManager (sphMarkersH)...\n");
    thrust::copy(sphMarkersH->posRadH.begin(), sphMarkersH->posRadH.end(), sphMarkersD1->posRadD.begin());
    thrust::copy(sphMarkersH->velMasH.begin(), sphMarkersH->velMasH.end(), sphMarkersD1->velMasD.begin());
    thrust::copy(sphMarkersH->rhoPresMuH.begin(), sphMarkersH->rhoPresMuH.end(), sphMarkersD1->rhoPresMuD.begin());
    thrust::copy(sphMarkersH->tauXxYyZzH.begin(), sphMarkersH->tauXxYyZzH.end(), sphMarkersD1->tauXxYyZzD.begin());
    thrust::copy(sphMarkersH->tauXyXzYzH.begin(), sphMarkersH->tauXyXzYzH.end(), sphMarkersD1->tauXyXzYzD.begin());

    printf("fsiData->ResizeDataManager (sphMarkersD)...\n");
    thrust::copy(sphMarkersD1->posRadD.begin(), sphMarkersD1->posRadD.end(), sphMarkersD2->posRadD.begin());
    thrust::copy(sphMarkersD1->velMasD.begin(), sphMarkersD1->velMasD.end(), sphMarkersD2->velMasD.begin());
    thrust::copy(sphMarkersD1->rhoPresMuD.begin(), sphMarkersD1->rhoPresMuD.end(), sphMarkersD2->rhoPresMuD.begin());
    thrust::copy(sphMarkersD1->tauXxYyZzD.begin(), sphMarkersD1->tauXxYyZzD.end(), sphMarkersD2->tauXxYyZzD.begin());
    thrust::copy(sphMarkersD1->tauXyXzYzD.begin(), sphMarkersD1->tauXyXzYzD.end(), sphMarkersD2->tauXyXzYzD.begin());
    
    printf("fsiData->ResizeDataManager (Rigid)...\n");
    fsiBodiesD1->resize(numObjects->numRigidBodies);
    fsiBodiesD2->resize(numObjects->numRigidBodies);
    fsiBodiesH->resize(numObjects->numRigidBodies);
    fsiGeneralData->rigid_FSI_ForcesD.resize(numObjects->numRigidBodies);
    fsiGeneralData->rigid_FSI_TorquesD.resize(numObjects->numRigidBodies);
    fsiGeneralData->rigidIdentifierD.resize(numObjects->numRigid_SphMarkers);
    fsiGeneralData->rigidSPH_MeshPos_LRF_D.resize(numObjects->numRigid_SphMarkers);
    fsiGeneralData->FlexSPH_MeshPos_LRF_D.resize(numObjects->numFlex_SphMarkers);
    fsiGeneralData->FlexSPH_MeshPos_LRF_H.resize(numObjects->numFlex_SphMarkers);
    printf("numObjects->numRigidBodies = %zd\n", numObjects->numRigidBodies);

    printf("fsiData->ResizeDataManager (Flexible)...\n");
    fsiGeneralData->FlexIdentifierD.resize(numObjects->numFlex_SphMarkers);
    if (fsiGeneralData->CableElementsNodesH.size() != numObjects->numFlexBodies1D) {
        printf("******************************************************************************\n");
        printf("******************************************************************************\n");
        printf("******************************Be Careful**************************************\n");
        printf("There might be 1D Flexible bodies in Chrono that are not a part of ChSystemFSI\n");
        printf("I am going to transfer nodal data for such elements back and forth although they\n");
        printf("are not part of FSI calculation. If you want to have some 1D element that are  \n");
        printf("inside the ChSystem mesh but not FSI system, you can ignore this warning ...\n");
        printf("******************************************************************************\n");
        printf("******************************************************************************\n");
        printf("******************************************************************************\n");
        fsiGeneralData->CableElementsNodes.resize(fsiGeneralData->CableElementsNodesH.size());
    } else
        fsiGeneralData->CableElementsNodes.resize(numObjects->numFlexBodies1D);
    fsiGeneralData->ShellElementsNodes.resize(numObjects->numFlexBodies2D);
    printf("numObjects->numFlexBodies1D = %zd\n", numObjects->numFlexBodies1D);
    printf("numObjects->numFlexBodies2D = %zd\n", numObjects->numFlexBodies2D);
    printf("fsiGeneralData->CableElementsNodesH.size() = %zd\n", fsiGeneralData->CableElementsNodesH.size());
    printf("fsiGeneralData->ShellElementsNodesH.size() = %zd\n", fsiGeneralData->ShellElementsNodesH.size());
    thrust::copy(fsiGeneralData->CableElementsNodesH.begin(), fsiGeneralData->CableElementsNodesH.end(),
                 fsiGeneralData->CableElementsNodes.begin());
    thrust::copy(fsiGeneralData->ShellElementsNodesH.begin(), fsiGeneralData->ShellElementsNodesH.end(),
                 fsiGeneralData->ShellElementsNodes.begin());

    fsiMeshD->resize(numObjects->numFlexNodes);
    fsiMeshH->resize(numObjects->numFlexNodes);
    fsiGeneralData->Flex_FSI_ForcesD.resize(numObjects->numFlexNodes);
}
}  // end namespace fsi
}  // end namespace chrono
